#include "hip/hip_runtime.h"
#include "cudahermite2d.h"


/*******************************************************************
*						  Hermite interpolation					   *
*******************************************************************/
// this computation is used very often, using it as a function greatly reduces code-redundancy
// however, here we have 16 scattered memory accesses
__device__ double device_hermite_mult(double *H, double b[][4], int I[], long int N, double h)
{
	return b[0][0]* H[    I[0]] + b[0][1]* H[    I[1]] + b[1][0]* H[    I[2]] + b[1][1]* H[    I[3]] // Point interpolation
	    + (b[0][2]* H[1*N+I[0]] + b[0][3]* H[1*N+I[1]] + b[1][2]* H[1*N+I[2]] + b[1][3]* H[1*N+I[3]]) * (h)  // dx
	    + (b[2][0]* H[2*N+I[0]] + b[2][1]* H[2*N+I[1]] + b[3][0]* H[2*N+I[2]] + b[3][1]* H[2*N+I[3]]) * (h)  // dy
	    + (b[2][2]* H[3*N+I[0]] + b[2][3]* H[3*N+I[1]] + b[3][2]* H[3*N+I[2]] + b[3][3]* H[3*N+I[3]]) * (h*h);  // dx dy
}


// save memory by not storing the matrix b but computing it in the function
__device__ double device_hermite_mult(double *H, double bX[], double bY[], int I[], long int N, double h)
{
	return bX[0]*bY[0]* H[    I[0]] + bX[1]*bY[0]* H[    I[1]] + bX[0]*bY[1]* H[    I[2]] + bX[1]*bY[1]* H[    I[3]] // Point interpolation
	    + (bX[2]*bY[0]* H[1*N+I[0]] + bX[3]*bY[0]* H[1*N+I[1]] + bX[2]*bY[1]* H[1*N+I[2]] + bX[3]*bY[1]* H[1*N+I[3]]) * (h)  // dx
	    + (bX[0]*bY[2]* H[2*N+I[0]] + bX[1]*bY[2]* H[2*N+I[1]] + bX[0]*bY[3]* H[2*N+I[2]] + bX[1]*bY[3]* H[2*N+I[3]]) * (h)  // dy
	    + (bX[2]*bY[2]* H[3*N+I[0]] + bX[3]*bY[2]* H[3*N+I[1]] + bX[2]*bY[3]* H[3*N+I[2]] + bX[3]*bY[3]* H[3*N+I[3]]) * (h*h);  // dx dy
}



__device__ double device_hermite_interpolate(double *H, double x, double y, int NX, int NY, double h)
{
	//cell index
	int Ix0 = floor(x/h);
	int Iy0 = floor(y/h);
	int Ix1 = Ix0 + 1;
	int Iy1 = Iy0 + 1;
	
	//dx, dy
	double dx = x/h - Ix0;
	double dy = y/h - Iy0;
		
	long int N = NX*NY;
	
	// project into domain, 100 is chosen so that all values are positiv, since negativ values return negativ reminder
	Ix0 = (Ix0+100 * NX)%NX; Iy0 = (Iy0+100 * NY)%NY;
	Ix1 = (Ix1+100 * NX)%NX; Iy1 = (Iy1+100 * NY)%NY;

	// I00, I10, I01, I11 in Vector to shorten function calls
	int I[4] = {Iy0 * NX + Ix0, Iy0 * NX + Ix1, Iy1 * NX + Ix0, Iy1 * NX + Ix1};
	
	double bX[4] = {Hf(dx), Hf(1-dx), Hg(dx), -Hg(1-dx)};
	double bY[4] = {Hf(dy), Hf(1-dy), Hg(dy), -Hg(1-dy)};
	
	return device_hermite_mult(H, bX, bY, I, N, h);
}


__device__ double device_hermite_interpolate_dx(double *H, double x, double y, int NX, int NY, double h)
{
	//cell index
	int Ix0 = floor(x/h);
	int Iy0 = floor(y/h);
	int Ix1 = Ix0 + 1;
	int Iy1 = Iy0 + 1;
	
	//dx, dy
	double dx = x/h - Ix0;
	double dy = y/h - Iy0;
		
	long int N = NX*NY;
	
	// project into domain, 100 is chosen so that all values are positiv, since negativ values return negativ reminder
	Ix0 = (Ix0+100 * NX)%NX; Iy0 = (Iy0+100 * NY)%NY;
	Ix1 = (Ix1+100 * NX)%NX; Iy1 = (Iy1+100 * NY)%NY;
	
	// I00, I10, I01, I11 in Vector to shorten function calls
	int I[4] = {Iy0 * NX + Ix0, Iy0 * NX + Ix1, Iy1 * NX + Ix0, Iy1 * NX + Ix1};
	
	double bX[4] = {Hfx(dx), -Hfx(1-dx), Hgx(dx), Hgx(1-dx)};
	double bY[4] = {Hf(dy), Hf(1-dy), Hg(dy), -Hg(1-dy)};
	
	return device_hermite_mult(H, bX, bY, I, N, h)/h;
}


__device__ double device_hermite_interpolate_dy(double *H, double x, double y, int NX, int NY, double h)
{
	// build up all needed positioning
	//cell index of footpoints
	int Ix0 = floor(x/h); int Iy0 = floor(y/h);
	int Ix1 = Ix0 + 1; int Iy1 = Iy0 + 1;

	//dx, dy, distance to footpoints
	double dx = x/h - Ix0;
	double dy = y/h - Iy0;

	long int N = NX*NY;

	// project into domain, 100 is chosen so that all values are positiv, since negativ values return negativ reminder
	Ix0 = (Ix0+100 * NX)%NX; Iy0 = (Iy0+100 * NY)%NY;
	Ix1 = (Ix1+100 * NX)%NX; Iy1 = (Iy1+100 * NY)%NY;
	
	// I00, I10, I01, I11 in Vector to shorten function calls
	int I[4] = {Iy0 * NX + Ix0, Iy0 * NX + Ix1, Iy1 * NX + Ix0, Iy1 * NX + Ix1};
	
	double bX[4] = {Hf(dx), Hf(1-dx), Hg(dx), -Hg(1-dx)};
	double bY[4] = {Hfx(dy), -Hfx(1-dy), Hgx(dy), Hgx(1-dy)};
	
	return device_hermite_mult(H, bX, bY, I, N, h)/h;
}


__device__ void device_hermite_interpolate_dx_dy(double *H, double x, double y, double *fx, double *fy, int NX, int NY, double h)
{
	*fx = device_hermite_interpolate_dx(H, x, y, NX, NY, h);
	*fy = device_hermite_interpolate_dy(H, x, y, NX, NY, h);
}


// special function for map advection to compute dx and dy directly at the same positions
__device__ void device_hermite_interpolate_dx_dy_1(double *H1, double x, double y, double *u1, double *v1, int NX, int NY, double h)
{
	// build up all needed positioning
	//cell index of footpoints
	int Ix0 = floor(x/h); int Iy0 = floor(y/h);
	int Ix1 = Ix0 + 1; int Iy1 = Iy0 + 1;

	//dx, dy, distance to footpoints
	double dx = x/h - Ix0;
	double dy = y/h - Iy0;

	long int N = NX*NY;

	// project into domain, 100 is chosen so that all values are positiv, since negativ values return negativ reminder
	Ix0 = (Ix0+100 * NX)%NX; Iy0 = (Iy0+100 * NY)%NY;
	Ix1 = (Ix1+100 * NX)%NX; Iy1 = (Iy1+100 * NY)%NY;

	// I00, I10, I01, I11 in Vector to shorten function calls
	int I[4] = {Iy0 * NX + Ix0, Iy0 * NX + Ix1, Iy1 * NX + Ix0, Iy1 * NX + Ix1};

	// computing all dx-interpolations, giving -v
	{
		double bX[4] = {Hfx(dx), -Hfx(1-dx), Hgx(dx), Hgx(1-dx)};
		double bY[4] = {Hf(dy), Hf(1-dy), Hg(dy), -Hg(1-dy)};

		*v1 =  -device_hermite_mult(H1, bX, bY, I, N, h)/h;

	}
	// compute all dy-interpolations, giving u
	{
		double bX[4] = {Hf(dx), Hf(1-dx), Hg(dx), -Hg(1-dx)};
		double bY[4] = {Hfx(dy), -Hfx(1-dy), Hgx(dy), Hgx(1-dy)};

		*u1 = 	device_hermite_mult(H1, bX, bY, I, N, h)/h;
	}
}


// make it easier for RKthree, avoid redundant operations, naming is set to u and v to avoid confusion
__device__ void  device_hermite_interpolate_dx_dy_3(double *H1, double *H2, double *H3, double x, double y, double *u1, double *v1, double *u2, double *v2, double *u3, double *v3, int NX, int NY, double h)
{
	// build up all needed positioning
	//cell index of footpoints
	int Ix0 = floor(x/h); int Iy0 = floor(y/h);
	int Ix1 = Ix0 + 1; int Iy1 = Iy0 + 1;

	//dx, dy, distance to footpoints
	double dx = x/h - Ix0;
	double dy = y/h - Iy0;

	long int N = NX*NY;

	// project into domain, 100 is chosen so that all values are positiv, since negativ values return negativ reminder
	Ix0 = (Ix0+100 * NX)%NX; Iy0 = (Iy0+100 * NY)%NY;
	Ix1 = (Ix1+100 * NX)%NX; Iy1 = (Iy1+100 * NY)%NY;

	// I00, I10, I01, I11 in Vector to shorten function calls
	int I[4] = {Iy0 * NX + Ix0, Iy0 * NX + Ix1, Iy1 * NX + Ix0, Iy1 * NX + Ix1};

	// computing all dx-interpolations, giving -v
	{
		double bX[4] = {Hfx(dx), -Hfx(1-dx), Hgx(dx), Hgx(1-dx)};
		double bY[4] = {Hf(dy), Hf(1-dy), Hg(dy), -Hg(1-dy)};
	
		// building b here is faster, as we only have to do it once for all three computations
		double b[4][4] = {
							bX[0]*bY[0], bX[1]*bY[0], bX[2]*bY[0], bX[3]*bY[0],
							bX[0]*bY[1], bX[1]*bY[1], bX[2]*bY[1], bX[3]*bY[1],
							bX[0]*bY[2], bX[1]*bY[2], bX[2]*bY[2], bX[3]*bY[2],
							bX[0]*bY[3], bX[1]*bY[3], bX[2]*bY[3], bX[3]*bY[3]
						};
		*v1 = -device_hermite_mult(H1, b, I, N, h)/h;
		*v2 = -device_hermite_mult(H2, b, I, N, h)/h;
		*v3 = -device_hermite_mult(H3, b, I, N, h)/h;
	}
	// compute all dy-interpolations, giving u
	{
		double bX[4] = {Hf(dx), Hf(1-dx), Hg(dx), -Hg(1-dx)};
		double bY[4] = {Hfx(dy), -Hfx(1-dy), Hgx(dy), Hgx(1-dy)};
	
		// building b here is faster, as we only have to do it once for all three computations
		double b[4][4] = {
							bX[0]*bY[0], bX[1]*bY[0], bX[2]*bY[0], bX[3]*bY[0],
							bX[0]*bY[1], bX[1]*bY[1], bX[2]*bY[1], bX[3]*bY[1],
							bX[0]*bY[2], bX[1]*bY[2], bX[2]*bY[2], bX[3]*bY[2],
							bX[0]*bY[3], bX[1]*bY[3], bX[2]*bY[3], bX[3]*bY[3]
						};
		*u1 = 	device_hermite_mult(H1, b, I, N, h)/h;
		*u2 = 	device_hermite_mult(H2, b, I, N, h)/h;
		*u3 = 	device_hermite_mult(H3, b, I, N, h)/h;
	}
}


//diffeomorphisms provide warped interpolations with a jump at the boundaries
__device__ void  device_diffeo_interpolate(double *Hx, double *Hy, double x, double y, double *x2,  double *y2, int NX, int NY, double h)
{
	// build up all needed positioning
	// cell index of footpoints
	int Ix0 = floor(x/h); int Iy0 = floor(y/h);
	int Ix1 = Ix0 + 1; int Iy1 = Iy0 + 1;

	//dx, dy, distance to footpoints
	double dx = x/h - Ix0;
	double dy = y/h - Iy0;
		
	long int N = NX*NY;
	
	//warping, compute projection needed to map onto LX/LY domain, add 1 for negative values to accommodate sign
	int Ix0W = Ix0/NX - (Ix0 < 0); int Ix1W = Ix1/NX - (Ix1 < 0);
	int Iy0W = Iy0/NY - (Iy0 < 0); int Iy1W = Iy1/NY - (Iy1 < 0);

	//jump on warping
	double LX = NX*h; double LY = NY*h;

	// project back into domain
	Ix0 -= Ix0W*NX; Iy0 -= Iy0W*NY; Ix1 -= Ix1W*NX; Iy1 -= Iy1W*NY;
	
	// I00, I10, I01, I11 in Vector to shorten function calls
	int I[4] = {Iy0 * NX + Ix0, Iy0 * NX + Ix1, Iy1 * NX + Ix0, Iy1 * NX + Ix1};
	
	double bX[4] = {Hf(dx), Hf(1-dx), Hg(dx), -Hg(1-dx)};
	double bY[4] = {Hf(dy), Hf(1-dy), Hg(dy), -Hg(1-dy)};
	
	
	double b[4][4] = {
						bX[0]*bY[0], bX[1]*bY[0], bX[2]*bY[0], bX[3]*bY[0],
						bX[0]*bY[1], bX[1]*bY[1], bX[2]*bY[1], bX[3]*bY[1],
						bX[0]*bY[2], bX[1]*bY[2], bX[2]*bY[2], bX[3]*bY[2],
						bX[0]*bY[3], bX[1]*bY[3], bX[2]*bY[3], bX[3]*bY[3]
					};
	
	*x2 =  b[0][0]* (Hx[I[0]] + Ix0W*LX) + b[0][1]* (Hx[I[1]] + Ix1W*LX) + b[1][0]* (Hx[I[2]] + Ix0W*LX) + b[1][1]* (Hx[I[3]] + Ix1W*LX) // Point interpolation
	    + (b[0][2]*  Hx[1*N+I[0]]        + b[0][3]*  Hx[1*N+I[1]]        + b[1][2]*  Hx[1*N+I[2]]        + b[1][3]*  Hx[1*N+I[3]]) * (h)  // dx
	    + (b[2][0]*  Hx[2*N+I[0]]        + b[2][1]*  Hx[2*N+I[1]]        + b[3][0]*  Hx[2*N+I[2]]        + b[3][1]*  Hx[2*N+I[3]]) * (h)  // dy
	    + (b[2][2]*  Hx[3*N+I[0]]        + b[2][3]*  Hx[3*N+I[1]]        + b[3][2]*  Hx[3*N+I[2]]        + b[3][3]*  Hx[3*N+I[3]]) * (h*h);  // dx dy

	*y2 =  b[0][0]* (Hy[I[0]] + Iy0W*LY) + b[0][1]* (Hy[I[1]] + Iy0W*LY) + b[1][0]* (Hy[I[2]] + Iy1W*LY) + b[1][1]* (Hy[I[3]] + Iy1W*LY) // Point interpolation
	    + (b[0][2]*  Hy[1*N+I[0]]        + b[0][3]*  Hy[1*N+I[1]]        + b[1][2]*  Hy[1*N+I[2]]        + b[1][3]*  Hy[1*N+I[3]]) * (h)  // dx
	    + (b[2][0]*  Hy[2*N+I[0]]        + b[2][1]*  Hy[2*N+I[1]]        + b[3][0]*  Hy[2*N+I[2]]        + b[3][1]*  Hy[2*N+I[3]]) * (h)  // dy
	    + (b[2][2]*  Hy[3*N+I[0]]        + b[2][3]*  Hy[3*N+I[1]]        + b[3][2]*  Hy[3*N+I[2]]        + b[3][3]*  Hy[3*N+I[3]]) * (h*h);  // dx dy
}


// compute determinant of gradient of flowmap
__device__ double  device_diffeo_grad(double *Hx, double *Hy, double x, double y, int NX, int NY, double h)																							// time cost
{
	// build up all needed positioning
	// cell index of footpoints
	int Ix0 = floor(x/h); int Iy0 = floor(y/h);
	int Ix1 = Ix0 + 1; int Iy1 = Iy0 + 1;

	//dx, dy, distance to footpoints
	double dx = x/h - Ix0;
	double dy = y/h - Iy0;
		
	long int N = NX*NY;
	
	//warping, compute projection needed to map onto LX/LY domain, add 1 for negative values to accommodate sign
	int Ix0W = Ix0/NX - (Ix0 < 0); int Ix1W = Ix1/NX - (Ix1 < 0);
	int Iy0W = Iy0/NY - (Iy0 < 0); int Iy1W = Iy1/NY - (Iy1 < 0);

	//jump on warping
	double LX = NX*h; double LY = NY*h;

	// project back into domain
	Ix0 -= Ix0W*NX; Iy0 -= Iy0W*NY; Ix1 -= Ix1W*NX; Iy1 -= Iy1W*NY;

	// I00, I10, I01, I11 in Vector to shorten function calls
	int I[4] = {Iy0 * NX + Ix0, Iy0 * NX + Ix1, Iy1 * NX + Ix0, Iy1 * NX + Ix1};
	
	double Xx, Xy, Yx, Yy;  // fx/dx, fx/dy fy/dx fy/dy
	// compute x derivatives
	{
		double bX[4] = {Hfx(dx), -Hfx(1-dx), Hgx(dx), Hgx(1-dx)};
		double bY[4] = {Hf(dy), Hf(1-dy), Hg(dy), -Hg(1-dy)};

		double b[4][4] = {
							bX[0]*bY[0], bX[1]*bY[0], bX[2]*bY[0], bX[3]*bY[0],
							bX[0]*bY[1], bX[1]*bY[1], bX[2]*bY[1], bX[3]*bY[1],
							bX[0]*bY[2], bX[1]*bY[2], bX[2]*bY[2], bX[3]*bY[2],
							bX[0]*bY[3], bX[1]*bY[3], bX[2]*bY[3], bX[3]*bY[3]
						};

		Xx =  (b[0][0]* (Hx[I[0]] + Ix0W*LX) + b[0][1]* (Hx[I[1]] + Ix1W*LX) + b[1][0]* (Hx[I[2]] + Ix0W*LX) + b[1][1]* (Hx[I[3]] + Ix1W*LX) // Point interpolation
			+ (b[0][2]*  Hx[1*N+I[0]]        + b[0][3]*  Hx[1*N+I[1]]        + b[1][2]*  Hx[1*N+I[2]]        + b[1][3]*  Hx[1*N+I[3]]) * (h)  // dx
			+ (b[2][0]*  Hx[2*N+I[0]]        + b[2][1]*  Hx[2*N+I[1]]        + b[3][0]*  Hx[2*N+I[2]]        + b[3][1]*  Hx[2*N+I[3]]) * (h)  // dy
			+ (b[2][2]*  Hx[3*N+I[0]]        + b[2][3]*  Hx[3*N+I[1]]        + b[3][2]*  Hx[3*N+I[2]]        + b[3][3]*  Hx[3*N+I[3]]) * (h*h))/h;  // dx dy

		Yx =  (b[0][0]* (Hy[I[0]] + Iy0W*LY) + b[0][1]* (Hy[I[1]] + Iy0W*LY) + b[1][0]* (Hy[I[2]] + Iy1W*LY) + b[1][1]* (Hy[I[3]] + Iy1W*LY) // Point interpolation
			+ (b[0][2]*  Hy[1*N+I[0]]        + b[0][3]*  Hy[1*N+I[1]]        + b[1][2]*  Hy[1*N+I[2]]        + b[1][3]*  Hy[1*N+I[3]]) * (h)  // dx
			+ (b[2][0]*  Hy[2*N+I[0]]        + b[2][1]*  Hy[2*N+I[1]]        + b[3][0]*  Hy[2*N+I[2]]        + b[3][1]*  Hy[2*N+I[3]]) * (h)  // dy
			+ (b[2][2]*  Hy[3*N+I[0]]        + b[2][3]*  Hy[3*N+I[1]]        + b[3][2]*  Hy[3*N+I[2]]        + b[3][3]*  Hy[3*N+I[3]]) * (h*h))/h;  // dx dy
	}
	// compute y derivatives
	{
		double bX[4] = {Hf(dx), Hf(1-dx), Hg(dx), -Hg(1-dx)};
		double bY[4] = {Hfx(dy), -Hfx(1-dy), Hgx(dy), Hgx(1-dy)};

		double b[4][4] = {
							bX[0]*bY[0], bX[1]*bY[0], bX[2]*bY[0], bX[3]*bY[0],
							bX[0]*bY[1], bX[1]*bY[1], bX[2]*bY[1], bX[3]*bY[1],
							bX[0]*bY[2], bX[1]*bY[2], bX[2]*bY[2], bX[3]*bY[2],
							bX[0]*bY[3], bX[1]*bY[3], bX[2]*bY[3], bX[3]*bY[3]
						};

		Xy =  (b[0][0]* (Hx[I[0]] + Ix0W*LX) + b[0][1]* (Hx[I[1]] + Ix1W*LX) + b[1][0]* (Hx[I[2]] + Ix0W*LX) + b[1][1]* (Hx[I[3]] + Ix1W*LX) // Point interpolation
			+ (b[0][2]*  Hx[1*N+I[0]]        + b[0][3]*  Hx[1*N+I[1]]        + b[1][2]*  Hx[1*N+I[2]]        + b[1][3]*  Hx[1*N+I[3]]) * (h)  // dx
			+ (b[2][0]*  Hx[2*N+I[0]]        + b[2][1]*  Hx[2*N+I[1]]        + b[3][0]*  Hx[2*N+I[2]]        + b[3][1]*  Hx[2*N+I[3]]) * (h)  // dy
			+ (b[2][2]*  Hx[3*N+I[0]]        + b[2][3]*  Hx[3*N+I[1]]        + b[3][2]*  Hx[3*N+I[2]]        + b[3][3]*  Hx[3*N+I[3]]) * (h*h))/h;  // dx dy

		Yy =  (b[0][0]* (Hy[I[0]] + Iy0W*LY) + b[0][1]* (Hy[I[1]] + Iy0W*LY) + b[1][0]* (Hy[I[2]] + Iy1W*LY) + b[1][1]* (Hy[I[3]] + Iy1W*LY) // Point interpolation
			+ (b[0][2]*  Hy[1*N+I[0]]        + b[0][3]*  Hy[1*N+I[1]]        + b[1][2]*  Hy[1*N+I[2]]        + b[1][3]*  Hy[1*N+I[3]]) * (h)  // dx
			+ (b[2][0]*  Hy[2*N+I[0]]        + b[2][1]*  Hy[2*N+I[1]]        + b[3][0]*  Hy[2*N+I[2]]        + b[3][1]*  Hy[2*N+I[3]]) * (h)  // dy
			+ (b[2][2]*  Hy[3*N+I[0]]        + b[2][3]*  Hy[3*N+I[1]]        + b[3][2]*  Hy[3*N+I[2]]        + b[3][3]*  Hy[3*N+I[3]]) * (h*h))/h;  // dx dy
	}
					
	return Xx*Yy - Xy*Yx;				
}





/******************************************************************/
/*******************************************************************
*							   Old								   *
*******************************************************************/
/******************************************************************/



void hermite_interpolation_test()
{
}


__global__ void kernel_hermite_interpolation(double *H, double *F, int NXH, int NYH, int NXF, int NYF, double hH, double hF)
{
}





