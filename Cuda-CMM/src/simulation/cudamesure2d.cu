#include "hip/hip_runtime.h"
#include "cudamesure2d.h"


void Compute_Energy(double *E, double *psi, TCudaGrid2D Grid){
	// parallel reduction using thrust
	thrust::device_ptr<double> psi_ptr = thrust::device_pointer_cast(psi);
	*E = 0.5*Grid.h*Grid.h * thrust::transform_reduce(psi_ptr + Grid.N, psi_ptr + 3*Grid.N, thrust::square<double>(), 0.0, thrust::plus<double>());
}


void Compute_Enstrophy(double *E, double *W, TCudaGrid2D Grid){
	// parallel reduction using thrust
	thrust::device_ptr<double> W_ptr = thrust::device_pointer_cast(W);
	*E = 0.5*Grid.h*Grid.h * thrust::transform_reduce(W_ptr, W_ptr + Grid.N, thrust::square<double>(), 0.0, thrust::plus<double>());
}


// compute palinstrophy using fourier transformations - a bit expensive with two temporary arrays but ca marche
void Compute_Palinstrophy(TCudaGrid2D Grid, double *Pal, double *W_real, hipfftDoubleComplex *Dev_Temp_C1, hipfftDoubleComplex *Dev_Temp_C2, hipfftHandle cufftPlan){
	// round 1: dx dervative
	k_real_to_comp<<<Grid.blocksPerGrid, Grid.threadsPerBlock>>>(W_real, Dev_Temp_C1, Grid.NX, Grid.NY);
	hipfftExecZ2Z(cufftPlan, Dev_Temp_C1, Dev_Temp_C2, HIPFFT_FORWARD);
	k_normalize<<<Grid.blocksPerGrid, Grid.threadsPerBlock>>>(Dev_Temp_C2, Grid.NX, Grid.NY);
	k_fft_dx<<<Grid.blocksPerGrid, Grid.threadsPerBlock>>>(Dev_Temp_C2, Dev_Temp_C1, Grid.NX, Grid.NY, Grid.h);
	hipfftExecZ2Z(cufftPlan, Dev_Temp_C1, Dev_Temp_C2, HIPFFT_BACKWARD);

	comp_to_real(Dev_Temp_C2, (hipfftDoubleReal*)Dev_Temp_C1, Grid.N);

	// parallel reduction using thrust
	thrust::device_ptr<double> Pal_ptr = thrust::device_pointer_cast((hipfftDoubleReal*)Dev_Temp_C1);
	*Pal = 0.5*Grid.h*Grid.h * thrust::transform_reduce(Pal_ptr, Pal_ptr + Grid.N, thrust::square<double>(), 0.0, thrust::plus<double>());

	// round 2: dy dervative
	k_real_to_comp<<<Grid.blocksPerGrid, Grid.threadsPerBlock>>>(W_real, Dev_Temp_C1, Grid.NX, Grid.NY);
	hipfftExecZ2Z(cufftPlan, Dev_Temp_C1, Dev_Temp_C2, HIPFFT_FORWARD);
	k_normalize<<<Grid.blocksPerGrid, Grid.threadsPerBlock>>>(Dev_Temp_C2, Grid.NX, Grid.NY);
	k_fft_dy<<<Grid.blocksPerGrid, Grid.threadsPerBlock>>>(Dev_Temp_C2, Dev_Temp_C1, Grid.NX, Grid.NY, Grid.h);
	hipfftExecZ2Z(cufftPlan, Dev_Temp_C1, Dev_Temp_C2, HIPFFT_BACKWARD);

	comp_to_real(Dev_Temp_C2, (hipfftDoubleReal*)Dev_Temp_C1, Grid.N);

	// parallel reduction using thrust
	*Pal += 0.5*Grid.h*Grid.h * thrust::transform_reduce(Pal_ptr, Pal_ptr + Grid.N, thrust::square<double>(), 0.0, thrust::plus<double>());

//	printf("Pal : %f\n", *Pal);
}


//// compute palinstrophy using hermite array - cheap, but we need the vorticity hermite for that, only works for fine array, can be kernelized too actually, but not now
//void Compute_Palinstrophy_hermite(TCudaGrid2D *Grid_fine, double *Pal, double *W_H_real){
//
//	// dx and dy values are in position 2/4 and 3/4, copy to host
//	double *Host_W_coarse_real_dx_dy = new double[2*Grid_fine.N];
//	hipMemcpy(Host_W_coarse_real_dx_dy, &W_H_real[Grid_fine.N], 2*Grid_fine.N, hipMemcpyDeviceToHost);
//	hipMemcpy(Host_W_coarse_real_dx_dy, &W_H_real[Grid_fine.N], 2*Grid_fine.N, hipMemcpyDeviceToHost);
//	hipMemcpy(Host_W_coarse_real_dx_dy, &W_H_real[Grid_fine.N], 2*Grid_fine.N, hipMemcpyDeviceToHost);
//
//	// now compute actual palinstrophy and add everything together
//    for(int i = 0; i < Grid_fine.N; i+=1){
//		*Pal += 0.5 * (Grid_fine.h) * (Grid_fine.h) * (Host_W_coarse_real_dx_dy[i] * Host_W_coarse_real_dx_dy[i] + Host_W_coarse_real_dx_dy[i + Grid_fine.N] * Host_W_coarse_real_dx_dy[i + Grid_fine.N]);
//	}
//}



/*******************************************************************
*							 Fourier							   *
*******************************************************************/



// Non-uniform discrete Fourier transform in 1D
void NDFT_1D(hipfftDoubleComplex *X_k, double *x_n, double *p_n, double *f_k, int N){
	
	// X_{k} = \sum_{n=0}^{N-1} x_{n} e^{-2\pi i p_n f_k} 
	// X_k is a complex in Fourier space 	; 	x_n are the values of the function in real space 	; 	p_n \in [0,1] are the sample points ; f_k \in [0,N] are frequencies
	
	for(int k = 0; k < N; k+=1){
		X_k[k].x = 0;
		X_k[k].y = 0;
		for(int n = 0; n < N; n+=1){
			// X_k[k] += x_n[n]*(exp(-2*i*PI*p_n[n]*f_k[k]))/N
			X_k[k].x +=  x_n[n]*cos(twoPI*p_n[n]*f_k[k])/N;
			X_k[k].y += -x_n[n]*sin(twoPI*p_n[n]*f_k[k])/N;
		}
	}
}


// Non-uniform inverse discrete Fourier transform in 1D
void iNDFT_1D(hipfftDoubleComplex *X_k, double *x_n, double *p_n, double *f_k, int N){
	
	// X_k is a complex in Fourier space 	; 	x_n are the values of the function in real space 	; 	p_n \in [0,1] are the sample points ; f_k \in [0,N] are frequencies
	
	for(int n = 0; n < N; n+=1){
		x_n[n] = 0;
		for(int k = 0; k < N; k+=1){
			// x_n[n] += X_k[k]*(exp(2*i*PI*p_n[n]*f_k[k]))
			x_n[n] +=  X_k[k].x*cos(twoPI*p_n[n]*f_k[k]);
			x_n[n] += -X_k[k].y*sin(twoPI*p_n[n]*f_k[k]);
			/*
			x_n[n].x +=  X_k[k].x*cos(twoPI*p_n[n]*f_k[k]);
			x_n[n].x += -X_k[k].y*sin(twoPI*p_n[n]*f_k[k]);
			x_n[n].y +=  X_k[k].x*sin(twoPI*p_n[n]*f_k[k]);
			x_n[n].y +=  X_k[k].y*cos(twoPI*p_n[n]*f_k[k]);
			*/
		}
	}
}


// Non-uniform discrete Fourier transform in 2D
__global__ void NDFT_2D(hipfftDoubleComplex *X_k, double *x_n, double *p_n, int *f_k, int NX, int Np){
	
	// X_{k} = \sum_{n=0}^{N-1} x_{n} e^{-2\pi i p_n . f_k} 
	// X_k is a complex in Fourier space 	; 	x_n are the values of the function in real space 	; 	p_n \in [0,1] are the sample points 	; 	f_k \in [0,N-1] are frequencies
	
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NX || iY >= NX)
		return;
	
	int In = iY*NX + iX;
	
	X_k[In].x = 0;
	X_k[In].y = 0;
	
	for(int n = 0; n < Np; n+=1){
		// X_k[k] += x_n[n]*(exp(-2*i*PI*p_n[n]*f_k[k]))/(N*N)
		X_k[In].x +=  x_n[n] * cos(twoPI * ( p_n[2*n]*f_k[iX] + p_n[2*n+1]*f_k[iY] ));
		X_k[In].y += -x_n[n] * sin(twoPI * ( p_n[2*n]*f_k[iX] + p_n[2*n+1]*f_k[iY] ));
	}
	X_k[In].x = X_k[In].x/(NX*NX);
	X_k[In].y = X_k[In].y/(NX*NX);
	
}


// Non-uniform inverse discrete Fourier transform in 2D
__global__ void iNDFT_2D(hipfftDoubleComplex *X_k, double *x_n, double *p_n, int *f_k, int N_grid){
	
	// X_k is a complex in Fourier space 	; 	x_n are the values of the function in real space 	; 	p_n \in [0,1] are the sample points 	; 	f_k \in [0,N-1] are frequencies
	
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	
	x_n[iX] = 0;
	
	for(int kx = 0; kx < N_grid; kx+=1){
		for(int ky = 0; ky < N_grid; ky+=1){
			x_n[iX] +=  X_k[ky*N_grid + kx].x * cos(twoPI * ( p_n[2*iX]*f_k[kx] + p_n[2*iX+1]*f_k[ky] ));
			x_n[iX] += -X_k[ky*N_grid + kx].y * sin(twoPI * ( p_n[2*iX]*f_k[kx] + p_n[2*iX+1]*f_k[ky] ));
			/*
			x_n[iX].x  =  X_k[ky*NX + kx].x * cos(twoPI * ( p_n[iX]*f_k[kx] + p_n[iX]*f_k[ky] ));
			x_n[iX].x += -X_k[ky*NX + kx].y * sin(twoPI * ( p_n[iX]*f_k[kx] + p_n[iX]*f_k[ky] ));
			x_n[iX].y  =  X_k[ky*NX + kx].x * sin(twoPI * ( p_n[iX]*f_k[kx] + p_n[iX]*f_k[ky] ));
			x_n[iX].y +=  X_k[ky*NX + kx].y * cos(twoPI * ( p_n[iX]*f_k[kx] + p_n[iX]*f_k[ky] ));
			*/
		}
	}
}


void Laplacian_vort(TCudaGrid2D Grid_fine, double *Dev_W_fine, hipfftDoubleComplex *Dev_Complex_fine, hipfftDoubleComplex *Dev_Hat_fine, double *Dev_lap_fine_real, hipfftDoubleComplex *Dev_lap_fine_complex, hipfftDoubleComplex *Dev_lap_fine_hat, hipfftHandle cufftPlan_fine){

    real_to_comp(Dev_W_fine, Dev_Complex_fine, Grid_fine.N);
    hipfftExecZ2Z(cufftPlan_fine, Dev_Complex_fine, Dev_Hat_fine, HIPFFT_FORWARD);
    k_normalize<<<Grid_fine.blocksPerGrid, Grid_fine.threadsPerBlock>>>(Dev_Complex_fine, Grid_fine.NX, Grid_fine.NY);

    k_fft_lap<<<Grid_fine.blocksPerGrid, Grid_fine.threadsPerBlock>>>(Dev_Hat_fine, Dev_lap_fine_hat, Grid_fine.NX, Grid_fine.NY, Grid_fine.h);
    hipfftExecZ2Z(cufftPlan_fine, Dev_lap_fine_hat, Dev_lap_fine_complex, HIPFFT_BACKWARD);
    comp_to_real(Dev_lap_fine_complex, Dev_lap_fine_real, Grid_fine.N);

}

__host__ __device__ double L1(double t, double tp, double tm, double tmm){
    return ((t-tm)*(t-tmm)/((tp-tm)*(tp-tmm)));
}

__host__ __device__ double L2(double t, double tp, double tm, double tmm){
    return ((t-tp)*(t-tmm)/((tm-tp)*(tm-tmm)));
}

__host__ __device__ double L3(double t, double tp, double tm, double tmm){
    return ((t-tp)*(t-tm)/((tmm-tp)*(tmm-tm)));
}
