#include "hip/hip_runtime.h"
#include "settings.h"

void SettingsCMM::setPresets() {
	// grid settings for coarse and fine grid
	// 32		64		128		256		512		1024		2048		4096		8192		16384
	// max working on V100 : grid_scale = 4096; fine_grid_scale = 16384;
	int grid_coarse = 512;
	int grid_fine = 2048;
	int grid_psi = 1024;  // psi will be upsampled to this grid

	/*
	 *  Initial conditions
	 *  "4_nodes" 				-	flow containing exactly 4 fourier modes with two vortices
	 *  "quadropole"			-	???
	 *  "three_vortices"		-	???
	 *  "single_shear_layer"	-	shear layer problem forming helmholtz-instabilities, merging into two vortices which then merges into one big vortex
	 *  "two_vortices"			-	???
	 *  "turbulence_gaussienne"	-	???
	 */
	string initial_condition = "4_nodes";

	// set minor properties
	double incomp_threshhold = 1e-4;  // the maximum allowance of map to deviate from grad_chi begin 1
	double map_epsilon = 1e-4;  // distance used for foot points for GALS map advection

	// set memory properties
	int mem_RAM_GPU_remaps = 128;  // mem_index in MB on the GPU
	int mem_RAM_CPU_remaps = 4096;  // mem_RAM_CPU_remaps in MB on the CPU
	int Nb_array_RAM = 4;  // fixed for four different stacks

	// set specific settings
	// Time integration, define by name, "RKThree", "ABTwo", "EulerExp", "RKFour"
	string time_integration = "RKThree";

	// mapupdate order, "2nd", "3rd", "4th"
	string map_update_order = "3rd";

	// mollification settings, stencil size, 0, 4, 8
	int molly_stencil = 4;


	// now set everything
	setGridCoarse(grid_coarse);
	setGridFine(grid_fine);
	setGridPsi(grid_psi);
	setInitialCondition(initial_condition);
	setIncompThreshold(incomp_threshhold);
	setMapEpsilon(map_epsilon);
	setMemRamGpuRemaps(mem_RAM_GPU_remaps);
	setMemRamCpuRemaps(mem_RAM_CPU_remaps);
	setNbArrayRam(Nb_array_RAM);
	setTimeIntegration(time_integration);
	setMapUpdateOrder(map_update_order);
	setMollyStencil(molly_stencil);
}


/*
 *  Function to apply values taken from command line
 *  general form:  COMMAND=VALUE
 */
void SettingsCMM::applyCommands(int argc, char *args[]) {
	// loop over all commands
	for( int count = 0; count < argc; count++ ) {
		// construct string for command
		string command_full = args[count];
		// check for = sign
		int pos_equal = command_full.find("=");
		if (pos_equal != string::npos) {
			// construct two substrings
			string command = command_full.substr(0, pos_equal);
			string value = command_full.substr(pos_equal+1, command_full.length());

			// big if else for different commands
			if (command == "grid_coarse") setGridCoarse(stoi(value));
			else if (command == "grid_fine") setGridFine(stoi(value));
			else if (command == "grid_psi") setGridPsi(stoi(value));
			else if (command == "initial_condition") setInitialCondition(value);
			else if (command == "incomp_threshold") setIncompThreshold(stoi(value));
			else if (command == "map_epsilon") setMapEpsilon(stoi(value));
			else if (command == "mem_RAM_GPU_remaps") setMemRamGpuRemaps(stoi(value));
			else if (command == "mem_RAM_CPU_remaps") setMemRamCpuRemaps(stoi(value));
			else if (command == "Nb_array_RAM") setNbArrayRam(stoi(value));
			else if (command == "time_integration") setTimeIntegration(value);
			else if (command == "map_update_order") setMapUpdateOrder(value);
			else if (command == "molly_stencil") setMollyStencil(stoi(value));
		}
	}
	//	 cout << "  args[" << count << "]   " << args[count] << "\n";
}


// class constructor from three main ingredients
SettingsCMM::SettingsCMM(int gridCoarse, int gridFine, string initialCondition) {
	// set presets
	setPresets();
	// override the three main components
	setGridCoarse(gridCoarse);
	setGridFine(gridFine);
	setInitialCondition(initialCondition);
	// assume psi will not be upsampled
	setGridPsi(gridCoarse);
}


// class constructor to build from presets
SettingsCMM::SettingsCMM() {
	// set presets
	setPresets();
}


// class constructor to take into account command line inputs
SettingsCMM::SettingsCMM(int argc, char *args[]) {
	// set presets
	setPresets();
	// override presets with command line arguments
	applyCommands(argc, args);
}
