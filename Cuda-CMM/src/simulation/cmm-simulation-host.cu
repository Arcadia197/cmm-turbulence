#include "hip/hip_runtime.h"
/******************************************************************************************************************************
*
*	This script is part of the code for the characteristic mapping method in 2D with particle flow
*	written in C++ (C) using Nvidia CUDA on Linux.
*
*   The code is managed under GNU General Public License v3.0. Everyone is permitted to copy
*   and distribute verbatim copies of this license document, but changing it is not allowed.
*
*   Documentation and further information can be taken from the GitHub page, located at:
*   https://github.com/CharacteristicMappingMethod/cmm-turbulence
*
******************************************************************************************************************************/

#include "cmm-simulation-host.h"
#include "cmm-simulation-kernel.h"

#include "../numerical/cmm-hermite.h"
#include "../numerical/cmm-timestep.h"

#include "../numerical/cmm-fft.h"

#include "../ui/globals.h"

// debugging, using printf
#include "stdio.h"
#include <math.h>

// parallel reduce
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include "../numerical/cmm-mesure.h"


// function to get difference to 1 for thrust parallel reduction
struct absto1
{
    __host__ __device__
        double operator()(const double &x) const {
            return fabs(1-x);
        }
};
double incompressibility_check(TCudaGrid2D Grid_check, CmmVar2D ChiX, CmmVar2D ChiY, double *grad_Chi) {
	// compute determinant of gradient and save in gradchi
	k_incompressibility_check<<<Grid_check.blocksPerGrid, Grid_check.threadsPerBlock>>>(Grid_check, *ChiX.Grid, ChiX.Dev_var, ChiY.Dev_var, grad_Chi);
	// compute maximum using thrust parallel reduction
	thrust::device_ptr<double> grad_Chi_ptr = thrust::device_pointer_cast(grad_Chi);
	return thrust::transform_reduce(grad_Chi_ptr, grad_Chi_ptr + Grid_check.N, absto1(), 0.0, thrust::maximum<double>());
}

double invertibility_check(TCudaGrid2D Grid_check, CmmVar2D ChiX_b, CmmVar2D ChiY_b, CmmVar2D ChiX_f, CmmVar2D ChiY_f, double *abs_invert) {
	// compute determinant of gradient and save in gradchi
	k_invertibility_check<<<Grid_check.blocksPerGrid, Grid_check.threadsPerBlock>>>(Grid_check, *ChiX_b.Grid, *ChiX_f.Grid,
			ChiX_b.Dev_var, ChiY_b.Dev_var, ChiX_f.Dev_var, ChiY_f.Dev_var, abs_invert);

	// compute maximum using thrust parallel reduction
	thrust::device_ptr<double> abs_invert_ptr = thrust::device_pointer_cast(abs_invert);
	return thrust::reduce(abs_invert_ptr, abs_invert_ptr + Grid_check.N, 0.0, thrust::maximum<double>());
}


// wrapper function for map advection
void advect_using_stream_hermite(SettingsCMM SettingsMain, CmmVar2D ChiX, CmmVar2D ChiY, CmmVar2D Psi,
		double *Chi_new_X, double *Chi_new_Y, double *t, double *dt, int loop_ctr, int direction) {
	
	// compute lagrange coefficients from dt vector for timesteps n+dt and n+dt/2, this makes them dynamic
	double h_L1[4], h_L12[4];  // constant memory for lagrange coefficient to be computed only once
	int loop_ctr_l = loop_ctr + SettingsMain.getLagrangeOrder()-1;  // dt and t are shifted because of initial previous steps
	
	for (int i_p = 0; i_p < SettingsMain.getLagrangeOrder(); ++i_p) {
		h_L1[i_p] = get_L_coefficient(t, t[loop_ctr_l+1], loop_ctr_l, i_p, SettingsMain.getLagrangeOrder());
		h_L12[i_p] = get_L_coefficient(t, t[loop_ctr_l] + dt[loop_ctr_l+1]/2.0, loop_ctr_l, i_p, SettingsMain.getLagrangeOrder());
	}

	double h_c[3];  																		// constant memory for map update coefficient to be computed only once
	switch (SettingsMain.getMapUpdateOrderNum()) {
		case 2: { h_c[0] = +3.0/8.0; h_c[1] = -3.0/20.0; h_c[2] = +1.0/40.0; break; }  		// 6th order interpolation
		case 1: { h_c[0] = +1.0/3.0; h_c[1] = -1.0/12.0; break; }  							// 4th order interpolation
		case 0: { h_c[0] = +1.0/4.0; break; }  												// 2th order interpolation
	}

	double h_c1[12], h_cx[12], h_cy[12], h_cxy[12];  										// compute coefficients for each direction only once
																							// already compute final coefficients with appropriate sign
	for (int i_foot = 0; i_foot < 4+4*SettingsMain.getMapUpdateOrderNum(); ++i_foot) {
		h_c1 [i_foot] = h_c[i_foot/4];
		h_cx [i_foot] = h_c[i_foot/4] * (1 - 2*((i_foot/2)%2))     / SettingsMain.getMapEpsilon() / double(i_foot/4 + 1);
		h_cy [i_foot] = h_c[i_foot/4] * (1 - 2*(((i_foot+1)/2)%2)) / SettingsMain.getMapEpsilon() / double(i_foot/4 + 1);
		h_cxy[i_foot] = h_c[i_foot/4] * (1 - 2*(i_foot%2)) / SettingsMain.getMapEpsilon() / SettingsMain.getMapEpsilon() / double(i_foot/4 + 1) / double(i_foot/4 + 1);
	}

	// copy to constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(d_L1), h_L1, sizeof(double)*4);  hipMemcpyToSymbol(HIP_SYMBOL(d_L12), h_L12, sizeof(double)*4);
	hipMemcpyToSymbol(HIP_SYMBOL(d_c1), h_c1, sizeof(double)*12); hipMemcpyToSymbol(HIP_SYMBOL(d_cx), h_cx, sizeof(double)*12);
	hipMemcpyToSymbol(HIP_SYMBOL(d_cy), h_cy, sizeof(double)*12); hipMemcpyToSymbol(HIP_SYMBOL(d_cxy), h_cxy, sizeof(double)*12);

//	printf("Time - %f \t dt - %f \t TimeInt - %d \t Lagrange - %d \n", t[loop_ctr_l+1], dt[loop_ctr_l+1], SettingsMain.getTimeIntegrationNum(), SettingsMain.getLagrangeOrder());

	// now launch the kernel
	k_advect_using_stream_hermite<<<ChiX.Grid->blocksPerGrid, ChiX.Grid->threadsPerBlock>>>(ChiX.Dev_var, ChiY.Dev_var, Chi_new_X, Chi_new_Y,
			Psi.Dev_var, *ChiX.Grid, *Psi.Grid, t[loop_ctr_l+1], dt[loop_ctr_l+1],
			SettingsMain.getMapEpsilon(), SettingsMain.getTimeIntegrationNum(),
			SettingsMain.getMapUpdateOrderNum(), SettingsMain.getLagrangeOrder(), direction);
}



/*******************************************************************
*		 Apply mapstacks to get full map to initial condition	   *
*******************************************************************/
//void apply_map_stack(TCudaGrid2D Grid, MapStack Map_Stack, double *ChiX, double *ChiY, double *Dev_Temp, int direction)
void apply_map_stack(TCudaGrid2D Grid, MapStack Map_Stack, double *ChiX, double *ChiY, double *Dev_Temp, int direction)
{
	// copy bounds to constant device memory
	hipMemcpyToSymbol(HIP_SYMBOL(d_bounds), Grid.bounds, sizeof(double)*4);

	// backwards map from last to first
	if (direction == -1) {
		// first application: current map
		k_h_sample_map_compact<<<Grid.blocksPerGrid, Grid.threadsPerBlock>>>(ChiX, ChiY, Dev_Temp, *Map_Stack.Grid, Grid);

		// afterwards: trace back all other maps
		for (int i_map = Map_Stack.map_stack_ctr-1; i_map >= 0; i_map--) {
			Map_Stack.copy_map_to_device(i_map);
			k_apply_map_compact<<<Grid.blocksPerGrid, Grid.threadsPerBlock>>>(Map_Stack.Dev_ChiX_stack, Map_Stack.Dev_ChiY_stack,
					Dev_Temp, *Map_Stack.Grid, Grid);
		}
	}
	// forward map from first to last
	else {
		// include remappings
		if (Map_Stack.map_stack_ctr > 0) {
			// first map to get map onto grid
			Map_Stack.copy_map_to_device(0);
			k_h_sample_map_compact<<<Grid.blocksPerGrid, Grid.threadsPerBlock>>>(Map_Stack.Dev_ChiX_stack, Map_Stack.Dev_ChiY_stack, Dev_Temp, *Map_Stack.Grid, Grid);

			// loop over all other maps
			for (int i_map = 1; i_map < Map_Stack.map_stack_ctr; i_map++) {
				Map_Stack.copy_map_to_device(i_map);
				k_apply_map_compact<<<Grid.blocksPerGrid, Grid.threadsPerBlock>>>(Map_Stack.Dev_ChiX_stack, Map_Stack.Dev_ChiY_stack,
						Dev_Temp, *Map_Stack.Grid, Grid);
			}

			// last map: current map
			k_apply_map_compact<<<Grid.blocksPerGrid, Grid.threadsPerBlock>>>(ChiX, ChiY, Dev_Temp, *Map_Stack.Grid, Grid);
		}
		// no remapping has occured yet
		else {
			k_h_sample_map_compact<<<Grid.blocksPerGrid, Grid.threadsPerBlock>>>(ChiX, ChiY, Dev_Temp, *Map_Stack.Grid, Grid);
		}
	}
}


/*******************************************************************
*		 Apply mapstacks to get full map to initial condition	   *
*		    and to map specific points / particles
*******************************************************************/
void apply_map_stack_points(TCudaGrid2D Grid, MapStack Map_Stack, double *ChiX, double *ChiY, double *Dev_Temp, int direction,
		double **fluid_particles_pos_in, double *fluid_particles_pos_out,
		SettingsCMM SettingsMain, int* fluid_particles_blocks, int fluid_particles_threads)
{
	// copy bounds to constant device memory
	hipMemcpyToSymbol(HIP_SYMBOL(d_bounds), Grid.bounds, sizeof(double)*4);

	// backwards map from last to first
	if (direction == -1) {
		// first application: current map
		// sample map
		k_h_sample_map_compact<<<Grid.blocksPerGrid, Grid.threadsPerBlock>>>(ChiX, ChiY, Dev_Temp, *Map_Stack.Grid, Grid);

		// sample particles / specific points, do this for all particles and append them to output
		ParticlesForwarded *particles_forwarded = SettingsMain.getParticlesForwarded();
		double *pos_out_counter = fluid_particles_pos_out;
		for (int i_p = 0; i_p < SettingsMain.getParticlesForwardedNum(); ++i_p) {
			if ((particles_forwarded[i_p].init_map != 0 && particles_forwarded[i_p].init_time != 0) || particles_forwarded[i_p].init_time == 0) {
				k_h_sample_points_map<<<fluid_particles_blocks[i_p], fluid_particles_threads>>>(*Map_Stack.Grid, Grid, ChiX, ChiY,
						fluid_particles_pos_in[i_p], pos_out_counter, particles_forwarded[i_p].num);
			}
			// particles cannot be traced back, just give back the particle positions
			else {
				hipMemcpy(pos_out_counter, fluid_particles_pos_in[i_p], 2*particles_forwarded[i_p].num*sizeof(double), hipMemcpyDeviceToDevice);
			}
			// shift pointer
			pos_out_counter += 2*particles_forwarded[i_p].num;
		}

		// afterwards: trace back all other maps
		for (int i_map = Map_Stack.map_stack_ctr-1; i_map >= 0; i_map--) {
			Map_Stack.copy_map_to_device(i_map);
			// sample map
			k_apply_map_compact<<<Grid.blocksPerGrid, Grid.threadsPerBlock>>>(Map_Stack.Dev_ChiX_stack, Map_Stack.Dev_ChiY_stack,
					Dev_Temp, *Map_Stack.Grid, Grid);

			// sample particles / specific points
			double *pos_out_counter = fluid_particles_pos_out;
			pos_out_counter = fluid_particles_pos_out;
			for (int i_p = 0; i_p < SettingsMain.getParticlesForwardedNum(); ++i_p) {
				// check if this map is applied too
				if ((i_map >= particles_forwarded[i_p].init_map && particles_forwarded[i_p].init_time != 0 && particles_forwarded[i_p].init_map != 0) || particles_forwarded[i_p].init_time == 0) {
					k_h_sample_points_map<<<fluid_particles_blocks[i_p], fluid_particles_threads>>>(*Map_Stack.Grid, Grid, ChiX, ChiY,
							pos_out_counter, pos_out_counter, particles_forwarded[i_p].num);
				}
				// shift pointer
				pos_out_counter += 2*particles_forwarded[i_p].num;
			}
		}
	}
	// forward map from first to last
	else {
		// include remappings
		if (Map_Stack.map_stack_ctr > 0) {
			// first map to get map onto grid
			Map_Stack.copy_map_to_device(0);
			// sample map
			k_h_sample_map_compact<<<Grid.blocksPerGrid, Grid.threadsPerBlock>>>(Map_Stack.Dev_ChiX_stack, Map_Stack.Dev_ChiY_stack, Dev_Temp, *Map_Stack.Grid, Grid);
			// sample particles / specific points, do this for all particles and append them to output
			ParticlesForwarded *particles_forwarded = SettingsMain.getParticlesForwarded();
			double *pos_out_counter = fluid_particles_pos_out;
			for (int i_p = 0; i_p < SettingsMain.getParticlesForwardedNum(); ++i_p) {
				// apply first map
				if (particles_forwarded[i_p].init_time == 0) {
					k_h_sample_points_map<<<fluid_particles_blocks[i_p], fluid_particles_threads>>>(*Map_Stack.Grid, Grid, Map_Stack.Dev_ChiX_stack, Map_Stack.Dev_ChiY_stack,
							fluid_particles_pos_in[i_p], pos_out_counter, particles_forwarded[i_p].num);
				}
				// copy initially as we do not start at t=0
				else {
					hipMemcpy(pos_out_counter, fluid_particles_pos_in[i_p], 2*particles_forwarded[i_p].num*sizeof(double), hipMemcpyDeviceToDevice);
				}
				// shift pointer
				pos_out_counter += 2*particles_forwarded[i_p].num;
			}

			// loop over all other maps
			for (int i_map = 1; i_map < Map_Stack.map_stack_ctr; i_map++) {
				Map_Stack.copy_map_to_device(i_map);
				k_apply_map_compact<<<Grid.blocksPerGrid, Grid.threadsPerBlock>>>(Map_Stack.Dev_ChiX_stack, Map_Stack.Dev_ChiY_stack,
						Dev_Temp, *Map_Stack.Grid, Grid);

				pos_out_counter = fluid_particles_pos_out;
				for (int i_p = 0; i_p < SettingsMain.getParticlesForwardedNum(); ++i_p) {
					// apply first map
					if ((i_map >= particles_forwarded[i_p].init_map && particles_forwarded[i_p].init_time != 0) || particles_forwarded[i_p].init_time == 0) {
						k_h_sample_points_map<<<fluid_particles_blocks[i_p], fluid_particles_threads>>>(*Map_Stack.Grid, Grid, Map_Stack.Dev_ChiX_stack, Map_Stack.Dev_ChiY_stack,
								pos_out_counter, pos_out_counter, particles_forwarded[i_p].num);
					}
					// shift pointer
					pos_out_counter += 2*particles_forwarded[i_p].num;
				}
			}

			// last map: current map
			k_apply_map_compact<<<Grid.blocksPerGrid, Grid.threadsPerBlock>>>(ChiX, ChiY, Dev_Temp, *Map_Stack.Grid, Grid);
			pos_out_counter = fluid_particles_pos_out;
			for (int i_p = 0; i_p < SettingsMain.getParticlesForwardedNum(); ++i_p) {
				// apply first map
				if ((particles_forwarded[i_p].init_map != 0 && particles_forwarded[i_p].init_time != 0) || particles_forwarded[i_p].init_time == 0) {
					k_h_sample_points_map<<<fluid_particles_blocks[i_p], fluid_particles_threads>>>(*Map_Stack.Grid, Grid, ChiX, ChiY,
							pos_out_counter, pos_out_counter, particles_forwarded[i_p].num);
				}
				// shift pointer
				pos_out_counter += 2*particles_forwarded[i_p].num;
			}
		}
		// no remapping has occured yet
		else {
			// sample map
			k_h_sample_map_compact<<<Grid.blocksPerGrid, Grid.threadsPerBlock>>>(ChiX, ChiY, Dev_Temp, *Map_Stack.Grid, Grid);
			// sample particles / specific points, do this for all particles and append them to output
			ParticlesForwarded *particles_forwarded = SettingsMain.getParticlesForwarded();
			double *pos_out_counter = fluid_particles_pos_out;
			for (int i_p = 0; i_p < SettingsMain.getParticlesForwardedNum(); ++i_p) {
				k_h_sample_points_map<<<fluid_particles_blocks[i_p], fluid_particles_threads>>>(*Map_Stack.Grid, Grid, ChiX, ChiY,
						fluid_particles_pos_in[i_p], pos_out_counter, particles_forwarded[i_p].num);

				pos_out_counter += 2*particles_forwarded[i_p].num;
			}
		}
	}
}


/*******************************************************************
*				Compute fine vorticity hermite					   *
*******************************************************************/

void translate_initial_condition_through_map_stack(MapStack Map_Stack, CmmVar2D ChiX, CmmVar2D ChiY, CmmVar2D Var,
		CmmVar2D Var_discrete_init, hipfftDoubleComplex *Dev_Temp_C1, int simulation_num_c, bool initial_discrete, int var_num /*=0*/)
{
	/*
	@param Var: output variable to be translated through all maps
	@param Dev_Temp_C1: complex space temporary variable
	@param Var_discrete_init: initial condition as discrete array
	@param initial_discrete: if true, initial condition is used from discrete array
	@param simulation_num_c: number of simulation (inicond), used for continuous initial condition
	@param var_num: 0 vorticity (default), 1 passive scalar, 2 distribution function (optional), for continuous IC
	*/

	// Sample vorticity on fine grid
	// Var is used as temporary variable and output
	apply_map_stack(*Var.Grid, Map_Stack, ChiX.Dev_var, ChiY.Dev_var, Var.Dev_var+Var.Grid->N, -1);
	// initial condition from either discrete array or condition
	k_h_sample_from_init<<<Var.Grid->blocksPerGrid, Var.Grid->threadsPerBlock>>>(Var.Dev_var, Var.Dev_var+Var.Grid->N,
			*Var.Grid, *Var_discrete_init.Grid, var_num, simulation_num_c, Var_discrete_init.Dev_var, initial_discrete);

	// go to comlex space
	hipfftExecD2Z(Var.plan_D2Z, Var.Dev_var, Dev_Temp_C1);
	k_normalize_h<<<Var.Grid->fft_blocks, Var.Grid->threadsPerBlock>>>(Dev_Temp_C1, *Var.Grid);

	// cut_off frequencies at N_fine/3 for turbulence (effectively 2/3)
//	k_fft_cut_off_scale<<<Grid_fineblocksPerGrid, Grid_finethreadsPerBlock>>>(Dev_Temp_C1, Grid_fineNX, (double)(Grid_fineNX)/3.0);

	// form hermite formulation
	fourier_hermite(*Var.Grid, Dev_Temp_C1, Var.Dev_var, Var.plan_Z2D);
}


/*******************************************************************
*						 Computation of Psi						   *
*******************************************************************/
void evaluate_stream_hermite(CmmVar2D ChiX, CmmVar2D ChiY, CmmVar2D Vort_fine_init, CmmVar2D Psi, CmmVar2D empty_vort,
		hipfftDoubleComplex *Dev_Temp_C1, int molly_stencil, double freq_cut_psi)
{
	/*
	This function computes the solution to $L \psi = w $ where $w$ is the vorticity and $\psi$ is the stream function
	@param Vort_fine_init: real space vorticity on fine grid as initial condition of sub-map
	@param Psi: real space stream function in Hermite form as output
	@param Dev_Temp_C1: complex space temporary variable
	@param empty_vort: this contains the grid and cufft_plans for the sampling size in real space, we then shift the grid in complex space
	*/


	// apply map to w and sample using mollifier, do it on a special grid for vorticity and apply mollification if wanted
	k_apply_map_and_sample_from_hermite<<<empty_vort.Grid->blocksPerGrid, empty_vort.Grid->threadsPerBlock>>>(ChiX.Dev_var, ChiY.Dev_var,
			(hipfftDoubleReal*)Dev_Temp_C1, Vort_fine_init.Dev_var, *ChiX.Grid, *empty_vort.Grid, *Vort_fine_init.Grid, molly_stencil, true);

	// forward fft, inline which is possible for forward fft
	hipfftExecD2Z(empty_vort.plan_D2Z, (hipfftDoubleReal*)Dev_Temp_C1, Dev_Temp_C1);
	k_normalize_h<<<empty_vort.Grid->fft_blocks, empty_vort.Grid->threadsPerBlock>>>(Dev_Temp_C1, *empty_vort.Grid);  // this is a normalization factor of FFT? if yes we dont need to do it everytime!!!

	// cut_off frequencies at N_psi/3 for turbulence (effectively 2/3) and compute smooth W
	// use Psi grid here for intermediate storage
	//	k_fft_cut_off_scale<<<Grid_coarse.blocksPerGrid, Grid_coarse.threadsPerBlock>>>(Dev_Temp_C1, Grid_coarse.NX, (double)(Grid_psi.NX)/3.0);

	// transition to stream function grid with three cases : grid_vort < grid_psi, grid_vort > grid_psi (a bit dumb) and grid_vort == grid_psi
	// grid change because inline data movement is nasty, we can use psi_real as buffer anyways
	if (empty_vort.Grid->NX != Psi.Grid->NX || empty_vort.Grid->NY != Psi.Grid->NY) {
		k_fft_grid_move<<<Psi.Grid->fft_blocks, Psi.Grid->threadsPerBlock>>>(Dev_Temp_C1, (hipfftDoubleComplex*) Psi.Dev_var, *Psi.Grid, *empty_vort.Grid);
	}
	// no movement needed, just copy data over
	else {
		hipMemcpy(Psi.Dev_var, Dev_Temp_C1, empty_vort.Grid->sizeNfft, hipMemcpyDeviceToDevice);
	}

	// cut high frequencies in fourier space, however not that much happens after zero move add from coarse grid
	k_fft_cut_off_scale_h<<<Psi.Grid->fft_blocks, Psi.Grid->threadsPerBlock>>>((hipfftDoubleComplex*) Psi.Dev_var, *Psi.Grid, freq_cut_psi);

	// Forming Psi hermite now on psi grid
	k_fft_iLap_h<<<Psi.Grid->fft_blocks, Psi.Grid->threadsPerBlock>>>((hipfftDoubleComplex*) Psi.Dev_var, Dev_Temp_C1, *Psi.Grid);

	// Inverse laplacian in Fourier space
	fourier_hermite(*Psi.Grid, Dev_Temp_C1, Psi.Dev_var, Psi.plan_Z2D);
}
// debugging lines, could be needed here to check psi
//	hipMemcpy(Host_Debug, Psi_real, 4*Grid_psi.sizeNReal, hipMemcpyDeviceToHost);
//	writeAllRealToBinaryFile(4*Grid_psi.N, Host_Debug, "psi_debug_4_nodes_C512_F2048_t64_T1", "Debug_2");


/*******************************************************************
*						 Computation of Psi						   *
*******************************************************************/

void evaluate_potential_from_density_hermite(SettingsCMM SettingsMain, TCudaGrid2D Grid_coarse, TCudaGrid2D Grid_fine, TCudaGrid2D Grid_Psi, TCudaGrid2D Grid_vort,
		double *Dev_ChiX, double *Dev_ChiY, double *Dev_W_H_fine_real, double *Psi_real,
		hipfftHandle cufft_plan_psi_D2Z, hipfftHandle cufft_plan_psi_Z2D, hipfftHandle cufft_plan_phi_1D, hipfftHandle cufft_plan_phi_1D_inverse,
		hipfftDoubleComplex *Dev_Temp_C1, int molly_stencil, double freq_cut_psi)	
{	/*
	This function computes the solution to $L_xx \phi = (1-l \int f dv) $ where $w$ is the vorticity and $\phi$ is the stream function
	*/	
	// apply map to w and sample using mollifier, do it on a special grid for vorticity and apply mollification if wanted
	k_apply_map_and_sample_from_hermite<<<Grid_vort.blocksPerGrid, Grid_vort.threadsPerBlock>>>(Dev_ChiX, Dev_ChiY,
			(hipfftDoubleReal*)Dev_Temp_C1, Dev_W_H_fine_real, Grid_coarse, Grid_vort, Grid_fine, molly_stencil, false);
	// this function solves the 1D laplace equation on the Grid_vort (coarse) and upsamples to Grid_Psi (fine)
	get_psi_hermite_from_distribution_function(Psi_real, (hipfftDoubleReal*)Dev_Temp_C1, Dev_Temp_C1, cufft_plan_phi_1D, cufft_plan_phi_1D_inverse, 
	cufft_plan_psi_D2Z, cufft_plan_psi_Z2D  ,Grid_vort, Grid_Psi);
	//  writeTranferToBinaryFile(Grid_Psi.N, (hipfftDoubleReal*)(Psi_real), SettingsMain, "/Stream_function_mollys", false);
	//  error("evaluate_potential_from_density_hermite: not nted yet",134);
// 
}



//// sample psi on a fixed grid with vorticity known - assumes periodicity is preserved (no zoom!)
//void psi_upsampling(TCudaGrid2D Grid, double *Dev_W, hipfftDoubleComplex *Dev_Temp_C1, double *Dev_Psi,
//		hipfftHandle cufft_plan_D2Z, hipfftHandle cufft_plan_Z2D){
//	hipfftExecD2Z(cufft_plan_D2Z, Dev_W, Dev_Temp_C1);
//	k_normalize_h<<<Grid.fft_blocks, Grid.threadsPerBlock>>>(Dev_Temp_C1, Grid);
//
//	// Forming Psi hermite
//	k_fft_iLap_h<<<Grid.fft_blocks, Grid.threadsPerBlock>>>(Dev_Temp_C1, Dev_Temp_C1, Grid);
//	fourier_hermite(Grid, Dev_Temp_C1, Dev_Psi, cufft_plan_Z2D);
//}



void get_psi_hermite_from_distribution_function(double *Psi_real_out, double *Dev_f_in, hipfftDoubleComplex *Dev_Temp_C1,
		hipfftHandle cufft_plan_phi_1D, hipfftHandle cufft_plan_phi_1D_inverse, hipfftHandle cufft_plan_psi_D2Z, hipfftHandle cufft_plan_psi_Z2D,
		TCudaGrid2D Grid, TCudaGrid2D Grid_psi){
	// #################################################################################################################			
	// this function solves the 1D laplace equation on the Grid_vort (coarse) and upsamples to Grid_Psi (fine)
	// #################################################################################################################
	integral_v<<<Grid.blocksPerGrid.x, Grid.threadsPerBlock.x >>>((hipfftDoubleReal*)Dev_f_in, (hipfftDoubleReal*)Dev_Temp_C1, Grid.NX, Grid.NY, Grid.hy);
	// forward fft
	hipfftExecD2Z (cufft_plan_phi_1D, (hipfftDoubleReal*)Dev_Temp_C1, Dev_Temp_C1+Grid.Nfft);
	// devide by NX to normalize FFT
	k_normalize_1D_h<<<Grid.fft_blocks.x, Grid.threadsPerBlock.x>>>(Dev_Temp_C1+Grid.Nfft, Grid);  // this is a normalization factor of FFT? if yes we dont need to do it everytime!!! 
	// inverse laplacian in fourier space - division by kx**2 and ky**2
	k_fft_iLap_h_1D<<<Grid.fft_blocks.x, Grid.threadsPerBlock.x>>>((hipfftDoubleComplex*) Dev_Temp_C1+Grid.Nfft, (hipfftDoubleComplex*) Dev_Temp_C1, Grid);
	// do zero padding if needed
	if (Grid.NX != Grid_psi.NX || Grid.NY != Grid_psi.NY) {
		zero_padding_1D<<<Grid_psi.fft_blocks.x, Grid_psi.threadsPerBlock.x>>>(Dev_Temp_C1, (hipfftDoubleComplex*) Psi_real_out, Grid_psi, Grid);
	}
	else { // no movement needed, just copy data over
		hipMemcpy(Psi_real_out, Dev_Temp_C1, Grid.NX, hipMemcpyDeviceToDevice);
	}

	// inverse fft (1D)
	hipfftExecZ2D (cufft_plan_phi_1D_inverse, (hipfftDoubleComplex*) Psi_real_out, (hipfftDoubleReal*)(Dev_Temp_C1));

	// assemble psi= phi  - v^2/2
	k_assemble_psi<<<Grid_psi.blocksPerGrid, Grid_psi.threadsPerBlock>>>((hipfftDoubleReal*)(Dev_Temp_C1), Psi_real_out, Grid_psi);
	// convert 2d psi to fourier space
	hipfftExecD2Z (cufft_plan_psi_D2Z, Psi_real_out, Dev_Temp_C1);
	k_normalize_h<<<Grid_psi.fft_blocks, Grid_psi.threadsPerBlock>>>(Dev_Temp_C1, Grid_psi);
	// cut high frequencies in fourier space, however not that much happens after zero move add from coarse grid
	// k_fft_cut_off_scale_h<<<Grid_Psi.fft_blocks, Grid_Psi.threadsPerBlock>>>((hipfftDoubleComplex*) Dev_Temp_C1+Grid_Psi.Nfft, Grid_Psi, freq_cut_psi);
	// Inverse laplacian in Fourier space
	fourier_hermite(Grid_psi, Dev_Temp_C1, Psi_real_out, cufft_plan_psi_Z2D);
}



//// compute laplacian on variable grid, needs Grid.sizeNfft + Grid.sizeN memory
//void laplacian(CmmVar2D Var_in, CmmVar2D Var_out, hipfftDoubleComplex *Dev_Temp_C1){
//    hipfftExecD2Z(Var_in.plan_D2Z, Var_in.Dev_var, Dev_Temp_C1);
//    k_normalize_h<<<Var_in.Grid->fft_blocks, Var_in.Grid->threadsPerBlock>>>(Dev_Temp_C1, *Var_in.Grid);
//
//    k_fft_lap_h<<<Var_in.Grid->fft_blocks, Var_in.Grid->threadsPerBlock>>>(Dev_Temp_C1, Dev_Temp_C1, *Var_in.Grid);
//    hipfftExecZ2D(Var_in.plan_Z2D, Dev_Temp_C1, Var_out.Dev_var);
//}
//
//// compute x-gradient on variable grid, needs Grid.sizeNfft + Grid.sizeN memory
//void grad_x(CmmVar2D Var_in, CmmVar2D Var_out, hipfftDoubleComplex *Dev_Temp_C1){
//    hipfftExecD2Z(Var_in.plan_D2Z, Var_in.Dev_var, Dev_Temp_C1);
//    k_normalize_h<<<Var_in.Grid->fft_blocks, Var_in.Grid->threadsPerBlock>>>(Dev_Temp_C1, *Var_in.Grid);
//
//    k_fft_dx_h<<<Var_in.Grid->fft_blocks, Var_in.Grid->threadsPerBlock>>>(Dev_Temp_C1, Dev_Temp_C1, *Var_in.Grid);
//    hipfftExecZ2D(Var_in.plan_Z2D, Dev_Temp_C1, Var_out.Dev_var);
//}
//
//// compute x-gradient on variable grid, needs Grid.sizeNfft + Grid.sizeN memory
//void grad_y(CmmVar2D Var_in, CmmVar2D Var_out, hipfftDoubleComplex *Dev_Temp_C1){
//    hipfftExecD2Z(Var_in.plan_D2Z, Var_in.Dev_var, Dev_Temp_C1);
//    k_normalize_h<<<Var_in.Grid->fft_blocks, Var_in.Grid->threadsPerBlock>>>(Dev_Temp_C1, *Var_in.Grid);
//
//    k_fft_dy_h<<<Var_in.Grid->fft_blocks, Var_in.Grid->threadsPerBlock>>>(Dev_Temp_C1, Dev_Temp_C1, *Var_in.Grid);
//    hipfftExecZ2D(Var_in.plan_Z2D, Dev_Temp_C1, Var_out.Dev_var);
//}




//// compute hermite with derivatives in fourier space, uniform helper function fitted for all grids to utilize only input temporary variable
//// input has size (NX+1)/2*NY and output 4*NX*NY, output is therefore used as temporary variable
//void fourier_hermite(TCudaGrid2D Grid, hipfftDoubleComplex *Dev_In, double *Dev_Out, hipfftHandle cufft_plan) {
//
//	// reshift for transforming so that we have enough space for everything
//	Dev_Out += Grid.N - 2*Grid.Nfft;
//
//	// dy and dxdy derivates are stored in later parts of output array, we can therefore use the first half as a temporary variable
//	// start with computing dy derivative
//	k_fft_dy_h<<<Grid.fft_blocks, Grid.threadsPerBlock>>>(Dev_In, (hipfftDoubleComplex*)Dev_Out, Grid);
//
//	// compute dxdy afterwards, to combine backwards transformations
//	k_fft_dx_h<<<Grid.fft_blocks, Grid.threadsPerBlock>>>((hipfftDoubleComplex*)Dev_Out, (hipfftDoubleComplex*)(Dev_Out) + Grid.Nfft, Grid);
//
//	// backwards transformation, store dx in position 3/4 and dy in position 4/4
//	hipfftExecZ2D(cufft_plan, (hipfftDoubleComplex*)(Dev_Out) + Grid.Nfft, Dev_Out + 2*Grid.N + 2*Grid.Nfft);
//	hipfftExecZ2D(cufft_plan, (hipfftDoubleComplex*)Dev_Out, Dev_Out + Grid.N + 2*Grid.Nfft);
//
//	// now compute dx derivative on itself and store it in the right place
//	k_fft_dx_h<<<Grid.fft_blocks, Grid.threadsPerBlock>>>(Dev_In, (hipfftDoubleComplex*)Dev_Out, Grid);
//	hipfftExecZ2D(cufft_plan, (hipfftDoubleComplex*)Dev_Out, Dev_Out + 2*Grid.Nfft);// x-derivative of the vorticity in Fourier space
//
//	/* Memory layout before shift of Dev_Out:
//	 f	 	... Dev_Out[2*Grid.Nfft-Grid.N	  , ..., 2*Grid.Nfft            - 1]
//	 d/dx 	... Dev_Out[2*Grid.Nfft			  , ..., 2*Grid.Nfft +   Grid.N - 1]
//	 d/dy 	... Dev_Out[2*Grid.Nfft +   Grid.N, ..., 2*Grid.Nfft + 2*Grid.N - 1]
//	 d/dxdy ... Dev_Out[2*Grid.Nfft + 2*Grid.N, ..., 2*Grid.Nfft + 3*Grid.N - 1]
//	*/
//	// shift again just before final store
//	Dev_Out += 2*Grid.Nfft - Grid.N;
//
//	// at last, store normal values
//	hipfftExecZ2D(cufft_plan, Dev_In, Dev_Out);
//
//}


/*******************************************************************
*		 Computation of Global conservation values				   *
*******************************************************************/
std::string compute_conservation_targets(SettingsCMM SettingsMain, double t_now, double dt_now, double dt,
		std::map<std::string, CmmVar2D*> cmmVarMap, hipfftDoubleComplex *Dev_Temp_C1)
{
	// check if we want to save at this time, combine all variables if so
	bool save_now = false;
	SaveComputational* save_comp = SettingsMain.getSaveComputational();
	for (int i_save = 0; i_save < SettingsMain.getSaveComputationalNum(); ++i_save) {
		// instants - distance to target is smaller than threshhold
		if (save_comp[i_save].is_instant && t_now - save_comp[i_save].time_start + dt*1e-5 < dt_now && t_now - save_comp[i_save].time_start + dt*1e-5 >= 0 && save_comp[i_save].conv) {
			save_now = true;
		}
		// intervals - modulo to steps with safety-increased targets is smaller than step
		if (!save_comp[i_save].is_instant
			&& ((fmod(t_now - save_comp[i_save].time_start + dt*1e-5, save_comp[i_save].time_step) < dt_now
			&& t_now + dt*1e-5 >= save_comp[i_save].time_start
			&& t_now - dt*1e-5 <= save_comp[i_save].time_end)
			|| t_now == save_comp[i_save].time_end)) {
			save_now = true;
		}
	}

	std::string message = "";
	if (save_now) {
		// compute mesure values
		double mesure[4];

		// compute quantities
		if (SettingsMain.getSimulationType() == "cmm_vlasov_poisson_1d"){
			Compute_Total_Energy(mesure[0], mesure[1], mesure[2], cmmVarMap["Psi"]->Dev_var, cmmVarMap["Vort"]->Dev_var, (hipfftDoubleReal*) Dev_Temp_C1, *cmmVarMap["Psi"]->Grid);
			Compute_Mass(mesure[3], cmmVarMap["Vort"]->Dev_var, *cmmVarMap["Vort"]->Grid); // is simply the mass in vlasov poisson (times 0.5)

			// save
			writeAppendToBinaryFile(1, &t_now, SettingsMain, "/Monitoring_data/Mesure/Time_s");  // time vector for data
			writeAppendToBinaryFile(1, mesure, SettingsMain, "/Monitoring_data/Mesure/Etot");
			writeAppendToBinaryFile(1, mesure+1, SettingsMain, "/Monitoring_data/Mesure/Ekin");
			writeAppendToBinaryFile(1, mesure+2, SettingsMain, "/Monitoring_data/Mesure/Epot");
			writeAppendToBinaryFile(1, mesure+3, SettingsMain, "/Monitoring_data/Mesure/Mass");

			// construct message
			message = "Computed coarse Cons : Etot = " + to_str(mesure[0], 8)
					+    " \t Ekin = " + to_str(mesure[1], 8)
					+ " \t Epot = " + to_str(mesure[2], 8)
					+ " \t Mass = " + to_str(mesure[3], 8);
		}
		else{
			Compute_Energy_H(mesure[0], *cmmVarMap["Psi"]);
//			Compute_Energy(mesure[0], *cmmVarMap["Psi"], Dev_Temp_C1);
			Compute_Enstrophy(mesure[1], *cmmVarMap["Vort"]);
			Compute_Palinstrophy(mesure[2], *cmmVarMap["Vort"], Dev_Temp_C1);
		
			// wmax
			thrust::device_ptr<double> w_ptr = thrust::device_pointer_cast(cmmVarMap["Vort"]->Dev_var);
			double w_max = thrust::reduce(w_ptr, w_ptr + cmmVarMap["Vort"]->Grid->N, 0.0, thrust::maximum<double>());
			double w_min = thrust::reduce(w_ptr, w_ptr + cmmVarMap["Vort"]->Grid->N, 0.0, thrust::minimum<double>());
			mesure[3] = std::max(w_max, -w_min);

			// hash of vorticity and stream function
			double w_hash[2]; Hash_array((char*)w_hash, cmmVarMap["Vort"]->Dev_var, cmmVarMap["Vort"]->Grid->N);
			double psi_hash[2]; Hash_array((char*)psi_hash, cmmVarMap["Psi"]->Dev_var, cmmVarMap["Psi"]->Grid->N);

			// save
			writeAppendToBinaryFile(1, &t_now, SettingsMain, "/Monitoring_data/Mesure/Time_s");  // time vector for data
			writeAppendToBinaryFile(1, mesure, SettingsMain, "/Monitoring_data/Mesure/Energy");
			writeAppendToBinaryFile(1, mesure+1, SettingsMain, "/Monitoring_data/Mesure/Enstrophy");
			writeAppendToBinaryFile(1, mesure+2, SettingsMain, "/Monitoring_data/Mesure/Palinstrophy");
			writeAppendToBinaryFile(1, mesure+3, SettingsMain, "/Monitoring_data/Mesure/Max_vorticity");
			writeAppendToBinaryFile(2, w_hash, SettingsMain, "/Monitoring_data/Mesure/Hash_vorticity");
			writeAppendToBinaryFile(2, psi_hash, SettingsMain, "/Monitoring_data/Mesure/Hash_stream_function");

			// construct message
			message = "Computed coarse Cons : Energ = " + to_str(mesure[0], 8)
					+    " \t Enstr = " + to_str(mesure[1], 8)
					+ " \t Palinstr = " + to_str(mesure[2], 8)
					+ " \t Wmax = " + to_str(mesure[3], 8);
		}
	}

	return message;
}



/*******************************************************************
*		 Sample on a specific grid and save everything	           *
*	i know this became quite a beast in terms of input parameters
*******************************************************************/
std::string sample_compute_and_write(SettingsCMM SettingsMain, double t_now, double dt_now, double dt,
		MapStack Map_Stack, MapStack Map_Stack_f, std::map<std::string, CmmVar2D*> cmmVarMap, hipfftDoubleComplex *Dev_Temp_C1,
		double **Host_forward_particles_pos, double **Dev_forward_particles_pos, int *forward_particles_block, int forward_particles_thread)
{

	// check if we want to save at this time, combine all variables if so
	std::string message = "";
	SaveSample* save_sample = SettingsMain.getSaveSample();
	double mesure[4];  // it's fine if we only output it last time, thats enough i guess
	for (int i_save = 0; i_save < SettingsMain.getSaveSampleNum(); ++i_save) {
		// check each save and execute it independent
		bool save_now = false;
		// instants - distance to target is smaller than threshhold
		if (save_sample[i_save].is_instant && t_now - save_sample[i_save].time_start + dt*1e-5 < dt_now && t_now - save_sample[i_save].time_start + dt*1e-5 >= 0) {
			save_now = true;
		}
		// intervals - modulo to steps with safety-increased targets is smaller than step
		if (!save_sample[i_save].is_instant
			&& ((fmod(t_now - save_sample[i_save].time_start + dt*1e-5, save_sample[i_save].time_step) < dt_now
			&& t_now + dt*1e-5 >= save_sample[i_save].time_start
			&& t_now - dt*1e-5 <= save_sample[i_save].time_end)
			|| t_now == save_sample[i_save].time_end)) {
			save_now = true;
		}

		if (save_now) {
			CmmVar2D *Sample_var = cmmVarMap["Sample_" + to_str(i_save)];  // extract sample variable
			std::string save_var = save_sample[i_save].var;  // extract save variables
			// forwards map to get it done
			if (SettingsMain.getForwardMap()) {
				// compute only if we actually want to save, elsewhise its a lot of computations for nothing
				if (save_var.find("Map_f") != std::string::npos or save_var.find("Chi_f") != std::string::npos
						or SettingsMain.getParticlesForwardedNum() > 0) {
					// apply mapstack to map or particle positions
					if (SettingsMain.getParticlesForwardedNum() == 0) {
						apply_map_stack(*Sample_var->Grid, Map_Stack_f, cmmVarMap["ChiX_f"]->Dev_var, cmmVarMap["ChiY_f"]->Dev_var, (hipfftDoubleReal*)Dev_Temp_C1, 1);
					}
					// forwarded particles: forward all particles regardless of if they will be saved, needs rework to be more clever
					else {
						apply_map_stack_points(*Sample_var->Grid, Map_Stack_f, cmmVarMap["ChiX_f"]->Dev_var, cmmVarMap["ChiY_f"]->Dev_var, (hipfftDoubleReal*)Dev_Temp_C1, 1,
								Dev_forward_particles_pos, (hipfftDoubleReal*)Dev_Temp_C1+2*Sample_var->Grid->N,
								SettingsMain, forward_particles_block, forward_particles_thread);
					}

					// save map by copying and saving offsetted data
					if (save_var.find("Map_f") != std::string::npos or save_var.find("Chi_f") != std::string::npos) {
						hipMemcpy2D(Sample_var->Dev_var, sizeof(double), (hipfftDoubleReal*)Dev_Temp_C1, sizeof(double)*2,
												 sizeof(double), Sample_var->Grid->N, hipMemcpyDeviceToDevice);
						writeTimeVariable(SettingsMain, "Map_ChiX_f_"+to_str(Sample_var->Grid->NX),
								t_now, Sample_var->Dev_var, Sample_var->Grid->N);
						hipMemcpy2D(Sample_var->Dev_var, sizeof(double), (hipfftDoubleReal*)Dev_Temp_C1+1, sizeof(double)*2,
												 sizeof(double), Sample_var->Grid->N, hipMemcpyDeviceToDevice);
						writeTimeVariable(SettingsMain, "Map_ChiY_f_"+to_str(Sample_var->Grid->NX),
								t_now, Sample_var->Dev_var, Sample_var->Grid->N);
					}

					// save position of forwarded particles, go through all and only safe the needed ones
					ParticlesForwarded* particles_forwarded = SettingsMain.getParticlesForwarded();
					double* particles_out_counter = (hipfftDoubleReal*)Dev_Temp_C1+2*Sample_var->Grid->N;
					for (int i_p = 0; i_p < SettingsMain.getParticlesForwardedNum(); ++i_p) {
						if (save_var.find("PartF_" + to_str_0(i_p+1, 2)) != std::string::npos) {
							// create particles folder if necessary
							std::string t_s_now = to_str(t_now); if (abs(t_now - T_MAX) < 1) t_s_now = "final";
							std::string sub_folder_name = "/Particle_data/Time_" + t_s_now;
							std::string folder_name_now = SettingsMain.getWorkspace() + "data/" + SettingsMain.getFileName() + sub_folder_name;
							struct stat st = {0};
							if (stat(folder_name_now.c_str(), &st) == -1) mkdir(folder_name_now.c_str(), 0777);
							// copy data to host and save
							hipMemcpy(Host_forward_particles_pos[i_p], particles_out_counter, 2*particles_forwarded[i_p].num*sizeof(double), hipMemcpyDeviceToHost);
							writeAllRealToBinaryFile(2*particles_forwarded[i_p].num, Host_forward_particles_pos[i_p], SettingsMain, "/Particle_data/Time_" + t_s_now + "/Particles_forwarded_pos_P" + to_str_0(i_p+1, 2));

						}
						particles_out_counter += 2*particles_forwarded[i_p].num;  // increase counter
					}
				}

			}

			// compute map to initial condition through map stack
			apply_map_stack(*Sample_var->Grid, Map_Stack, cmmVarMap["ChiX"]->Dev_var, cmmVarMap["ChiY"]->Dev_var, (hipfftDoubleReal*)Dev_Temp_C1, -1);

			// save map by copying and saving offsetted data
			if (save_var.find("Map_b") != std::string::npos or save_var.find("Chi_b") != std::string::npos) {
				hipMemcpy2D(Sample_var->Dev_var, sizeof(double), (hipfftDoubleReal*)Dev_Temp_C1, sizeof(double)*2,
										 sizeof(double), Sample_var->Grid->N, hipMemcpyDeviceToDevice);
				writeTimeVariable(SettingsMain, "Map_ChiX_"+to_str(Sample_var->Grid->NX),
						t_now, Sample_var->Dev_var, Sample_var->Grid->N);
				hipMemcpy2D(Sample_var->Dev_var, sizeof(double), (hipfftDoubleReal*)Dev_Temp_C1+1, sizeof(double)*2,
										 sizeof(double), Sample_var->Grid->N, hipMemcpyDeviceToDevice);
				writeTimeVariable(SettingsMain, "Map_ChiY_"+to_str(Sample_var->Grid->NX),
						t_now, Sample_var->Dev_var, Sample_var->Grid->N);
			}

			// passive scalar theta - 1 to switch for passive scalar
			if (save_var.find("Scalar") != std::string::npos or save_var.find("Theta") != std::string::npos) {
				k_h_sample_from_init<<<Sample_var->Grid->blocksPerGrid, Sample_var->Grid->threadsPerBlock>>>(Sample_var->Dev_var, (hipfftDoubleReal*)Dev_Temp_C1,
						*Sample_var->Grid, *cmmVarMap["Vort_discrete_init"]->Grid, 1, SettingsMain.getScalarNum(), cmmVarMap["Vort_discrete_init"]->Dev_var, SettingsMain.getScalarDiscrete());

				writeTimeVariable(SettingsMain, "Scalar_Theta_"+to_str(Sample_var->Grid->NX),
						t_now, Sample_var->Dev_var, Sample_var->Grid->N);

				thrust::device_ptr<double> scal_ptr = thrust::device_pointer_cast(Sample_var->Dev_var);
				double scal_int = Sample_var->Grid->hx * Sample_var->Grid->hy * thrust::reduce(scal_ptr, scal_ptr + Sample_var->Grid->N, 0.0, thrust::plus<double>());
				writeAppendToBinaryFile(1, &scal_int, SettingsMain, "/Monitoring_data/Mesure/Scalar_integral_"+ to_str(Sample_var->Grid->NX));
			}


			int varnum = 0;
			// compute vorticity - 0 to switch for vorticity
			k_h_sample_from_init<<<Sample_var->Grid->blocksPerGrid, Sample_var->Grid->threadsPerBlock>>>(Sample_var->Dev_var, (hipfftDoubleReal*)Dev_Temp_C1,
					*Sample_var->Grid, *cmmVarMap["Vort_discrete_init"]->Grid, varnum, SettingsMain.getInitialConditionNum(), cmmVarMap["Vort_discrete_init"]->Dev_var, SettingsMain.getInitialDiscrete());

			// save vorticity
			if (save_var.find("Vorticity") != std::string::npos or save_var.find("W") != std::string::npos  or save_var.find("F") != std::string::npos) {
				writeTimeVariable(SettingsMain, "Vorticity_W_"+to_str(Sample_var->Grid->NX),
						t_now, Sample_var->Dev_var, Sample_var->Grid->N);
			}

			// compute enstrophy and palinstrophy
			Compute_Enstrophy(mesure[1], *Sample_var);
			Compute_Palinstrophy(mesure[2], *Sample_var, Dev_Temp_C1);

			// compute wmax
			thrust::device_ptr<double> w_ptr = thrust::device_pointer_cast(Sample_var->Dev_var);
			double w_max = thrust::reduce(w_ptr, w_ptr + Sample_var->Grid->N, 0.0, thrust::maximum<double>());
			double w_min = thrust::reduce(w_ptr, w_ptr + Sample_var->Grid->N, 0.0, thrust::minimum<double>());
			mesure[3] = std::max(w_max, -w_min);

			// hash of vorticity
			double w_hash[2]; Hash_array((char*)w_hash, Sample_var->Dev_var, Sample_var->Grid->N);

			// compute laplacian of vorticity
			if (save_var.find("Laplacian_W") != std::string::npos) {
				laplacian(*Sample_var, Sample_var->Dev_var+Sample_var->Grid->N, Dev_Temp_C1);

				writeTimeVariable(SettingsMain, "Laplacian_W_"+to_str(Sample_var->Grid->NX),
						t_now, Sample_var->Dev_var+Sample_var->Grid->N, Sample_var->Grid->N);
			}

			// compute gradient of vorticity
			if (save_var.find("Grad_W") != std::string::npos) {
				grad_x(*Sample_var, Sample_var->Dev_var+Sample_var->Grid->N, Dev_Temp_C1);

				writeTimeVariable(SettingsMain, "GradX_W_"+to_str(Sample_var->Grid->NX),
						t_now, Sample_var->Dev_var+Sample_var->Grid->N, Sample_var->Grid->N);

				grad_y(*Sample_var, Sample_var->Dev_var+Sample_var->Grid->N, Dev_Temp_C1);

				writeTimeVariable(SettingsMain, "GradY_W_"+to_str(Sample_var->Grid->NX),
						t_now, Sample_var->Dev_var+Sample_var->Grid->N, Sample_var->Grid->N);
			}

			// reuse sampled vorticity to compute psi, take fourier_hermite reshifting into account
			i_laplacian(*Sample_var, Sample_var->Dev_var, Dev_Temp_C1);
			Compute_Energy(mesure[0], *Sample_var, Dev_Temp_C1);

//			size_t shift = 2*Sample_var->Grid->Nfft - Sample_var->Grid->N;
//			i_laplacian_h(*Sample_var, Sample_var->Dev_var+shift, Dev_Temp_C1);
//			Sample_var->Dev_var = Sample_var->Dev_var+shift;
//
//			Compute_Energy_H(mesure[0], *Sample_var);

			// hash of stream function
			double psi_hash[2]; Hash_array((char*)psi_hash, Sample_var->Dev_var, Sample_var->Grid->N);

			if (save_var.find("Stream") != std::string::npos or save_var.find("Psi") != std::string::npos) {
				writeTimeVariable(SettingsMain, "Stream_function_Psi_"+to_str(Sample_var->Grid->NX),
						t_now, Sample_var->Dev_var, Sample_var->Grid->N);
			}
			// disable Hermite of stream function, saves space and all parts could be computed individually anyways
//			if (save_var.find("Stream_H") != std::string::npos or save_var.find("Psi_H") != std::string::npos) {
//				writeTimeVariable(SettingsMain, "Stream_function_Psi_"+to_str(Sample_var->Grid->NX),
//						t_now, Sample_var->Dev_var, 4*Sample_var->Grid->N);
//			}
			if (save_var.find("Velocity") != std::string::npos or save_var.find("U") != std::string::npos) {
				grad_x(*Sample_var, Sample_var->Dev_var+Sample_var->Grid->N, Dev_Temp_C1);
				writeTimeVariable(SettingsMain, "Velocity_UX_"+to_str(Sample_var->Grid->NX),
						t_now, Sample_var->Dev_var+Sample_var->Grid->N, Sample_var->Grid->N);

				grad_y(*Sample_var, Sample_var->Dev_var+Sample_var->Grid->N, Dev_Temp_C1);

				writeTimeVariable(SettingsMain, "Velocity_UY_"+to_str(Sample_var->Grid->NX),
						t_now, Sample_var->Dev_var+Sample_var->Grid->N, Sample_var->Grid->N);
			}

			// shift back - only needed if computing hermite form
//			Sample_var->Dev_var = Sample_var->Dev_var-shift;

			// save conservation properties
			writeAppendToBinaryFile(1, &t_now, SettingsMain, "/Monitoring_data/Mesure/Time_s_"+ to_str(Sample_var->Grid->NX));  // time vector for data
			writeAppendToBinaryFile(1, mesure, SettingsMain, "/Monitoring_data/Mesure/Energy_"+ to_str(Sample_var->Grid->NX));
			writeAppendToBinaryFile(1, mesure+1, SettingsMain, "/Monitoring_data/Mesure/Enstrophy_"+ to_str(Sample_var->Grid->NX));
			writeAppendToBinaryFile(1, mesure+2, SettingsMain, "/Monitoring_data/Mesure/Palinstrophy_"+ to_str(Sample_var->Grid->NX));
			writeAppendToBinaryFile(1, mesure+3, SettingsMain, "/Monitoring_data/Mesure/Max_vorticity_"+ to_str(Sample_var->Grid->NX));
			writeAppendToBinaryFile(2, w_hash, SettingsMain, "/Monitoring_data/Mesure/Hash_vorticity_"+ to_str(Sample_var->Grid->NX));
			writeAppendToBinaryFile(2, psi_hash, SettingsMain, "/Monitoring_data/Mesure/Hash_stream_function_"+ to_str(Sample_var->Grid->NX));

			// construct message
			message = message + "Processed sample data " + to_str(i_save + 1) + " on grid " + to_str(Sample_var->Grid->NX) + ", Cons : Energ = " + to_str(mesure[0], 8)
					+    " \t Enstr = " + to_str(mesure[1], 8)
					+ " \t Palinstr = " + to_str(mesure[2], 8)
					+ " \t Wmax = " + to_str(mesure[3], 8);
		}
	}
	return message;
}





std::string sample_compute_and_write_vlasov(SettingsCMM SettingsMain, double t_now, double dt_now, double dt,
		MapStack Map_Stack, MapStack Map_Stack_f, TCudaGrid2D* Grid_sample, TCudaGrid2D Grid_discrete, double *Dev_sample,
		hipfftHandle* cufft_plan_sample_phi_1D, hipfftHandle* cufft_plan_sample_phi_1D_inverse, hipfftHandle* cufft_plan_sample_phi_2D, hipfftHandle* cufft_plan_sample_phi_2D_inverse, hipfftDoubleComplex *Dev_Temp_C1,
		double **Host_forward_particles_pos, double **Dev_forward_particles_pos, int *forward_particles_block, int forward_particles_thread,
		double *Dev_ChiX, double *Dev_ChiY, double *Dev_ChiX_f, double *Dev_ChiY_f, double *W_initial_discrete) {

//	// check if we want to save at this time, combine all variables if so
//	std::string message = "";
//	SaveSample* save_sample = SettingsMain.getSaveSample();
//	double mesure[4];  // it's fine if we only output it last time, thats enough i guess
//	for (int i_save = 0; i_save < SettingsMain.getSaveSampleNum(); ++i_save) {
//		// check each save and execute it independent
//		bool save_now = false;
//		// instants - distance to target is smaller than threshhold
//		if (save_sample[i_save].is_instant && t_now - save_sample[i_save].time_start + dt*1e-5 < dt_now && t_now - save_sample[i_save].time_start + dt*1e-5 >= 0) {
//			save_now = true;
//		}
//		// intervals - modulo to steps with safety-increased targets is smaller than step
//		if (!save_sample[i_save].is_instant
//			&& ((fmod(t_now - save_sample[i_save].time_start + dt*1e-5, save_sample[i_save].time_step) < dt_now
//			&& t_now + dt*1e-5 >= save_sample[i_save].time_start
//			&& t_now - dt*1e-5 <= save_sample[i_save].time_end)
//			|| t_now == save_sample[i_save].time_end)) {
//			save_now = true;
//		}
//
//		if (save_now) {
//			std::string save_var = save_sample[i_save].var;  // extract save variables
//			// forwards map to get it done
//			if (SettingsMain.getForwardMap()) {
//				// compute only if we actually want to save, elsewhise its a lot of computations for nothing
//				if (save_var.find("Map_f") != std::string::npos or save_var.find("Chi_f") != std::string::npos
//						or SettingsMain.getParticlesForwardedNum() > 0) {
//					// apply mapstack to map or particle positions
//					if (SettingsMain.getParticlesForwardedNum() == 0) {
//						apply_map_stack(Grid_sample[i_save], Map_Stack_f, Dev_ChiX_f, Dev_ChiY_f, (hipfftDoubleReal*)Dev_Temp_C1, 1);
//					}
//					// forwarded particles: forward all particles regardless of if they will be saved, needs rework to be more clever
//					else {
//						apply_map_stack_points(Grid_sample[i_save], Map_Stack_f, Dev_ChiX_f, Dev_ChiY_f, (hipfftDoubleReal*)Dev_Temp_C1, 1,
//								Dev_forward_particles_pos, (hipfftDoubleReal*)Dev_Temp_C1+2*Grid_sample[i_save].N,
//								SettingsMain, forward_particles_block, forward_particles_thread);
//					}
//
//					// save map by copying and saving offsetted data
//					if (save_var.find("Map_f") != std::string::npos or save_var.find("Chi_f") != std::string::npos) {
//						hipMemcpy2D(Dev_sample, sizeof(double), (hipfftDoubleReal*)Dev_Temp_C1, sizeof(double)*2,
//												 sizeof(double), Grid_sample[i_save].N, hipMemcpyDeviceToDevice);
//						writeTimeVariable(SettingsMain, "Map_ChiX_f_"+to_str(Grid_sample[i_save].NX),
//								t_now, Dev_sample, Grid_sample[i_save].N);
//						hipMemcpy2D(Dev_sample, sizeof(double), (hipfftDoubleReal*)Dev_Temp_C1+1, sizeof(double)*2,
//												 sizeof(double), Grid_sample[i_save].N, hipMemcpyDeviceToDevice);
//						writeTimeVariable(SettingsMain, "Map_ChiY_f_"+to_str(Grid_sample[i_save].NX),
//								t_now, Dev_sample, Grid_sample[i_save].N);
//					}
//
//				}
//
//			}
//
//			// compute map to initial condition through map stack
//			apply_map_stack(Grid_sample[i_save], Map_Stack, Dev_ChiX, Dev_ChiY, (hipfftDoubleReal*)Dev_Temp_C1, -1);
//
//			// save map by copying and saving offsetted data
//			if (save_var.find("Map_b") != std::string::npos or save_var.find("Chi_b") != std::string::npos) {
//				hipMemcpy2D(Dev_sample, sizeof(double), (hipfftDoubleReal*)Dev_Temp_C1, sizeof(double)*2,
//										 sizeof(double), Grid_sample[i_save].N, hipMemcpyDeviceToDevice);
//				writeTimeVariable(SettingsMain, "Map_ChiX_"+to_str(Grid_sample[i_save].NX),
//						t_now, Dev_sample, Grid_sample[i_save].N);
//				hipMemcpy2D(Dev_sample, sizeof(double), (hipfftDoubleReal*)Dev_Temp_C1+1, sizeof(double)*2,
//										 sizeof(double), Grid_sample[i_save].N, hipMemcpyDeviceToDevice);
//				writeTimeVariable(SettingsMain, "Map_ChiY_"+to_str(Grid_sample[i_save].NX),
//						t_now, Dev_sample, Grid_sample[i_save].N);
//			}
//
//			// passive scalar theta - 1 to switch for passive scalar
//			if (save_var.find("Scalar") != std::string::npos or save_var.find("Theta") != std::string::npos) {
//				k_h_sample_from_init<<<Grid_sample[i_save].blocksPerGrid, Grid_sample[i_save].threadsPerBlock>>>(Dev_sample, (hipfftDoubleReal*)Dev_Temp_C1,
//						Grid_sample[i_save], Grid_discrete, 1, SettingsMain.getScalarNum(), W_initial_discrete, SettingsMain.getScalarDiscrete());
//
//				writeTimeVariable(SettingsMain, "Scalar_Theta_"+to_str(Grid_sample[i_save].NX),
//						t_now, Dev_sample, Grid_sample[i_save].N);
//
//				thrust::device_ptr<double> scal_ptr = thrust::device_pointer_cast(Dev_sample);
//				double scal_int = Grid_sample[i_save].hx * Grid_sample[i_save].hy * thrust::reduce(scal_ptr, scal_ptr + Grid_sample[i_save].N, 0.0, thrust::plus<double>());
//				writeAppendToBinaryFile(1, &scal_int, SettingsMain, "/Monitoring_data/Mesure/Scalar_integral_"+ to_str(Grid_sample[i_save].NX));
//			}
//
//
//			int varnum = 2;
//			// compute distribution function
//			k_h_sample_from_init<<<Grid_sample[i_save].blocksPerGrid, Grid_sample[i_save].threadsPerBlock>>>(Dev_sample, (hipfftDoubleReal*)Dev_Temp_C1,
//					Grid_sample[i_save], Grid_discrete, varnum, SettingsMain.getInitialConditionNum(), W_initial_discrete, SettingsMain.getInitialDiscrete());
//
//			// save particle distribution function
//			if (save_var.find("Vorticity") != std::string::npos or save_var.find("W") != std::string::npos  or save_var.find("F") != std::string::npos) {
//				writeTimeVariable(SettingsMain, "Distribution_F_"+to_str(Grid_sample[i_save].NX),
//						t_now, Dev_sample, Grid_sample[i_save].N);
//			}
//
//			// compute enstrophy and palinstrophy
//			Compute_Kinetic_Energy(mesure[1], Dev_sample,(hipfftDoubleReal*) Dev_Temp_C1,  Grid_sample[i_save]);
//			Compute_Mass(mesure[3], Dev_sample, Grid_sample[i_save]);
//
//			// reuse sampled vorticity to compute psi, take fourier_hermite reshifting into account
//			long int shift = 2*Grid_sample[i_save].Nfft - Grid_sample[i_save].N;
//			get_psi_hermite_from_distribution_function(Dev_sample+shift, Dev_sample, Dev_Temp_C1, cufft_plan_sample_phi_1D[i_save], cufft_plan_sample_phi_1D_inverse[i_save],
//			cufft_plan_sample_phi_2D[i_save], cufft_plan_sample_phi_2D_inverse[i_save], Grid_sample[i_save], Grid_sample[i_save]); // Grid_sample[i_save] is the grid of the distribution function and the grid of Psi_real_out
//			Compute_Potential_Energy(mesure[2], Dev_sample+shift, Grid_sample[i_save]);
//
//			if (save_var.find("Stream") != std::string::npos or save_var.find("Psi") != std::string::npos) {
//				writeTimeVariable(SettingsMain, "Stream_function_Psi_"+to_str(Grid_sample[i_save].NX),
//						t_now, Dev_sample+shift, Grid_sample[i_save].N);
//			}
//
//			mesure[0] = mesure[1] + mesure[2]; // total energy
//			// printf("DEBUG:    Etot = %.4f, Ekin = %.4f, Epot = %.4f, Mass = %.4f\n", mesure[0], mesure[1], mesure[2], mesure[3]);
//			// error("babu", 1014);
//			writeAppendToBinaryFile(1, &t_now, SettingsMain, "/Monitoring_data/Mesure/Time_s");  // time vector for data
//			writeAppendToBinaryFile(1, mesure, SettingsMain, "/Monitoring_data/Mesure/Etot");
//			writeAppendToBinaryFile(1, mesure+1, SettingsMain, "/Monitoring_data/Mesure/Ekin");
//			writeAppendToBinaryFile(1, mesure+2, SettingsMain, "/Monitoring_data/Mesure/Epot");
//			writeAppendToBinaryFile(1, mesure+3, SettingsMain, "/Monitoring_data/Mesure/Mass");
//
//				// construct message
//			message = message + "Saved sample data " + to_str(i_save + 1) + " on grid " + to_str(Grid_sample[i_save].NX) + ", Cons : Etot = " + to_str(mesure[0], 8)
//					+    " \t Ekin = " + to_str(mesure[1], 8)
//					+ " \t Epot = " + to_str(mesure[2], 8)
//					+ " \t Mass = " + to_str(mesure[3], 8);
//		}
//	}
//	return message;
	return "";
}



/*******************************************************************
*							   Zoom								   *
*			sample vorticity with mapstack at arbitrary frame
*******************************************************************/
std::string compute_zoom(SettingsCMM SettingsMain, double t_now, double dt_now, double dt,
		MapStack Map_Stack, MapStack Map_Stack_f, std::map<std::string, CmmVar2D*> cmmVarMap, hipfftDoubleComplex *Dev_Temp_C1,
		double **Host_particles_pos, double **Dev_particles_pos,
		double **Host_forward_particles_pos, double **Dev_forward_particles_pos, int* forward_particles_block, int forward_particles_thread)
{
	// check if we want to save at this time, combine all variables if so
	std::string i_num = to_str(t_now); if (abs(t_now - T_MAX) < 1) i_num = "final";
	SaveZoom* save_zoom = SettingsMain.getSaveZoom();
	std::string message = "";
	for (int i_save = 0; i_save < SettingsMain.getSaveZoomNum(); ++i_save) {
		// check each save and execute it independent
		bool save_now = false;
		// instants - distance to target is smaller than threshhold
		if (save_zoom[i_save].is_instant && t_now - save_zoom[i_save].time_start + dt*1e-5 < dt_now && t_now - save_zoom[i_save].time_start + dt*1e-5 >= 0) {
			save_now = true;
		}
		// intervals - modulo to steps with safety-increased targets is smaller than step
		if (!save_zoom[i_save].is_instant
			&& ((fmod(t_now - save_zoom[i_save].time_start + dt*1e-5, save_zoom[i_save].time_step) < dt_now
			&& t_now + dt*1e-5 >= save_zoom[i_save].time_start
			&& t_now - dt*1e-5 <= save_zoom[i_save].time_end)
			|| t_now == save_zoom[i_save].time_end)) {
			save_now = true;
		}
		if (save_now) {
			CmmVar2D *Zoom_var = cmmVarMap["Zoom_" + to_str(i_save)];  // extract sample variable

			// create folder
			std::string sub_folder_name = "/Zoom_data/Time_" + i_num;
			std::string folder_name_now = SettingsMain.getWorkspace() + "data/" + SettingsMain.getFileName() + sub_folder_name;
			mkdir(folder_name_now.c_str(), 0777);
			message = message + "Processed zoom data " + to_str(i_save + 1) + " on grid " + to_str(Zoom_var->Grid->NX);

			std::string save_var = save_zoom[i_save].var;

			double x_min, x_max, y_min, y_max;

			double x_width = save_zoom[i_save].width_x/2.0;
			double y_width = save_zoom[i_save].width_y/2.0;

			// do repetetive zooms
			for(int zoom_ctr = 0; zoom_ctr < save_zoom[i_save].rep; zoom_ctr++){
				// create new subfolder for current zoom
				sub_folder_name = "/Zoom_data/Time_" + i_num + "/Zoom_" + to_str(i_save + 1) + "_rep_" + to_str(zoom_ctr);
				folder_name_now = SettingsMain.getWorkspace() + "data/" + SettingsMain.getFileName() + sub_folder_name;
				mkdir(folder_name_now.c_str(), 0777);

				// construct frame bounds for this zoom
				x_min = save_zoom[i_save].pos_x - x_width;
				x_max = save_zoom[i_save].pos_x + x_width;
				y_min = save_zoom[i_save].pos_y - y_width;
				y_max = save_zoom[i_save].pos_y + y_width;
				// safe bounds in array
				double bounds[4] = {x_min, x_max, y_min, y_max};

				printf("bounds - %f,  %f,  %f, %f", x_min, x_max, y_min, y_max);

				TCudaGrid2D Grid_zoom_i(Zoom_var->Grid->NX, Zoom_var->Grid->NY, Zoom_var->Grid->NZ, bounds);

				// compute forwards map for map stack of zoom window first, as it can be discarded afterwards
				if (SettingsMain.getForwardMap()) {
					// compute only if we actually want to save, elsewhise its a lot of computations for nothing
					if (save_var.find("Map_f") != std::string::npos or save_var.find("Chi_f") != std::string::npos
							or SettingsMain.getParticlesForwardedNum() > 0) {
						// apply mapstack to map or particle positions
						if (SettingsMain.getParticlesForwardedNum() == 0) {
							apply_map_stack(Grid_zoom_i, Map_Stack_f, cmmVarMap["ChiX_f"]->Dev_var, cmmVarMap["ChiY_f"]->Dev_var, (hipfftDoubleReal*)Dev_Temp_C1, 1);
						}
						// forwarded particles: forward all particles regardless of if they will be saved, needs rework to be more clever
						else {
							apply_map_stack_points(Grid_zoom_i, Map_Stack_f, cmmVarMap["ChiX_f"]->Dev_var, cmmVarMap["ChiY_f"]->Dev_var, (hipfftDoubleReal*)Dev_Temp_C1, 1,
									Dev_forward_particles_pos, (hipfftDoubleReal*)Dev_Temp_C1+2*Zoom_var->Grid->N,
									SettingsMain, forward_particles_block, forward_particles_thread);
						}

						// save map by copying and saving offsetted data
						if (save_var.find("Map_f") != std::string::npos or save_var.find("Chi_f") != std::string::npos) {
							hipMemcpy2D(Zoom_var->Dev_var, sizeof(double), (hipfftDoubleReal*)Dev_Temp_C1, sizeof(double)*2,
													 sizeof(double), Zoom_var->Grid->N, hipMemcpyDeviceToDevice);
							writeTimeVariable(SettingsMain, "Map_ChiX_f_"+to_str(Zoom_var->Grid->NX),
									t_now, Zoom_var->Dev_var, Zoom_var->Grid->N);
							hipMemcpy2D(Zoom_var->Dev_var, sizeof(double), (hipfftDoubleReal*)Dev_Temp_C1+1, sizeof(double)*2,
													 sizeof(double), Zoom_var->Grid->N, hipMemcpyDeviceToDevice);
							writeTimeVariable(SettingsMain, "Map_ChiY_f_"+to_str(Zoom_var->Grid->NX),
									t_now, Zoom_var->Dev_var, Zoom_var->Grid->N);
						}

						// save position of forwarded particles, go through all and only safe the needed ones
						ParticlesForwarded* particles_forwarded = SettingsMain.getParticlesForwarded();
						double* particles_out_counter = (hipfftDoubleReal*)Dev_Temp_C1+2*Grid_zoom_i.N;
						for (int i_p = 0; i_p < SettingsMain.getParticlesForwardedNum(); ++i_p) {
							if (save_var.find("PartF_" + to_str_0(i_p+1, 2)) != std::string::npos) {
								// copy data to host and save
								hipMemcpy(Host_forward_particles_pos[i_p], particles_out_counter, 2*particles_forwarded[i_p].num*sizeof(double), hipMemcpyDeviceToHost);
								double* part_pos = Host_particles_pos[i_p];

								int part_counter = 0;
								for (int p_num = 0; p_num < SettingsMain.getParticlesForwardedNum(); p_num++) {
									// check if particle in frame and then save it inside itself by checking for NaN values
									if (part_pos[2*p_num  ] == part_pos[2*p_num] and
										part_pos[2*p_num+1] == part_pos[2*p_num+1]) {
										// transcribe particle
										part_pos[2*part_counter  ] = part_pos[2*p_num];
										part_pos[2*part_counter+1] = part_pos[2*p_num+1];
										// increment counter
										part_counter++;
									}
								}
								// save particles
							    writeAllRealToBinaryFile(2*part_counter, Host_forward_particles_pos[i_p], SettingsMain, sub_folder_name+"/Particles_forwarded_pos_P" + to_str_0(i_p, 2));
							}
							particles_out_counter += 2*particles_forwarded[i_p].num;  // increase counter
						}
					}
				}



				// compute backwards map for map stack of zoom window
				apply_map_stack(Grid_zoom_i, Map_Stack, cmmVarMap["ChiX"]->Dev_var, cmmVarMap["ChiY"]->Dev_var, (hipfftDoubleReal*)Dev_Temp_C1+Grid_zoom_i.N, -1);

				// save map by copying and saving offsetted data
				if (save_var.find("Map_b") != std::string::npos or save_var.find("Chi_b") != std::string::npos) {
					hipMemcpy2D((hipfftDoubleReal*)Dev_Temp_C1, sizeof(double), (hipfftDoubleReal*)Dev_Temp_C1+Grid_zoom_i.N, sizeof(double)*2,
							sizeof(double), Grid_zoom_i.N, hipMemcpyDeviceToDevice);
					writeTranferToBinaryFile(Grid_zoom_i.N, (hipfftDoubleReal*)Dev_Temp_C1, SettingsMain, sub_folder_name+"/Map_ChiX_"+to_str(Grid_zoom_i.NX), false);
					hipMemcpy2D((hipfftDoubleReal*)Dev_Temp_C1, sizeof(double), (hipfftDoubleReal*)Dev_Temp_C1+Grid_zoom_i.N+1, sizeof(double)*2,
												sizeof(double), Grid_zoom_i.N, hipMemcpyDeviceToDevice);
					writeTranferToBinaryFile(Grid_zoom_i.N, (hipfftDoubleReal*)Dev_Temp_C1, SettingsMain, sub_folder_name+"/Map_ChiY_"+to_str(Grid_zoom_i.NX), false);
				}

				// passive scalar theta - 1 to switch for passive scalar
				if (save_var.find("Scalar") != std::string::npos or save_var.find("Theta") != std::string::npos) {
					k_h_sample_from_init<<<Grid_zoom_i.blocksPerGrid, Grid_zoom_i.threadsPerBlock>>>((hipfftDoubleReal*)Dev_Temp_C1, (hipfftDoubleReal*)Dev_Temp_C1+Grid_zoom_i.N,
							Grid_zoom_i, *cmmVarMap["Vort_discrete_init"]->Grid, 1, SettingsMain.getScalarNum(), cmmVarMap["Vort_discrete_init"]->Dev_var, SettingsMain.getScalarDiscrete());

					writeTranferToBinaryFile(Grid_zoom_i.N, (hipfftDoubleReal*)Dev_Temp_C1, SettingsMain, sub_folder_name+"/Scalar_Theta_"+to_str(Grid_zoom_i.NX), false);
				}

				// compute and save vorticity zoom
				if (save_var.find("Vorticity") != std::string::npos or save_var.find("W") != std::string::npos) {
					// compute vorticity - 0 to switch for vorticity
					k_h_sample_from_init<<<Grid_zoom_i.blocksPerGrid, Grid_zoom_i.threadsPerBlock>>>((hipfftDoubleReal*)Dev_Temp_C1, (hipfftDoubleReal*)Dev_Temp_C1+Grid_zoom_i.N,
							Grid_zoom_i, *cmmVarMap["Vort_discrete_init"]->Grid, 0, SettingsMain.getInitialConditionNum(), cmmVarMap["Vort_discrete_init"]->Dev_var, SettingsMain.getInitialDiscrete());

					writeTranferToBinaryFile(Grid_zoom_i.N, (hipfftDoubleReal*)Dev_Temp_C1, SettingsMain, sub_folder_name+"/Vorticity_W_"+to_str(Grid_zoom_i.NX), false);

					// hash
					double w_hash[2]; Hash_array((char*)w_hash, (hipfftDoubleReal*)Dev_Temp_C1, Grid_zoom_i.N);
					writeAppendToBinaryFile(2, w_hash, SettingsMain, "/Monitoring_data/Mesure/Hash_vorticity_Zoom_" + to_str(i_save + 1) + "_rep_" + to_str(zoom_ctr));
				}

				// compute sample of stream function - it's not a zoom though!
				if (save_var.find("Stream") != std::string::npos or save_var.find("Psi") != std::string::npos) {
					// sample stream function from hermite
					k_h_sample<<<Grid_zoom_i.blocksPerGrid,Grid_zoom_i.threadsPerBlock>>>(cmmVarMap["Psi"]->Dev_var, (hipfftDoubleReal*)Dev_Temp_C1, *cmmVarMap["Vort_discrete_init"]->Grid, Grid_zoom_i);

					// save psi zoom
					writeTranferToBinaryFile(Grid_zoom_i.N, (hipfftDoubleReal*)Dev_Temp_C1, SettingsMain, sub_folder_name+"/Stream_function_Psi_"+to_str(Grid_zoom_i.NX), false);

					// hash
					double psi_hash[2]; Hash_array((char*)psi_hash, (hipfftDoubleReal*)Dev_Temp_C1, Grid_zoom_i.N);
					writeAppendToBinaryFile(2, psi_hash, SettingsMain, "/Monitoring_data/Mesure/Hash_vorticity_Zoom_" + to_str(i_save + 1) + "_rep_" + to_str(zoom_ctr));

				}

				// safe particles in zoomframe if wanted
				ParticlesAdvected* particles_advected = SettingsMain.getParticlesAdvected();
				for (int i_p = 0; i_p < SettingsMain.getParticlesAdvectedNum(); ++i_p) {
					// particle position first
					if (save_var.find("PartA_" + to_str_0(i_p+1, 2)) != std::string::npos) {
						// copy particles to host
						hipMemcpy(Host_particles_pos[i_p], Dev_particles_pos[i_p], 2*particles_advected[i_p].num*sizeof(double), hipMemcpyDeviceToHost);
						double* part_pos = Host_particles_pos[i_p];

						int part_counter = 0;
						for (int i_pn = 0; i_pn < particles_advected[i_pn].num; i_pn++) {
							// check if particle in frame and then save it inside itself
							if (part_pos[2*i_pn  ] > x_min and part_pos[2*i_pn  ] < x_max and
								part_pos[2*i_pn+1] > y_min and part_pos[2*i_pn+1] < y_max) {
								// transcribe particle
								part_pos[2*part_counter  ] = part_pos[2*i_pn  ];
								part_pos[2*part_counter+1] = part_pos[2*i_pn+1];
								// increment counter
								part_counter++;
							}
						}
						// save particles
						writeAllRealToBinaryFile(2*part_counter, Host_particles_pos[i_p], SettingsMain, sub_folder_name+"/Particles_advected_pos_P" + to_str_0(i_p, 2));
					}
				}

				x_width *=  save_zoom[i_save].rep_fac;
				y_width *=  save_zoom[i_save].rep_fac;
			}
		}
	}
	return message;
}


// avoid overstepping specific time targets
double compute_next_timestep(SettingsCMM SettingsMain, double t, double dt) {
	double dt_now = dt;
	double dt_e = dt*1e-5;  // check for floating point arithmetic

	// 1st - particles computation start positions for advected and forwarded particles
	ParticlesAdvected* particles_advected = SettingsMain.getParticlesAdvected();
	for (int i_p = 0; i_p < SettingsMain.getParticlesAdvectedNum(); ++i_p) {
		if (t + dt_e - particles_advected[i_p].init_time < 0 && t + dt_e + dt - particles_advected[i_p].init_time > 0) {
			dt_now = fmin(dt_now, particles_advected[i_p].init_time - t);
		}
	}
	ParticlesForwarded* particles_forwarded = SettingsMain.getParticlesForwarded();
	for (int i_p = 0; i_p < SettingsMain.getParticlesForwardedNum(); ++i_p) {
		if (t + dt_e - particles_forwarded[i_p].init_time < 0 && t + dt_e + dt - particles_forwarded[i_p].init_time > 0) {
			dt_now = fmin(dt_now, particles_forwarded[i_p].init_time - t);
		}
	}

	// 2nd - save_computational for instant and interval
	SaveComputational* save_comp = SettingsMain.getSaveComputational();
	for (int i_save = 0; i_save < SettingsMain.getSaveComputationalNum(); ++i_save) {
		// instants - distance to target goes from negative to positive
		if (save_comp[i_save].is_instant && t + dt_e - save_comp[i_save].time_start < 0 && t + dt_e + dt - save_comp[i_save].time_start > 0) {
			dt_now = fmin(dt_now, save_comp[i_save].time_start - t);
		}
		// intervals - modulo to steps with safety-increased targets is smaller than current timestep
		if (!save_comp[i_save].is_instant
			&& (fmod(t + dt_e - save_comp[i_save].time_start, save_comp[i_save].time_step) > fmod(t + dt + dt_e - save_comp[i_save].time_start, save_comp[i_save].time_step)
			|| fmod(t + dt_e - save_comp[i_save].time_start, save_comp[i_save].time_step) < 0 && fmod(t + dt + dt_e - save_comp[i_save].time_start, save_comp[i_save].time_step) > 0)
			&& t + dt + dt_e >= save_comp[i_save].time_start && t - dt_e <= save_comp[i_save].time_end) {
			dt_now = fmin(dt_now, save_comp[i_save].time_step - fmod(t, save_comp[i_save].time_step));
		}
	}

	// 3nd - save_sample for instant and interval
	SaveSample* save_sample = SettingsMain.getSaveSample();
	for (int i_save = 0; i_save < SettingsMain.getSaveSampleNum(); ++i_save) {
		// instants - distance to target goes from negative to positive
		if (save_sample[i_save].is_instant && t + dt_e - save_sample[i_save].time_start < 0 && t + dt_e + dt - save_sample[i_save].time_start > 0) {
			dt_now = fmin(dt_now, save_sample[i_save].time_start - t);
		}
		// intervals - modulo to steps with safety-increased targets is smaller than current timestep
		if (!save_sample[i_save].is_instant
			&& (fmod(t + dt_e - save_sample[i_save].time_start, save_sample[i_save].time_step) > fmod(t + dt + dt_e - save_sample[i_save].time_start, save_sample[i_save].time_step)
			|| fmod(t + dt_e - save_sample[i_save].time_start, save_sample[i_save].time_step) < 0 && fmod(t + dt + dt_e - save_sample[i_save].time_start, save_sample[i_save].time_step) > 0)
			&& t + dt + dt_e >= save_sample[i_save].time_start && t - dt_e <= save_sample[i_save].time_end) {
			dt_now = fmin(dt_now, save_sample[i_save].time_step - fmod(t, save_sample[i_save].time_step));
		}
	}

	// 4th - save_zoom for instant and interval
	SaveZoom* save_zoom = SettingsMain.getSaveZoom();
	for (int i_save = 0; i_save < SettingsMain.getSaveZoomNum(); ++i_save) {
		// instants - distance to target goes from negative to positive
		if (save_zoom[i_save].is_instant && t + dt_e - save_zoom[i_save].time_start < 0 && t + dt_e + dt - save_zoom[i_save].time_start > 0) {
			dt_now = fmin(dt_now, save_zoom[i_save].time_start - t);
		}
		// intervals - modulo to steps with safety-increased targets is smaller than current timestep
		if (!save_zoom[i_save].is_instant
			&& (fmod(t + dt_e - save_zoom[i_save].time_start, save_zoom[i_save].time_step) > fmod(t + dt + dt_e - save_zoom[i_save].time_start, save_zoom[i_save].time_step)
			|| fmod(t + dt_e - save_zoom[i_save].time_start, save_zoom[i_save].time_step) < 0 && fmod(t + dt + dt_e - save_zoom[i_save].time_start, save_zoom[i_save].time_step) > 0)
			&& t + dt + dt_e >= save_zoom[i_save].time_start && t - dt_e <= save_zoom[i_save].time_end) {
			dt_now = fmin(dt_now, save_zoom[i_save].time_step - fmod(t, save_zoom[i_save].time_step));
		}
	}

	return dt_now;
}


/*******************************************************************
*				Zoom for a specific time instant				   *
*******************************************************************/

// We have to check that it still works.
/*
void Zoom_load_frame(string File, int grid_scale, int fine_grid_scale, string t_nb){


	double LX;
	int NXc, NYc;
	int NXsf, NYsf;
	int map_stack_ctr = 23;									// don't need it, it can be tertemined by the size of data loaded...

	LX = twoPI;
	NXc = NYc = grid_scale;
	NXsf = NYsf = fine_grid_scale;

	string simulationName = File;

	TCudaGrid2D Gc(NXc, NYc, LX);
	TCudaGrid2D Gsf(NXsf, NYsf, LX);


	double *ChiX, *ChiY, *ChiX_stack, *ChiY_stack;
	ChiX = new double[4*grid_scale*grid_scale];
	ChiY = new double[4*grid_scale*grid_scale];
	ChiX_stack = new double[map_stack_ctr * 4*Grid_coarse.sizeNReal];
	ChiY_stack = new double[map_stack_ctr * 4*Grid_coarse.sizeNReal];


	readAllRealFromBinaryFile(4*grid_scale*grid_scale, ChiX, simulationName, "ChiX_" + t_nb);
	readAllRealFromBinaryFile(4*grid_scale*grid_scale, ChiY, simulationName, "ChiY_" + t_nb);
	readAllRealFromBinaryFile(map_stack_ctr * 4*grid_scale*grid_scale, ChiX_stack, simulationName, "ChiX_stack_" + t_nb);
	readAllRealFromBinaryFile(map_stack_ctr * 4*grid_scale*grid_scale, ChiY_stack, simulationName, "ChiY_stack_" + t_nb);


	double *Dev_W_fine;
	hipMalloc((void**)&Dev_W_fine,  Grid_fine.sizeNReal);

	double *Dev_ChiX, *Dev_ChiY;
	hipMalloc((void**)&Dev_ChiX, 4*Grid_coarse.sizeNReal);
	hipMalloc((void**)&Dev_ChiY, 4*Grid_coarse.sizeNReal);

	double *Dev_ChiX_stack, *Dev_ChiY_stack;
	hipMalloc((void **) &Dev_ChiX_stack, map_stack_ctr * 4*Grid_coarse.sizeNReal);
	hipMalloc((void **) &Dev_ChiY_stack, map_stack_ctr * 4*Grid_coarse.sizeNReal);


	hipMemcpy(Dev_ChiX, ChiX, 4*Grid_coarse.sizeNReal, hipMemcpyHostToDevice);
	hipMemcpy(Dev_ChiY, ChiY, 4*Grid_coarse.sizeNReal, hipMemcpyHostToDevice);
	hipMemcpy(Dev_ChiX_stack, ChiX_stack, map_stack_ctr * 4*Grid_coarse.sizeNReal, hipMemcpyHostToDevice);
	hipMemcpy(Dev_ChiY_stack, ChiY_stack, map_stack_ctr * 4*Grid_coarse.sizeNReal, hipMemcpyHostToDevice);


	Zoom(simulationName, LX, Grid_coarse, Grid_fine, Dev_ChiX_stack, Dev_ChiY_stack, Dev_ChiX, Dev_ChiY, Dev_W_fine, map_stack_ctr);


	delete [] ChiX;
	delete [] ChiY;

	hipFree(Dev_W_fine);
	hipFree(Dev_ChiX);
	hipFree(Dev_ChiY);
	hipFree(Dev_ChiX_stack);
	hipFree(Dev_ChiY_stack);


	printf("Finished\n");

}
*/
