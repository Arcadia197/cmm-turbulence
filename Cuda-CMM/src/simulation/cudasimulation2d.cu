#include "hip/hip_runtime.h"
#include "cudasimulation2d.h"


////////////////////////////////////////////////////////////////////////
__global__ void kernel_init_diffeo(ptype *ChiX, ptype *ChiY, int NX, int NY, ptype h)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NX || iY >= NY)
		return;
		
	long int N = NX*NY;
	
	int In = iY*NX + iX;	
	
	ChiX[In] = iX*h;
	ChiY[In] = iY*h;
	
	ChiX[1*N+In] = ChiY[2*N+In] = 1; 
	
	ChiX[2*N+In] = ChiY[1*N+In] = 
	ChiX[3*N+In] = ChiY[3*N+In] = 0;
}


__global__ void kernel_update_map_from_dual(ptype *ChiX, ptype *ChiY, ptype *X, ptype *Y, int NX, int NY, ptype ep)																												
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NX || iY >= NY)
		return;
	
	int In = iY*NX + iX;	
	
	double c1 = 1.0/(4.0);
	double c2 = 1.0/(4.0*ep);
	double c3 = 1.0/(4.0*ep*ep);
	
	long int N = NX*NY;
	
	ChiX[    In] = ( (X[0*N+In] + X[1*N+In] + X[2*N+In] + X[3*N+In])*c1 );
	ChiY[    In] = ( (Y[0*N+In] + Y[1*N+In] + Y[2*N+In] + Y[3*N+In])*c1 );
		
	ChiX[1*N+In] = ( (X[0*N+In] + X[1*N+In] - X[2*N+In] - X[3*N+In])*c2 );
	ChiY[1*N+In] = ( (Y[0*N+In] + Y[1*N+In] - Y[2*N+In] - Y[3*N+In])*c2 );
		
	ChiX[2*N+In] = ( (X[0*N+In] - X[1*N+In] - X[2*N+In] + X[3*N+In])*c2 );
	ChiY[2*N+In] = ( (Y[0*N+In] - Y[1*N+In] - Y[2*N+In] + Y[3*N+In])*c2 );
		
	ChiX[3*N+In] = ( (X[0*N+In] - X[1*N+In] + X[2*N+In] - X[3*N+In])*c3 );
	ChiY[3*N+In] = ( (Y[0*N+In] - Y[1*N+In] + Y[2*N+In] - Y[3*N+In])*c3 );
}


__global__ void upsample(ptype *ChiX, ptype *ChiY, ptype *ChiX_2048, ptype *ChiY_2048, int NXc, int NYc, ptype hc, int NXs, int NYs, ptype hs)															// time cost
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NXs || iY >= NYs)
		return;
	
	int In = iY*NXs + iX;	
	
	//position
	ptype x = iX*hs;
	ptype y = iY*hs;
	
	device_diffeo_interpolate(ChiX, ChiY, x, y, &x, &y, NXc, NYc, hc);		
	
	ChiX_2048[In] = x;
	ChiY_2048[In] = y;
}


__global__ void kernel_incompressibility_check(ptype *ChiX, ptype *ChiY, ptype *gradChi, int NXc, int NYc, ptype hc, int NXs, int NYs, ptype hs)															// time cost
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NXs || iY >= NYs)
		return;
	
	int In = iY*NXs + iX;	
	
	//position
	ptype x = iX*hs + 0.5*hs;
	ptype y = iY*hs + 0.5*hs;
	
	gradChi[In] = device_diffeo_grad(ChiX, ChiY, x, y, NXc, NYc, hc);
}


__global__ void kernel_advect_using_stream_hermite2(ptype *ChiX, ptype *ChiY, ptype *ChiDualX, ptype *ChiDualY, ptype *phi, ptype *phi_p, ptype *phi_p_p, int NXc, int NYc, ptype hc, ptype t, ptype dt, ptype ep)			// time cost
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
    /*int NX = 512; //Dans l'interpolation remettre NXc Nyc hc
    int NY = 512;
    ptype h = twoPI/(float)NX;*/

    if(iX >= NXc || iY >= NYc)
		return;
	
	int In = iY*NXc + iX;
	
	long int N = NXc*NYc;
	
	//position
	ptype x = iX*hc;
	ptype y = iY*hc;
	
	//running through 4 neighbours (unrolled loops)
	ptype xep, yep;
	ptype phi_x, phi_y;
	
	
	ptype u, v, u_p, v_p, u_p_p, v_p_p;  // velocity at current and previous time steps
	ptype k1_x, k1_y, k2_x, k2_y, k3_x, k3_y;  // different intermediate functions for RKThree
	ptype l[6] = {3.0,-3.0,1,1.875,-1.25,0.375};  // coefficients for lagrangian time interpolation of velocity
    ptype xf, yf;  // coordinates in loop
    
	ptype xf_p, yf_p;  // variables used for fixed-point iteration in ABTwo

    //ptype l[6] = {L1(t+dt, t, t-dt, t-2*dt), L2(t+dt, t, t-dt, t-2*dt), L3(t+dt, t, t-dt, t-2*dt), L1(t+dt/2, t, t-dt, t-2*dt), L2(t+dt/2, t, t-dt, t-2*dt), L3(t+dt/2, t, t-dt, t-2*dt)}; Work but slow

	// repeat for all 4 footpoints
    #pragma unroll 4
	FOR(k, 4)
	{
		// get position of footpoint
		xep = x+ep*(1 - 2*((k/2)%2 == 1));
		yep = y+ep*(1 - 2*(((k+1)/2)%2 == 1));
		
		xf = xep;
		yf = yep;

        if (TIME_INTEGRATION == "EulerExp") {
			device_hermite_interpolate_dx_dy(phi, xf, yf, &phi_x, &phi_y, NXc, NYc, hc);
			u_p =  phi_y;
			v_p = -phi_x;

			xf = xep - dt * u_p;
			yf = yep - dt * v_p;
        }

		// Adam-Bashfords of order two, where a fixed point iteration with 10 iterations is utilized
        else if (TIME_INTEGRATION == "ABTwo") {
			xf_p = xep; yf_p = yep;

			// fixed point iteration for xf,yf using previous foot points (self-correction)
            #pragma unroll 10
            FOR(ctr, 10)
            {
				//step 1
                device_hermite_interpolate_dx_dy(phi_p, xf_p, yf_p, &phi_x, &phi_y, NXc, NYc, hc);
                u_p =  phi_y; v_p = -phi_x;

                xf_p = xf - dt * u_p;
                yf_p = yf - dt * v_p;

				//step 2
                device_hermite_interpolate_dx_dy(phi_p, xf_p, yf_p, &phi_x, &phi_y, NXc, NYc, hc);
                u_p =  phi_y; v_p = -phi_x;

                device_hermite_interpolate_dx_dy(phi, xf , yf, &phi_x, &phi_y, NXc, NYc, hc);
                u =  phi_y; v = -phi_x;

                xf = xep - dt * (1.5*u - 0.5*u_p);
                yf = yep - dt * (1.5*v - 0.5*v_p);

            }
		}

		// RKThree time step utilizing some intermediate steps
        else if (TIME_INTEGRATION == "RKThree") {
			// compute u_tilde(X,t_n+1)
			device_hermite_interpolate_dx_dy(phi_p_p, xep, yep, &phi_x, &phi_y, NXc, NYc, hc);
			u_p_p = phi_y; v_p_p = -phi_x;

			device_hermite_interpolate_dx_dy(phi_p, xep, yep, &phi_x, &phi_y, NXc, NYc, hc);
			u_p = phi_y; v_p = -phi_x;

			device_hermite_interpolate_dx_dy(phi, xep, yep, &phi_x, &phi_y, NXc, NYc, hc);
			u = phi_y; v = -phi_x;

			// k1 = u_tilde(x,t_n+1)
			k1_x = l[0] * u + l[1] * u_p + l[2] * u_p_p;
			k1_y = l[0] * v + l[1] * v_p + l[2] * v_p_p;

			// compute u_tilde(x - dt*k1/2, t_n+1 - dt/2)
			device_hermite_interpolate_dx_dy(phi_p_p, xep - dt*k1_x/2, yep - dt*k1_y/2, &phi_x, &phi_y, NXc, NYc, hc);
			u_p_p = phi_y; v_p_p = -phi_x;

			device_hermite_interpolate_dx_dy(phi_p, xep - dt*k1_x/2, yep - dt*k1_y/2, &phi_x, &phi_y, NXc, NYc, hc);
			u_p = phi_y; v_p = -phi_x;

			device_hermite_interpolate_dx_dy(phi, xep - dt*k1_x/2, yep - dt*k1_y/2, &phi_x, &phi_y, NXc, NYc, hc);
			u = phi_y; v = -phi_x;

			//k2 = u_tilde(x - k1 dt/2, t_n+1 - dt/2)
			k2_x = l[3] * u + l[4]* u_p + l[5] * u_p_p;
			k2_y = l[3] * v + l[4] * v_p + l[5] * v_p_p;

			//compute u_tilde(x + dt * k1 - 2*dt*k2, t_n+1 - dt)
			device_hermite_interpolate_dx_dy(phi, xep + dt*k1_x - 2*dt*k2_x, yep + dt*k1_y - 2*dt*k2_y , &phi_x, &phi_y, NXc, NYc, hc);
			u = phi_y; v = -phi_x;

			// k3 = u_tilde(x = k1 dt - 2 k2 dt, t_n)
			k3_x = u;
			k3_y = v;

			xf = xep - dt * (k1_x + 4*k2_x + k3_x)/6;
			yf = yep - dt * (k1_y + 4*k2_y + k3_y)/6;
		}

        // scheme name not known, do euler explicit to avoid errors (a bit cheeky, i know, but i dont know how to throw errors yet)
        else {
			device_hermite_interpolate_dx_dy(phi, xf, yf, &phi_x, &phi_y, NXc, NYc, hc);
			u_p =  phi_y; v_p = -phi_x; xf = xep - dt * u_p; yf = yep - dt * v_p;
        }

		device_diffeo_interpolate(ChiX, ChiY, xf, yf, &ChiDualX[k*N+In], &ChiDualY[k*N+In], NXc, NYc, hc);
	}
}


/*******************************************************************
*						 Real and Complex						   *
*******************************************************************/
	
__global__ void kernel_real_to_complex(ptype *varR, cuPtype *varC, int NX, int NY)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NX || iY >= NY)
		return;
	
	int In = iY*NX + iX;	
	
	varC[In].x = varR[In];
	varC[In].y = 0.0;
}


__global__ void kernel_complex_to_real(ptype *varR, cuPtype *varC, int NX, int NY)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NX || iY >= NY)
		return;
	
	int In = iY*NX + iX;	
	
	varR[In] = varC[In].x;
}


__global__ void kernel_real_to_complex_H(ptype *varR, cuPtype *varC, int NX, int NY)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NX || iY >= NY)
		return;
	
	int In = iY*NX + iX;	
	long int N = NX*NY;
	
	varC[In].x = varR[In];
	varC[N+In].x = varR[N+In];
	varC[2*N+In].x = varR[2*N+In];
	varC[3*N+In].x = varR[3*N+In];
	varC[In].y = varC[N+In].y = varC[2*N+In].y = varC[3*N+In].y = 0.0;
	
}


__global__ void kernel_complex_to_real_H(ptype *varR, cuPtype *varC, int NX, int NY)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NX || iY >= NY)
		return;
	
	int In = iY*NX + iX;	
	long int N = NX*NY;
	
	varR[In] = varC[In].x;
	varR[N+In] = varC[N+In].x;
	varR[2*N+In] = varC[2*N+In].x;
	varR[3*N+In] = varC[3*N+In].x;
}



/*******************************************************************
*						 Apply remapping						   *
*******************************************************************/


__global__ void kernel_apply_map_stack_to_W(ptype *ChiX_stack, ptype *ChiY_stack, ptype *ChiX, ptype *ChiY, ptype *ws, int stack_length, int NXc, int NYc, ptype hc, int NXs, int NYs, ptype hs, ptype *W_initial)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NXs || iY >= NYs)
		return;
	
	int In = iY*NXs + iX;
	long int N = NXc*NYc;	
	
	//position
	ptype x = iX*hs;
	ptype y = iY*hs;
	
	device_diffeo_interpolate(ChiX, ChiY, x, y, &x, &y, NXc, NYc, hc);		
	for(int k = stack_length - 1; k >= 0; k--)
		device_diffeo_interpolate(&ChiX_stack[k*N*4], &ChiY_stack[k*N*4], x, y, &x, &y, NXc, NYc, hc);		
	
	#ifndef DISCRET
		ws[In] = device_initial_W(x, y);
	#endif
	
	#ifdef DISCRET
		ws[In] = device_hermite_interpolate(W_initial, x, y, NXs, NYs, hs);
		//ws[In] = device_initial_W_discret(x, y, W_initial, NXs, NYs);
	#endif
	
}


__global__ void kernel_apply_map_stack_to_W_custom(ptype *ChiX_stack, ptype *ChiY_stack, ptype *ChiX, ptype *ChiY, ptype *ws, int stack_length, int NXc, int NYc, ptype hc, int NXs, int NYs, ptype hs, ptype xl, ptype xr, ptype yl, ptype yr, ptype *W_initial)		// Zoom
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NXs || iY >= NYs)
		return;
	
	int In = iY*NXs + iX;
	long int N = NXc*NYc;	
	
	ptype htemp = (xr - xl)/NXs;
	
	//position
	ptype x = xl + iX*htemp;
	ptype y = yl + iY*htemp;
	
	device_diffeo_interpolate(ChiX, ChiY, x, y, &x, &y, NXc, NYc, hc);
	for(int k = stack_length - 1; k >= 0; k--)
		device_diffeo_interpolate(&ChiX_stack[k*N*4], &ChiY_stack[k*N*4], x, y, &x, &y, NXc, NYc, hc);
		
	ws[In] = device_initial_W(x, y); //device_initial_W_discret(x, y)
	
	#ifndef DISCRET
		ws[In] = device_initial_W(x, y);
	#endif
	
	#ifdef DISCRET
		ws[In] = device_hermite_interpolate(W_initial, x, y, NXs, NYs, hs);
		//ws[In] = device_initial_W_discret(x, y, W_initial, NXs, NYs);
	#endif
	
}

/*
void kernel_apply_map_stack_to_W_part_All(TCudaGrid2D *Grid_coarse, TCudaGrid2D *Grid_fine, ptype *ChiX_stack, ptype *ChiY_stack, ptype *ChiX, ptype *ChiY, ptype *ChiX_stack_RAM, ptype *ChiY_stack_RAM, ptype *W_real, cuPtype *Dev_Complex_fine, int stack_length, int map_stack_length, int stack_length_RAM, int NXc, int NYc, ptype hc, int NXs, int NYs, ptype hs, ptype *W_initial)
{
	
	kernel_apply_map_stack_to_W_part_1<<<Grid_fine->blocksPerGrid, Grid_fine->threadsPerBlock>>>(ChiX, ChiY, Dev_Complex_fine, Grid_coarse->NX, Grid_coarse->NY, Grid_coarse->h, Grid_fine->NX, Grid_fine->NY, Grid_fine->h);
	for(int K = stack_length_RAM; K >= 0; K--){
	 	hipMemcpy(ChiX_stack, &ChiX_stack_RAM[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
		hipMemcpy(ChiY_stack, &ChiY_stack_RAM[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
		if (K == stack_length_RAM){
			for(int k = stack_length - stack_length_RAM*map_stack_length - 1; k >= 0; k--){
				kernel_apply_map_stack_to_W_part_2<<<Grid_fine->blocksPerGrid, Grid_fine->threadsPerBlock>>>(ChiX_stack, ChiY_stack, Dev_Complex_fine, Grid_coarse->NX, Grid_coarse->NY, Grid_coarse->h, Grid_fine->NX, Grid_fine->NY, k);
			}
		}
		else{
			for(int k = map_stack_length - 1; k >= 0; k--){
				kernel_apply_map_stack_to_W_part_2<<<Grid_fine->blocksPerGrid, Grid_fine->threadsPerBlock>>>(ChiX_stack, ChiY_stack, Dev_Complex_fine, Grid_coarse->NX, Grid_coarse->NY, Grid_coarse->h, Grid_fine->NX, Grid_fine->NY, k);
			}
		}
	}
	kernel_apply_map_stack_to_W_part_3<<<Grid_fine->blocksPerGrid, Grid_fine->threadsPerBlock>>>(W_real, Dev_Complex_fine, Grid_fine->NX, Grid_fine->NY, Grid_fine->h, W_initial);
	
}*/


void kernel_apply_map_stack_to_W_part_All(TCudaGrid2D *Grid_coarse, TCudaGrid2D *Grid_fine, ptype *ChiX_stack, ptype *ChiY_stack, ptype *ChiX, ptype *ChiY, ptype *Host_ChiX_stack_RAM_0, ptype *Host_ChiY_stack_RAM_0, ptype *Host_ChiX_stack_RAM_1, ptype *Host_ChiY_stack_RAM_1, ptype *Host_ChiX_stack_RAM_2, ptype *Host_ChiY_stack_RAM_2, ptype *Host_ChiX_stack_RAM_3, ptype *Host_ChiY_stack_RAM_3, ptype *W_real, cuPtype *Dev_Complex_fine, int stack_length, int map_stack_length, int stack_length_RAM, int stack_length_Nb_array_RAM, int mem_RAM, int NXc, int NYc, ptype hc, int NXs, int NYs, ptype hs, ptype *W_initial)
{
	
	kernel_apply_map_stack_to_W_part_1<<<Grid_fine->blocksPerGrid, Grid_fine->threadsPerBlock>>>(ChiX, ChiY, Dev_Complex_fine, Grid_coarse->NX, Grid_coarse->NY, Grid_coarse->h, Grid_fine->NX, Grid_fine->NY, Grid_fine->h);
	
	for(int K_RAM = stack_length_Nb_array_RAM; K_RAM >= 0; K_RAM--){
		if (K_RAM == stack_length_Nb_array_RAM){
			for(int K = stack_length_RAM%mem_RAM; K >= 0; K--){
				if (K_RAM == 0){
					hipMemcpy(ChiX_stack, &Host_ChiX_stack_RAM_0[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
					hipMemcpy(ChiY_stack, &Host_ChiY_stack_RAM_0[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
				}
				if (K_RAM == 1){
					hipMemcpy(ChiX_stack, &Host_ChiX_stack_RAM_1[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
					hipMemcpy(ChiY_stack, &Host_ChiY_stack_RAM_1[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
				}
				if (K_RAM == 2){
					hipMemcpy(ChiX_stack, &Host_ChiX_stack_RAM_2[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
					hipMemcpy(ChiY_stack, &Host_ChiY_stack_RAM_2[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
				}
				if (K_RAM == 3){
					hipMemcpy(ChiX_stack, &Host_ChiX_stack_RAM_3[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
					hipMemcpy(ChiY_stack, &Host_ChiY_stack_RAM_3[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
				}
				if (K == stack_length_RAM%mem_RAM){
					for(int k = stack_length - stack_length_RAM*map_stack_length - 1; k >= 0; k--){
						kernel_apply_map_stack_to_W_part_2<<<Grid_fine->blocksPerGrid, Grid_fine->threadsPerBlock>>>(ChiX_stack, ChiY_stack, Dev_Complex_fine, Grid_coarse->NX, Grid_coarse->NY, Grid_coarse->h, Grid_fine->NX, Grid_fine->NY, k);
					}
				}
				else{
					for(int k = map_stack_length - 1; k >= 0; k--){
						kernel_apply_map_stack_to_W_part_2<<<Grid_fine->blocksPerGrid, Grid_fine->threadsPerBlock>>>(ChiX_stack, ChiY_stack, Dev_Complex_fine, Grid_coarse->NX, Grid_coarse->NY, Grid_coarse->h, Grid_fine->NX, Grid_fine->NY, k);
					}
				}
			}
		}
		else{
			for(int K = mem_RAM-1; K >= 0; K--){
				if (K_RAM == 0){
					hipMemcpy(ChiX_stack, &Host_ChiX_stack_RAM_0[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
					hipMemcpy(ChiY_stack, &Host_ChiY_stack_RAM_0[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
				}
				if (K_RAM == 1){
					hipMemcpy(ChiX_stack, &Host_ChiX_stack_RAM_1[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
					hipMemcpy(ChiY_stack, &Host_ChiY_stack_RAM_1[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
				}
				if (K_RAM == 2){
					hipMemcpy(ChiX_stack, &Host_ChiX_stack_RAM_2[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
					hipMemcpy(ChiY_stack, &Host_ChiY_stack_RAM_2[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
				}
				if (K_RAM == 3){
					hipMemcpy(ChiX_stack, &Host_ChiX_stack_RAM_3[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
					hipMemcpy(ChiY_stack, &Host_ChiY_stack_RAM_3[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
				}
				for(int k = map_stack_length - 1; k >= 0; k--){
					kernel_apply_map_stack_to_W_part_2<<<Grid_fine->blocksPerGrid, Grid_fine->threadsPerBlock>>>(ChiX_stack, ChiY_stack, Dev_Complex_fine, Grid_coarse->NX, Grid_coarse->NY, Grid_coarse->h, Grid_fine->NX, Grid_fine->NY, k);
				}
			}
		}
	}
	
	kernel_apply_map_stack_to_W_part_3<<<Grid_fine->blocksPerGrid, Grid_fine->threadsPerBlock>>>(W_real, Dev_Complex_fine, Grid_fine->NX, Grid_fine->NY, Grid_fine->h, W_initial);
	
}


__global__ void kernel_apply_map_stack_to_W_part_1(ptype *ChiX, ptype *ChiY, cuPtype *x_y, int NXc, int NYc, ptype hc, int NXs, int NYs, ptype hs)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NXs || iY >= NYs)
		return;
	
	int In = iY*NXs + iX;
	
	//position
	ptype x = iX*hs;
	ptype y = iY*hs;
	
	device_diffeo_interpolate(ChiX, ChiY, x, y, &x, &y, NXc, NYc, hc);
	
	x_y[In].x = x;
	x_y[In].y = y;
	
}

__global__ void kernel_apply_map_stack_to_W_part_2(ptype *ChiX_stack, ptype *ChiY_stack, cuPtype *x_y, int NXc, int NYc, ptype hc, int NXs, int NYs, int k)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NXs || iY >= NYs)
		return;
	
	int In = iY*NXs + iX;
	long int N = NXc*NYc;	
	
	//for(int k = stack_length - 1; k >= 0; k--)
	device_diffeo_interpolate(&ChiX_stack[k*N*4], &ChiY_stack[k*N*4], x_y[In].x, x_y[In].y, &x_y[In].x, &x_y[In].y, NXc, NYc, hc);
	
}

__global__ void kernel_apply_map_stack_to_W_part_3(ptype *ws, cuPtype *x_y, int NXs, int NYs, ptype hs, ptype *W_initial)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NXs || iY >= NYs)
		return;
	
	int In = iY*NXs + iX;
	
	#ifndef DISCRET
		ws[In] = device_initial_W(x_y[In].x, x_y[In].y);
	#endif
	
	#ifdef DISCRET
		ws[In] = device_hermite_interpolate(W_initial, x_y[In].x, x_y[In].y, NXs, NYs, hs);
	#endif
	
}


void kernel_apply_map_stack_to_W_custom_part_All(TCudaGrid2D *Grid_coarse, TCudaGrid2D *Grid_fine, ptype *ChiX_stack, ptype *ChiY_stack, ptype *ChiX, ptype *ChiY, ptype *Host_ChiX_stack_RAM_0, ptype *Host_ChiY_stack_RAM_0, ptype *Host_ChiX_stack_RAM_1, ptype *Host_ChiY_stack_RAM_1, ptype *Host_ChiX_stack_RAM_2, ptype *Host_ChiY_stack_RAM_2, ptype *Host_ChiX_stack_RAM_3, ptype *Host_ChiY_stack_RAM_3, ptype *W_real, cuPtype *Dev_Complex_fine, int stack_length, int map_stack_length, int stack_length_RAM, int stack_length_Nb_array_RAM, int mem_RAM, int NXc, int NYc, ptype hc, int NXs, int NYs, ptype hs, ptype xl, ptype xr, ptype yl, ptype yr, ptype *W_initial)
{
	
	kernel_apply_map_stack_to_W_custom_part_1<<<Grid_fine->blocksPerGrid, Grid_fine->threadsPerBlock>>>(ChiX, ChiY, Dev_Complex_fine, Grid_coarse->NX, Grid_coarse->NY, Grid_coarse->h, Grid_fine->NX, Grid_fine->NY, Grid_fine->h, xl, xr, yl, yr);
	
	for(int K_RAM = stack_length_Nb_array_RAM; K_RAM >= 0; K_RAM--){
		if (K_RAM == stack_length_Nb_array_RAM){
			for(int K = stack_length_RAM%mem_RAM; K >= 0; K--){
				if (K_RAM == 0){
					hipMemcpy(ChiX_stack, &Host_ChiX_stack_RAM_0[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
					hipMemcpy(ChiY_stack, &Host_ChiY_stack_RAM_0[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
				}
				if (K_RAM == 1){
					hipMemcpy(ChiX_stack, &Host_ChiX_stack_RAM_1[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
					hipMemcpy(ChiY_stack, &Host_ChiY_stack_RAM_1[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
				}
				if (K_RAM == 2){
					hipMemcpy(ChiX_stack, &Host_ChiX_stack_RAM_2[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
					hipMemcpy(ChiY_stack, &Host_ChiY_stack_RAM_2[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
				}
				if (K_RAM == 3){
					hipMemcpy(ChiX_stack, &Host_ChiX_stack_RAM_3[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
					hipMemcpy(ChiY_stack, &Host_ChiY_stack_RAM_3[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
				}
				if (K == stack_length_RAM%mem_RAM){
					for(int k = stack_length - stack_length_RAM*map_stack_length - 1; k >= 0; k--){
						kernel_apply_map_stack_to_W_part_2<<<Grid_fine->blocksPerGrid, Grid_fine->threadsPerBlock>>>(ChiX_stack, ChiY_stack, Dev_Complex_fine, Grid_coarse->NX, Grid_coarse->NY, Grid_coarse->h, Grid_fine->NX, Grid_fine->NY, k);
					}
				}
				else{
					for(int k = map_stack_length - 1; k >= 0; k--){
						kernel_apply_map_stack_to_W_part_2<<<Grid_fine->blocksPerGrid, Grid_fine->threadsPerBlock>>>(ChiX_stack, ChiY_stack, Dev_Complex_fine, Grid_coarse->NX, Grid_coarse->NY, Grid_coarse->h, Grid_fine->NX, Grid_fine->NY, k);
					}
				}
			}
		}
		else{
			for(int K = mem_RAM-1; K >= 0; K--){
				if (K_RAM == 0){
					hipMemcpy(ChiX_stack, &Host_ChiX_stack_RAM_0[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
					hipMemcpy(ChiY_stack, &Host_ChiY_stack_RAM_0[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
				}
				if (K_RAM == 1){
					hipMemcpy(ChiX_stack, &Host_ChiX_stack_RAM_1[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
					hipMemcpy(ChiY_stack, &Host_ChiY_stack_RAM_1[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
				}
				if (K_RAM == 2){
					hipMemcpy(ChiX_stack, &Host_ChiX_stack_RAM_2[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
					hipMemcpy(ChiY_stack, &Host_ChiY_stack_RAM_2[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
				}
				if (K_RAM == 3){
					hipMemcpy(ChiX_stack, &Host_ChiX_stack_RAM_3[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
					hipMemcpy(ChiY_stack, &Host_ChiY_stack_RAM_3[K*map_stack_length*Grid_coarse->N*4], map_stack_length * 4*Grid_coarse->sizeNReal, hipMemcpyHostToDevice);
				}
				for(int k = map_stack_length - 1; k >= 0; k--){
					kernel_apply_map_stack_to_W_part_2<<<Grid_fine->blocksPerGrid, Grid_fine->threadsPerBlock>>>(ChiX_stack, ChiY_stack, Dev_Complex_fine, Grid_coarse->NX, Grid_coarse->NY, Grid_coarse->h, Grid_fine->NX, Grid_fine->NY, k);
				}
			}
		}
	}
	
	kernel_apply_map_stack_to_W_part_3<<<Grid_fine->blocksPerGrid, Grid_fine->threadsPerBlock>>>(W_real, Dev_Complex_fine, Grid_fine->NX, Grid_fine->NY, Grid_fine->h, W_initial);
	
}


__global__ void kernel_apply_map_stack_to_W_custom_part_1(ptype *ChiX, ptype *ChiY, cuPtype *x_y, int NXc, int NYc, ptype hc, int NXs, int NYs, ptype hs, ptype xl, ptype xr, ptype yl, ptype yr)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NXs || iY >= NYs)
		return;
	
	int In = iY*NXs + iX;
	
	ptype htemp = (xr - xl)/NXs;
	
	//position
	ptype x = xl + iX*htemp;
	ptype y = yl + iY*htemp;
	
	device_diffeo_interpolate(ChiX, ChiY, x, y, &x, &y, NXc, NYc, hc);
	
	x_y[In].x = x;
	x_y[In].y = y;
	
}


__global__ void cut_off_scale(cuPtype *W, int NX)
{
	
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	int In = iY*NX + iX;
	
	int i = In/NX;
	int j = In%NX;
	if (i <= NX/2 && j <= NX/2)
		if ((i*i + j*j) > NX*NX/9){
			W[In].x = 0;
			W[In].y = 0;
		}
	if (i >= NX/2 && j <= NX/2)
		if (((NX-i)*(NX-i) + j*j) > NX*NX/9){
			W[In].x = 0;
			W[In].y = 0;
		}
	if (i <= NX/2 && j >= NX/2)
		if ((i*i + (NX-j)*(NX-j)) > NX*NX/9){
			W[In].x = 0;
			W[In].y = 0;
		}
	if (i >= NX/2 && j >= NX/2)
		if (((NX-i)*(NX-i) + (NX-j)*(NX-j)) > NX*NX/9){
			W[In].x = 0;
			W[In].y = 0;
		}
	
	if (In == 0){
		W[In].x = 0;
		W[In].y = 0;
	}
	
}


/*******************************************************************
*						 										   *
*******************************************************************/


__global__ void kernel_compare_vorticity_with_initial(ptype *ChiX_stack, ptype *ChiY_stack, ptype *ChiX, ptype *ChiY, ptype *error, int stack_length, int NXc, int NYc, ptype hc, int NXs, int NYs, ptype hs)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NXs || iY >= NYs)
		return;
	
	int In = iY*NXs + iX;
	long int N = NXc*NYc;	
	
	//position
	ptype x = iX*hs;
	ptype y = iY*hs;
	
	device_diffeo_interpolate(ChiX, ChiY, x, y, &x, &y, NXc, NYc, hc);		
	for(int k = stack_length - 1; k >= 0; k--)
		device_diffeo_interpolate(&ChiX_stack[k*N*4], &ChiY_stack[k*N*4], x, y, &x, &y, NXc, NYc, hc);		
	
	error[In] = fabs(device_initial_W(x, y) - device_initial_W(iX*hs, iY*hs));
}




__global__ void kernel_apply_map_and_sample_from_hermite(ptype *ChiX, ptype *ChiY, ptype *fs, ptype *H, int NXc, int NYc, ptype hc, int NXs, int NYs, ptype hs, int NXh, int NYh, ptype hh)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
    /*int NX = 128; //Dans l'interpolation remettre NXc Nyc hc
    int NY = 128;
    ptype h = twoPI/(float)NX;*/

	if(iX >= NXs || iY >= NYs)
		return;
	
	int In = iY*NXs + iX;	
	
	//position
	ptype x = iX*hs;
	ptype y = iY*hs;
	
	ptype x2, y2;
	
	device_diffeo_interpolate(ChiX, ChiY, x, y, &x2, &y2, NXc, NYc, hc);
	
	
	fs[In] = device_hermite_interpolate(H, x2, y2, NXh, NYh, hh);
	
}


/*******************************************************************
*						 										   *
*******************************************************************/

////////////////////////////////////////////////////////////////////////
__global__ void kernel_sample_on_coarse_grid(cuPtype *AcOut, cuPtype *AfOut, int NXc, int NYc, ptype hc, int NXf, int NYf, ptype hf)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NXc || iY >= NYc)
		return;
	
	int In = iY*NXc + iX;	
		
		////////////////////////////////////////////////////
		//same sampling grid
		AcOut[In].x = AfOut[In].x;
		AcOut[In].y = AfOut[In].y;
		return;
	
}


__global__ void kernel_normalize(cuPtype *F, int NX, int NY)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NX || iY >= NY)
		return;
	
	int In = iY*NX + iX;	
	
	ptype N = NX*NY;
	
	F[In].x /= (ptype)N;
	F[In].y /= (ptype)N;
}


/*******************************************************************
*						 Initial condition						   *
*******************************************************************/

__device__ ptype device_initial_W(ptype x, ptype y)
{
// "4_nodes"		"quadropole"		"three_vortices"		"single_shear_layer"		"two_votices"

	if(PROBLEM_CODE == "4_nodes")
	{
		x = x - (x>0)*((int)(x/twoPI))*twoPI - (x<0)*((int)(x/twoPI)-1)*twoPI;
		y = y - (y>0)*((int)(y/twoPI))*twoPI - (y<0)*((int)(y/twoPI)-1)*twoPI;
		
		return cos(x) + cos(y) + 0.6*cos(2*x) + 0.2*cos(3*x);
	}
	else if(PROBLEM_CODE == "quadropole")
	{
		ptype ret = 0;
		for(int iy = -2; iy <= 2; iy++)
			for(int ix = -2; ix <= 2; ix++)
			{
				ptype dx = x - PI/2 + ix * 2*PI; 
				ptype dy = y - PI/2 + iy * 2*PI;
				ptype A = 0.6258473;
				ptype s = 0.5;
				ptype B = A/(s*s*s*s) * (dx * dy) * (dx*dx + dy*dy - 6*s*s);
				ptype D = (dx*dx + dy*dy)/(2*s*s);
				ret += B * exp(-D);
			}
			return ret;
	}
	else if(PROBLEM_CODE == "two_votices")
	{
		ptype ret = 0;
		for(int iy = -1; iy <= 1; iy++)
			for(int ix = -1; ix <= 1; ix++)
			{
				ret += sin(0.5*(x + twoPI*ix))*sin(0.5*(x + twoPI*ix))*sin(0.5*((y + twoPI*iy) + twoPI*iy))*sin(0.5*((y + twoPI*iy) + twoPI*iy)) * (exp(-(((x + twoPI*ix) - PI)*((x + twoPI*ix) - PI) + ((y + twoPI*iy) - 0.33*twoPI)*((y + twoPI*iy) - 0.33*twoPI))*5) + 
											exp(-(((x + twoPI*ix) - PI)*((x + twoPI*ix) - PI) + ((y + twoPI*iy) - 0.67*twoPI)*((y + twoPI*iy) - 0.67*twoPI))*5));		 //two votices of same size
			}
		return ret;
	}
	else if(PROBLEM_CODE == "three_vortices")
	{
		//three vortices
		ptype ret = 0;
		ptype LX = PI/2;
		ptype LY = PI/(2.0*sqrt(2.0));
		
		for(int iy = -1; iy <= 1; iy++)
			for(int ix = -1; ix <= 1; ix++)
			{
				ret += sin(0.5*(x + twoPI*ix))*sin(0.5*(x + twoPI*ix))*sin(0.5*((y + twoPI*iy) + twoPI*iy))*sin(0.5*((y + twoPI*iy) + twoPI*iy)) * 
							(
							+	exp(-(((x + twoPI*ix) - PI - LX)*((x + twoPI*ix) - PI - LX) + ((y + twoPI*iy) - PI)*((y + twoPI*iy) - PI))*5) 
							+	exp(-(((x + twoPI*ix) - PI + LX)*((x + twoPI*ix) - PI + LX) + ((y + twoPI*iy) - PI)*((y + twoPI*iy) - PI))*5) 
							-	exp(-(((x + twoPI*ix) - PI + LX)*((x + twoPI*ix) - PI + LX) + ((y + twoPI*iy) - PI - LY)*((y + twoPI*iy) - PI - LY))*5) 
							);		 //two votices of same size
			}
		return ret;
	}
	else if(PROBLEM_CODE == "single_shear_layer")
	{
		//single shear layer
		ptype delta = 50;
		ptype delta2 = 0.01;
		ptype ret = 0;
		for(int iy = -1; iy <= 1; iy++)
			for(int iy = -1; iy <= 1; iy++)
				{
					ret +=    (1 + delta2 * cos(2*x))  *    exp( - delta * (y - PI) * (y - PI) ); 
				}
		ret /= 9;
		return ret;
	}
	else if(PROBLEM_CODE == "turbulence_gaussienne")
	{	
		x = fmod(x, twoPI);
		x = (x < 0)*(twoPI+x) + (x > 0)*(x);
		y = fmod(y, twoPI);
		y = (y < 0)*(twoPI+y) + (y > 0)*(y);
		int NB_gaus = 8;		//NB_gaus = 6;sigma = 0.24;
		ptype sigma = 0.2;
		ptype ret = 0;
		for(int mu_x = 0; mu_x < NB_gaus; mu_x++){
			for(int mu_y = 0; mu_y < NB_gaus; mu_y++){
				ret += 1/(twoPI*sigma*sigma)*exp(-((x-mu_x*twoPI/(NB_gaus-1))*(x-mu_x*twoPI/(NB_gaus-1))/(2*sigma*sigma)+(y-mu_y*twoPI/(NB_gaus-1))*(y-mu_y*twoPI/(NB_gaus-1))/(2*sigma*sigma))); 
			}
		}
		for(int mu_x = 0; mu_x < NB_gaus-1; mu_x++){
			for(int mu_y = 0; mu_y < NB_gaus-1; mu_y++){
				hiprandState_t state_x;
				hiprand_init((mu_x+1)*mu_y*mu_y, 0, 0, &state_x);
				ptype RAND_gaus_x = ((ptype)(hiprand(&state_x)%1000)-500)/100000;
				hiprandState_t state_y;
				hiprand_init((mu_y+1)*mu_x*mu_x, 0, 0, &state_y);
				ptype RAND_gaus_y = ((ptype)(hiprand(&state_y)%1000)-500)/100000;
				ret -= 1/(twoPI*sigma*sigma)*exp(-((x-(2*mu_x+1)*twoPI/(2*(NB_gaus-1))+RAND_gaus_x)*(x-(2*mu_x+1)*twoPI/(2*(NB_gaus-1))+RAND_gaus_x)/(2*sigma*sigma)+(y-(2*mu_y+1)*twoPI/(2*(NB_gaus-1))+RAND_gaus_y)*(y-(2*mu_y+1)*twoPI/(2*(NB_gaus-1))+RAND_gaus_y)/(2*sigma*sigma))); 
			}
		}
		//hiprandState_t state;
		//hiprand_init(floor(y * 16384) * 16384 + floor(x * 16384), 0, 0, &state);
		//ret *= 1+((ptype)(hiprand(&state)%1000)-500)/100000;
		return ret - 0.008857380480028442;
	}
	else	//default case goes to stationary
	{
		x = x - (x>0)*((int)(x/twoPI))*twoPI - (x<0)*((int)(x/twoPI)-1)*twoPI;
		y = y - (y>0)*((int)(y/twoPI))*twoPI - (y<0)*((int)(y/twoPI)-1)*twoPI;
		
		return cos(x)*cos(y);
	}

}



__device__ ptype device_initial_W_discret(ptype x, ptype y, ptype *W_initial, int NX, int NY){
	
	int In; 
	
	
	x = fmod(x, twoPI);
	x = (x < 0)*(twoPI+x) + (x > 0)*(x);
	y = fmod(y, twoPI);
	y = (y < 0)*(twoPI+y) + (y > 0)*(y);
	
	In = floor(y/twoPI * NY) * NX + floor(x/twoPI * NX); 
	//In = floor(x * NY) * NX + floor(y * NX); 
	
	return W_initial[In];

}








/*******************************************************************
*							   Zoom								   *
*******************************************************************/


void Zoom(TCudaGrid2D *Grid_coarse, TCudaGrid2D *Grid_fine, ptype *Dev_ChiX_stack, ptype *Dev_ChiY_stack, ptype *Host_ChiX_stack_RAM_0, ptype *Host_ChiY_stack_RAM_0, ptype *Host_ChiX_stack_RAM_1, ptype *Host_ChiY_stack_RAM_1, ptype *Host_ChiX_stack_RAM_2, ptype *Host_ChiY_stack_RAM_2, ptype *Host_ChiX_stack_RAM_3, ptype *Host_ChiY_stack_RAM_3, ptype *Dev_ChiX, ptype *Dev_ChiY, int stack_length, int map_stack_length, int stack_length_RAM, int stack_length_Nb_array_RAM, int mem_RAM, ptype *W_real, hipfftHandle cufftPlan_fine, ptype *W_initial, cuPtype *Dev_Complex_fine, string simulationName, ptype L)
{
	ptype *ws;
	ws = new ptype[Grid_fine->N];
	int save_ctr = 0;
	
	ptype xCenter = 0.54; 
	ptype yCenter = 0.51; 
	ptype width = 0.5;
	
	ptype xMin = xCenter - width/2;
	ptype xMax = xMin + width;
	ptype yMin = yCenter - width/2;
	ptype yMax = yMin + width;
	
	std::ostringstream ss;
	ss<<save_ctr;
	
	
	//save zooming effects
	for(int zoom_ctr = 0; zoom_ctr<10; zoom_ctr++){
		
		width *=  0.5;//0.99
		xMin = xCenter - width/2;
		xMax = xMin + width;
		yMin = yCenter - width/2;
		yMax = yMin + width;
		
		
		//kernel_apply_map_stack_to_W_custom<<<Gsf->blocksPerGrid, Gsf->threadsPerBlock>>>(devChiX_stack, devChiY_stack, devChiX, devChiY, devWs, stack_map_passed, Gc->NX, Gc->NY, Gc->h, Gsf->NX, Gsf->NY, Gsf->h, xMin*L, xMax*L, yMin*L, yMax*L, W_initial);	
		kernel_apply_map_stack_to_W_custom_part_All(Grid_coarse, Grid_fine, Dev_ChiX_stack, Dev_ChiY_stack, Dev_ChiX, Dev_ChiY, Host_ChiX_stack_RAM_0, Host_ChiY_stack_RAM_0, Host_ChiX_stack_RAM_1, Host_ChiY_stack_RAM_1, Host_ChiX_stack_RAM_2, Host_ChiY_stack_RAM_2, Host_ChiX_stack_RAM_3, Host_ChiY_stack_RAM_3, W_real, Dev_Complex_fine, stack_length, map_stack_length, stack_length_RAM, stack_length_Nb_array_RAM, mem_RAM, Grid_coarse->NX, Grid_coarse->NY, Grid_coarse->h, Grid_fine->NX, Grid_fine->NY, Grid_fine->h, xMin*L, xMax*L, yMin*L, yMax*L, W_initial);
		
		
		hipMemcpy(ws, W_real, Grid_fine->sizeNReal, hipMemcpyDeviceToHost);
		
		std::ostringstream ss2;
		ss2<<zoom_ctr;
		
		writeAllRealToBinaryFile(Grid_fine->N, ws, simulationName, "zoom_" + ss2.str());
	}
	
}


/*******************************************************************
*							    								   *
*******************************************************************/



















/******************************************************************/
/*******************************************************************
*							   Old								   *
*******************************************************************/
/******************************************************************/












////////////////////////////////////////////////////////////////////////
void test_fft_operations()
{
}

//void recompute_output_files(){}

////////////////////////////////////////////////////////////////////////
ptype compare_map_with_identity(ptype *chiX, ptype *chiY, int NX, int NY, ptype h)
{
return 0;
}

__global__ void kernel_compute_total_grid_Chi(ptype *ChiX_stack, ptype *ChiY_stack, ptype *ChiX, ptype *ChiY, ptype *gradChi, int stack_length, int NXc, int NYc, ptype hc, int NXs, int NYs, ptype hs)
{
}

__global__ void kernel_compute_enstropy_increase_rate_factors(ptype *w, ptype *phi, ptype *div1, ptype *div2, int NXc, int NYc, ptype hc, ptype ep)
{
}

__global__ void kernel_compute_enstropy_increase_rate_factors(ptype *wHsc, ptype *ChiX, ptype *ChiY, ptype *phi, ptype *div1, ptype *div2, int NXc, int NYc, ptype hc, int NXsc, int NYsc, ptype hsc, ptype ep)
{
}

////////////////////////////////////////////////////////////////////////
__global__ void kernel_advect_using_velocity_function(ptype *ChiX, ptype *ChiY, ptype *ChiDualX, ptype *ChiDualY,  int NXc, int NYc, ptype hc, ptype t, ptype dt, ptype ep)
{
}

__global__ void kernel_advect_using_stream_hermite(ptype *ChiX, ptype *ChiY, ptype *ChiDualX, ptype *ChiDualY, ptype *phi, int NXc, int NYc, ptype hc, ptype t, ptype dt, ptype ep)
{
}

////////////////////////////////////////////////////////////////////////
__global__ void kernel_apply_map_to_W(ptype *ChiX, ptype *ChiY, ptype *ws, int NXc, int NYc, ptype hc, int NXs, int NYs, ptype hs)
{
}

__global__ void kernel_compare_map_stack_with_identity(ptype *ChiX_stack, ptype *ChiY_stack, ptype *ChiX, ptype *ChiY, ptype *error, int stack_length, int NXc, int NYc, ptype hc, int NXs, int NYs, ptype hs)
{
}




























