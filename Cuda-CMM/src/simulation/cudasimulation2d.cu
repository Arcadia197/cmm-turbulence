#include "hip/hip_runtime.h"
#include "cudasimulation2d.h"

#include "stdio.h"

#include "../numerical/cmm-hermite.h"
#include "../numerical/cmm-timestep.h"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// parallel reduce
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>

__constant__ double d_L1[4], d_L12[4], d_c1[12], d_cx[12], d_cy[12], d_cxy[12];

////////////////////////////////////////////////////////////////////////
__global__ void kernel_init_diffeo(double *ChiX, double *ChiY, int NX, int NY, double h)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NX || iY >= NY)
		return;
		
	long int N = NX*NY;
	
	int In = iY*NX + iX;	
	
	ChiX[In] = iX*h;
	ChiY[In] = iY*h;
	
	// x dx = y dy = 1
	ChiX[1*N+In] = ChiY[2*N+In] = 1;
	
	// x dy = y dx = dxdy = 0
	ChiX[2*N+In] = ChiY[1*N+In] = ChiX[3*N+In] = ChiY[3*N+In] = 0;
}


__global__ void k_sample(double *ChiX, double *ChiY, double *ChiX_s, double *ChiY_s, int NXc, int NYc, double hc, int NXs, int NYs, double hs)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NXs || iY >= NYs)
		return;
	
	int In = iY*NXs + iX;	
	
	//position
	double x = iX*hs;
	double y = iY*hs;
	
	device_diffeo_interpolate_2D(ChiX, ChiY, x, y, &x, &y, NXc, NYc, hc);
	
	ChiX_s[In] = x;
	ChiY_s[In] = y;
}


// function to get difference to 1 for thrust parallel reduction
struct absto1
{
    __host__ __device__
        double operator()(const double &x) const {
            return fabs(1-x);
        }
};
double incompressibility_check(double *ChiX, double *ChiY, double *gradChi, TCudaGrid2D Grid_fine, TCudaGrid2D Grid_coarse) {
	// compute determinant of gradient and save in gradchi
	kernel_incompressibility_check<<<Grid_fine.blocksPerGrid, Grid_fine.threadsPerBlock>>>(ChiX, ChiY, gradChi, Grid_coarse.NX, Grid_coarse.NY, Grid_coarse.h, Grid_fine.NX, Grid_fine.NY, Grid_fine.h);  // time cost		A optimiser

	// compute maximum using thrust parallel reduction
	thrust::device_ptr<double> gradChi_ptr = thrust::device_pointer_cast(gradChi);
	return thrust::transform_reduce(gradChi_ptr, gradChi_ptr + Grid_fine.N, absto1(), 0.0, thrust::maximum<double>());
}


__global__ void kernel_incompressibility_check(double *ChiX, double *ChiY, double *gradChi, int NXc, int NYc, double hc, int NXs, int NYs, double hs)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NXs || iY >= NYs)
		return;
	
	int In = iY*NXs + iX;
	
	//position shifted by half a point to compute at off-grid
	double x = iX*hs + 0.5*hs;
	double y = iY*hs + 0.5*hs;
	gradChi[In] = device_diffeo_grad_2D(ChiX, ChiY, x, y, NXc, NYc, hc);
}


// advect stream where footpoints are just neighbouring points
void advect_using_stream_hermite_grid(SettingsCMM SettingsMain, TCudaGrid2D Grid_map, TCudaGrid2D Grid_psi, double *ChiX, double *ChiY, double *Chi_new_X, double *Chi_new_Y, double *psi, double *t, double *dt, int loop_ctr) {
	// compute lagrange coefficients from dt vector for timesteps n+dt and n+dt/2, this makes them dynamic
	double h_L1[4], h_L12[4];  // constant memory for lagrange coefficient to be computed only once
	int loop_ctr_l = loop_ctr + SettingsMain.getLagrangeOrder()-1;  // dt and t are shifted because of initial previous steps
	for (int i_p = 0; i_p < SettingsMain.getLagrangeOrder(); ++i_p) {
		h_L1[i_p] = get_L_coefficient(t, t[loop_ctr_l+1], loop_ctr_l, i_p, SettingsMain.getLagrangeOrder());
		h_L12[i_p] = get_L_coefficient(t, t[loop_ctr_l] + dt[loop_ctr_l+1]/2.0, loop_ctr_l, i_p, SettingsMain.getLagrangeOrder());
	}

	// copy to constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(d_L1), h_L1, sizeof(double)*4); hipMemcpyToSymbol(HIP_SYMBOL(d_L12), h_L12, sizeof(double)*4);

	// first of all: compute footpoints at gridpoints, here we could speedup the first sampling of u by directly using the values, as we start at exact grid point locations
	k_compute_footpoints<<<Grid_map.blocksPerGrid, Grid_map.threadsPerBlock>>>(ChiX, ChiY, Chi_new_X, Chi_new_Y, psi,
			Grid_map.NX, Grid_map.NY, Grid_map.h, Grid_psi, t[loop_ctr_l+1], dt[loop_ctr_l+1],
			SettingsMain.getTimeIntegrationNum(), SettingsMain.getLagrangeOrder());

	// update map, x and y can be done seperately
	int shared_size = (18+2*SettingsMain.getMapUpdateOrderNum())*(18+2*SettingsMain.getMapUpdateOrderNum());  // how many points do we want to load?
	k_map_update<<<Grid_map.blocksPerGrid, Grid_map.threadsPerBlock, shared_size*sizeof(double)>>>(ChiX, Chi_new_X, Grid_map.NX, Grid_map.NY, Grid_map.h, SettingsMain.getMapUpdateOrderNum()+1, 0);
	k_map_update<<<Grid_map.blocksPerGrid, Grid_map.threadsPerBlock, shared_size*sizeof(double)>>>(ChiY, Chi_new_Y, Grid_map.NX, Grid_map.NY, Grid_map.h, SettingsMain.getMapUpdateOrderNum()+1, 1);
}


// compute footpoints at exact grid locations
__global__ void k_compute_footpoints(double *ChiX, double *ChiY, double *Chi_new_X, double *Chi_new_Y, double *psi, int NXc, int NYc, double hc, TCudaGrid2D Grid_psi, double t, double dt, int time_integration_num, int l_order) {
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
    /*int NX = 512; //Dans l'interpolation remettre NXc Nyc hc
    int NY = 512;
    double h = twoPI/(float)NX;*/

    if(iX >= NXc || iY >= NYc)
		return;

    int In = iY*NXc + iX;

    double x_ep[2], x_f[2];

	x_ep[0] = iX*hc; x_ep[1] = iY*hc;

	// time integration - note, we go backwards in time!
	switch (time_integration_num) {
		case 10: { euler_exp         (psi, d_L1,        x_ep, x_f, Grid_psi, dt, l_order); break; }
		// ABTwo
		case 20: { adam_bashford_2_pc(psi,              x_ep, x_f, Grid_psi, dt         ); break; }
		// ABTwo
		case 21: { RK2_heun          (psi, d_L1,        x_ep, x_f, Grid_psi, dt, l_order); break; }
		// RKThree
		case 30: { RK3_classical     (psi, d_L1, d_L12, x_ep, x_f, Grid_psi, dt, l_order); break; }
		// RKFour
		case 40: { RK4_classical     (psi, d_L1, d_L12, x_ep, x_f, Grid_psi, dt, l_order); break; }
		// custom RKThree
		case 31: { RK3_optimized     (psi, d_L1,        x_ep, x_f, Grid_psi, dt, l_order); break; }
		// custom RKFour case IV
		case 41: { RK4_optimized     (psi, d_L1, d_L12, x_ep, x_f, Grid_psi, dt, l_order); break; }
		// zero on default
		default: { x_f[0] = x_f[1] = 0; }
	}

	// apply map deformation
	device_diffeo_interpolate_2D(ChiX, ChiY, x_f[0], x_f[1], &x_f[0], &x_f[1], NXc, NYc, hc);

	// transcribe
	Chi_new_X[In] = x_f[0];	Chi_new_Y[In] = x_f[1];
}


// update map, this should hopefully use some funky cuda techniques
// do template for map update orders to help the compiler
__global__ void k_map_update(double *Chi, double *Chi_new, int NXc, int NYc, double hc, int map_ord_1, int warp_direc) {
	// shared memory for loading the data - block widened by map_ord_1
    extern __shared__ double sdata[];

	int thread_id_xy = threadIdx.x + threadIdx.y * blockDim.x;  // unique ID of thread in block

	// get positions in whole grid
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);

	bool debug = false;
//	if ((blockIdx.x == 0) && (blockIdx.y == 0)) { debug = true; }

    if(iX >= NXc || iY >= NYc) return;  // safety precaution

    // index in arrays
    int In = iY*NXc + iX;
    long int N = NXc*NYc;

	if (debug) {
		printf("Id : %d \t block x : %d \t block y : %d \t In : %d\n", thread_id_xy, iX, iY, In);
	}

    int ring_num = (16 + 2 * map_ord_1)* (16 + 2* map_ord_1) - 256;  // how many elements do we have in the ring
    int change_num = (16 + 2 * map_ord_1) * map_ord_1;  // how many elements at top or bottom

    // load values in shared memory, this makes it efficient as we have to access less storage in total
    // central block first, each thread loads memory at its location, offset to fit
    int sdata_pos = change_num + map_ord_1 + thread_id_xy + 2*map_ord_1*(thread_id_xy / 16);
    sdata[sdata_pos] = Chi_new[In];

	if (debug) {
		printf("Id : %d \t store 1 : %d \t load 1 : %d\n", thread_id_xy, sdata_pos, In);
		__syncthreads();  // only for debugging
	}

	if (thread_id_xy < ring_num) {
		// load in ring values computed without ifs so that all important threads have something to do
		int load_id, store_id;

		// where should we store in shared memory?
		store_id = thread_id_xy
				 + ((thread_id_xy >= change_num)*(thread_id_xy < ring_num-change_num) * ((thread_id_xy + map_ord_1 - change_num)/(2*map_ord_1))*16)
				 + (thread_id_xy >= ring_num-change_num)*256;

		// due to periodic boundaries, we have to check for x and y values to perform waprs
		int load_differ_x = - map_ord_1 + (thread_id_xy < change_num)*(thread_id_xy%(16+map_ord_1*2))
				+ (thread_id_xy < ring_num-change_num)*(thread_id_xy >= change_num) * (thread_id_xy%(2*map_ord_1) + ((thread_id_xy/map_ord_1)%2)*16)
				+ (thread_id_xy >= ring_num-change_num)*((thread_id_xy-ring_num+change_num)%(16+map_ord_1*2));
		int load_differ_y = - map_ord_1 + (thread_id_xy < change_num)*thread_id_xy/(16+map_ord_1*2)
				+ (thread_id_xy < ring_num-change_num)*(thread_id_xy >= change_num) * (map_ord_1+(thread_id_xy - change_num)/(2*map_ord_1))
				+ (thread_id_xy >= ring_num-change_num)*(16+map_ord_1+(thread_id_xy-ring_num+change_num)/(16+map_ord_1*2));

		if (debug) {
			printf("Id : %d \t load_x : %d \t load y : %d\n", thread_id_xy, load_differ_x, load_differ_y);
		}

		// compute
		int load_id_x = blockDim.x * blockIdx.x + load_differ_x;
		int load_id_y = blockDim.y * blockIdx.y + load_differ_y;

		int warp_x = (load_id_x/NXc - (load_id_x < 0));  // needed for warping the values back
		int warp_y = (load_id_y/NYc - (load_id_y < 0));  // needed for warping the values back
		load_id_x -= warp_x*NXc;  // warp
		load_id_y -= warp_y*NYc;  // warp

		load_id = load_id_y * NXc + load_id_x;

		sdata[store_id] = Chi_new[load_id] + twoPI*((1 - warp_direc)*warp_x + warp_direc*warp_y);

		if (debug) {
			printf("Id : %d \t store 2 : %d \t load 2 : %d\n", thread_id_xy, store_id, load_id);
		}
	}


    // synch threads needed, as different warps compute different things
    __syncthreads();

    // now, lets modify the values
    double chi_new[4];

	// chi values - exact center point
    int shift_y = change_num/map_ord_1;  // construct stencils easily
    chi_new[0] = sdata[sdata_pos];

    switch (map_ord_1) {
    	// second order
		case 1: {
			// chi grad x - central differences with stencil +NE +SE -SW -NW
			chi_new[1] = (sdata[sdata_pos + 1] - sdata[sdata_pos - 1])
					   / (2.0 * hc);

			// chi grad y - central differences with stencil +NE -SE -SW +NW
		    chi_new[2] = (sdata[sdata_pos + shift_y] - sdata[sdata_pos - shift_y])
		    		   / (2.0 * hc);

			// chi grad x y - cross central differences with stencil +NE -SE +SW -NW
		    chi_new[3] = (sdata[sdata_pos + shift_y + 1] - sdata[sdata_pos - shift_y + 1] + sdata[sdata_pos - shift_y - 1]  - sdata[sdata_pos + shift_y - 1])
		    		   / (4.0 * hc * hc);
			break;
		}
		// fourth order
		case 2: {
			// chi grad x - central differences with stencil +NE +SE -SW -NW
			chi_new[1] = (8*(sdata[sdata_pos + 1] - sdata[sdata_pos - 1])
					   -    (sdata[sdata_pos + 2] - sdata[sdata_pos - 2]))
					   / (12.0 * hc);

			// chi grad y - central differences with stencil +NE -SE -SW +NW
		    chi_new[2] = (8*(sdata[sdata_pos +   shift_y] - sdata[sdata_pos -   shift_y])
		    		   -    (sdata[sdata_pos + 2*shift_y] - sdata[sdata_pos - 2*shift_y]))
					   / (12.0 * hc);

			// chi grad x y - cross central differences with stencil +NE -SE +SW -NW
		    chi_new[3] =  (8*(sdata[sdata_pos +   shift_y + 1] - sdata[sdata_pos -   shift_y + 1] + sdata[sdata_pos -   shift_y - 1]  - sdata[sdata_pos +   shift_y - 1])
		    		   -     (sdata[sdata_pos + 2*shift_y + 2] - sdata[sdata_pos - 2*shift_y + 2] + sdata[sdata_pos - 2*shift_y - 2]  - sdata[sdata_pos + 2*shift_y - 2]))
		    	       / (24.0 * hc * hc);
			break;
		}
		// sixth order
		case 3: {
			// chi grad x - central differences with stencil +NE +SE -SW -NW
			chi_new[1] = (45*(sdata[sdata_pos + 1] - sdata[sdata_pos - 1])
					   -   9*(sdata[sdata_pos + 2] - sdata[sdata_pos - 2])
					   +     (sdata[sdata_pos + 3] - sdata[sdata_pos - 3]))
					   / (60.0 * hc);

			// chi grad y - central differences with stencil +NE -SE -SW +NW
		    chi_new[2] = (45*(sdata[sdata_pos +   shift_y] - sdata[sdata_pos -   shift_y])
		    		   -   9*(sdata[sdata_pos + 2*shift_y] - sdata[sdata_pos - 2*shift_y])
					   +     (sdata[sdata_pos + 3*shift_y] - sdata[sdata_pos - 3*shift_y]))
					   / (60.0 * hc);

			// chi grad x y - cross central differences with stencil +NE -SE +SW -NW
		    chi_new[3] =  (45*(sdata[sdata_pos +   shift_y + 1] - sdata[sdata_pos -   shift_y + 1] + sdata[sdata_pos -   shift_y - 1]  - sdata[sdata_pos +   shift_y - 1])
		    		   -   18*(sdata[sdata_pos + 2*shift_y + 2] - sdata[sdata_pos - 2*shift_y + 2] + sdata[sdata_pos - 2*shift_y - 2]  - sdata[sdata_pos + 2*shift_y - 2])
		    		   +      (sdata[sdata_pos + 3*shift_y + 3] - sdata[sdata_pos - 3*shift_y + 3] + sdata[sdata_pos - 3*shift_y - 3]  - sdata[sdata_pos + 3*shift_y - 3]))
		    	       / (120.0 * hc * hc);
			break;
		}
	}

    // transcribe final results
	Chi[    In] = chi_new[0];
	Chi[1*N+In] = chi_new[1];
	Chi[2*N+In] = chi_new[2];
	Chi[3*N+In] = chi_new[3];
}



// wrapper function for map advection
void advect_using_stream_hermite(SettingsCMM SettingsMain, TCudaGrid2D Grid_map, TCudaGrid2D Grid_psi, double *ChiX, double *ChiY, double *Chi_new_X, double *Chi_new_Y, double *psi, double *t, double *dt, int loop_ctr) {
	// compute lagrange coefficients from dt vector for timesteps n+dt and n+dt/2, this makes them dynamic
	double h_L1[4], h_L12[4];  // constant memory for lagrange coefficient to be computed only once
	int loop_ctr_l = loop_ctr + SettingsMain.getLagrangeOrder()-1;  // dt and t are shifted because of initial previous steps
	for (int i_p = 0; i_p < SettingsMain.getLagrangeOrder(); ++i_p) {
		h_L1[i_p] = get_L_coefficient(t, t[loop_ctr_l+1], loop_ctr_l, i_p, SettingsMain.getLagrangeOrder());
		h_L12[i_p] = get_L_coefficient(t, t[loop_ctr_l] + dt[loop_ctr_l+1]/2.0, loop_ctr_l, i_p, SettingsMain.getLagrangeOrder());
	}

	double h_c[3];  // constant memory for map update coefficient to be computed only once
	switch (SettingsMain.getMapUpdateOrderNum()) {
		case 2: { h_c[0] = +3.0/8.0; h_c[1] = -3.0/20.0; h_c[2] = +1.0/40.0; break; }  // 6th order interpolation
		case 1: { h_c[0] = +1.0/3.0; h_c[1] = -1.0/12.0; break; }  // 4th order interpolation
		case 0: { h_c[0] = +1.0/4.0; break; }  // 2th order interpolation
	}

	double h_c1[12], h_cx[12], h_cy[12], h_cxy[12];  // compute coefficients for each direction only once
	// already compute final coefficients with appropriate sign
	for (int i_foot = 0; i_foot < 4+4*SettingsMain.getMapUpdateOrderNum(); ++i_foot) {
		h_c1 [i_foot] = h_c[i_foot/4];
		h_cx [i_foot] = h_c[i_foot/4] * (1 - 2*((i_foot/2)%2))     / SettingsMain.getMapEpsilon() / double(i_foot/4 + 1);
		h_cy [i_foot] = h_c[i_foot/4] * (1 - 2*(((i_foot+1)/2)%2)) / SettingsMain.getMapEpsilon() / double(i_foot/4 + 1);
		h_cxy[i_foot] = h_c[i_foot/4] * (1 - 2*(i_foot%2)) / SettingsMain.getMapEpsilon() / SettingsMain.getMapEpsilon() / double(i_foot/4 + 1) / double(i_foot/4 + 1);
	}

	// copy to constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(d_L1), h_L1, sizeof(double)*4); hipMemcpyToSymbol(HIP_SYMBOL(d_L12), h_L12, sizeof(double)*4);
	hipMemcpyToSymbol(HIP_SYMBOL(d_c1), h_c1, sizeof(double)*12); hipMemcpyToSymbol(HIP_SYMBOL(d_cx), h_cx, sizeof(double)*12);
	hipMemcpyToSymbol(HIP_SYMBOL(d_cy), h_cy, sizeof(double)*12); hipMemcpyToSymbol(HIP_SYMBOL(d_cxy), h_cxy, sizeof(double)*12);

	// now launch the kernel
	kernel_advect_using_stream_hermite<<<Grid_map.blocksPerGrid, Grid_map.threadsPerBlock>>>(ChiX, ChiY, Chi_new_X, Chi_new_Y,
			psi, Grid_map, Grid_psi, t[loop_ctr_l+1], dt[loop_ctr_l+1],
			SettingsMain.getMapEpsilon(), SettingsMain.getTimeIntegrationNum(),
			SettingsMain.getMapUpdateOrderNum(), SettingsMain.getLagrangeOrder());
}


/*
 * Main advection function of the flow map using the stream function
 * Loop over footpoints to apply GALS
 * For each foot point: advect using the stream function and time stepping scheme
 * At the end: combine results of all footpoints using specific map update scheme
 */
__global__ void kernel_advect_using_stream_hermite(double *ChiX, double *ChiY, double *Chi_new_X, double *Chi_new_Y, double *psi,
		TCudaGrid2D Grid_map, TCudaGrid2D Grid_psi, double t, double dt, double ep, int time_integration_num, int map_update_order_num, int l_order)
{
	bool debug = false;
//	if ((blockIdx.x == 0) && (blockIdx.y == 0)) { debug = true; }

	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);

    if(iX >= Grid_map.NX || iY >= Grid_map.NY)
		return;

	int In = iY*Grid_map.NX + iX;

	//running through neighbours
	double x_ep[2], x_f[2];

	// initialize new intermediate values as zeros, helpful to not write to array every point
	double Chi_full_x[4] = {0, 0, 0, 0};
	double Chi_full_y[4] = {0, 0, 0, 0};

	// repeat for all footpoints, 4 for 2th order, 8 for 4th order and 12 for 6th order
	for (int k_foot = 0; k_foot< map_update_order_num*4 + 4; k_foot++) {
		int i_foot_now = (k_foot/4);  // used for getting
		double i_dist_now = i_foot_now+1;

		// get position of footpoint, NE, SE, SW, NW
		// for higher orders repeat cross shape stencil with more points
		x_ep[0] = Grid_map.bounds[0] + iX*Grid_map.hx + i_dist_now * ep*(1 - 2*((k_foot/2)%2));
		x_ep[1] = Grid_map.bounds[2] + iY*Grid_map.hy + i_dist_now * ep*(1 - 2*(((k_foot+1)/2)%2));

		// time integration - note, we go backwards in time!
		switch (time_integration_num) {
			case 10: { euler_exp         (psi, d_L1,        x_ep, x_f, Grid_psi, dt, l_order); break; }
			// ABTwo
			case 20: { adam_bashford_2_pc(psi,              x_ep, x_f, Grid_psi, dt         ); break; }
			// ABTwo
			case 21: { RK2_heun          (psi, d_L1,        x_ep, x_f, Grid_psi, dt, l_order); break; }
			// RKThree
			case 30: { RK3_classical     (psi, d_L1, d_L12, x_ep, x_f, Grid_psi, dt, l_order); break; }
			// RKFour
			case 40: { RK4_classical     (psi, d_L1, d_L12, x_ep, x_f, Grid_psi, dt, l_order); break; }
			// custom RKThree
			case 31: { RK3_optimized     (psi, d_L1,        x_ep, x_f, Grid_psi, dt, l_order); break; }
			// custom RKFour case IV
			case 41: { RK4_optimized     (psi, d_L1, d_L12, x_ep, x_f, Grid_psi, dt, l_order); break; }
			// zero on default
			default: { x_f[0] = x_f[1] = 0; }
		}

		// apply map deformation
		device_diffeo_interpolate_2D(ChiX, ChiY, x_f[0], x_f[1], &x_f[0], &x_f[1], Grid_map.NX, Grid_map.NY, Grid_map.h);

		if (debug) {
			printf("Id : %d \t ifoot : %d \t fpoint : %f \n", In, k_foot, x_f[0]);
		}

		// directly apply map update
		// chi values - central average with stencil +NE +SE +SW +NW
		Chi_full_x[0] += x_f[0] * d_c1 [k_foot]; Chi_full_y[0] += x_f[1] * d_c1 [k_foot];

		// chi grad x - central differences with stencil +NE +SE -SW -NW
		Chi_full_x[1] += x_f[0] * d_cx [k_foot]; Chi_full_y[1] += x_f[1] * d_cx [k_foot];

		// chi grad y - central differences with stencil +NE -SE -SW +NW
		Chi_full_x[2] += x_f[0] * d_cy [k_foot]; Chi_full_y[2] += x_f[1] * d_cy [k_foot];

		// chi grad x y - cross central differences with stencil +NE -SE +SW -NW
		Chi_full_x[3] += x_f[0] * d_cxy[k_foot]; Chi_full_y[3] += x_f[1] * d_cxy[k_foot];
	}

	if (debug) {
		printf("Id : %d \t Val : %f \t x : %f \t y : %f \t xy : %f \n", In, Chi_full_x[0], Chi_full_x[1], Chi_full_x[2], Chi_full_x[3]);
	}

	// can't use Chi because we still use it for diffeo_interpolate
	Chi_new_X[             In] = Chi_full_x[0];	Chi_new_Y[             In] = Chi_full_y[0];
	Chi_new_X[1*Grid_map.N+In] = Chi_full_x[1];	Chi_new_Y[1*Grid_map.N+In] = Chi_full_y[1];
	Chi_new_X[2*Grid_map.N+In] = Chi_full_x[2];	Chi_new_Y[2*Grid_map.N+In] = Chi_full_y[2];
	Chi_new_X[3*Grid_map.N+In] = Chi_full_x[3];	Chi_new_Y[3*Grid_map.N+In] = Chi_full_y[3];
}


/*******************************************************************
*						 Apply remapping						   *
*******************************************************************/


void apply_map_stack_to_W_part_All(TCudaGrid2D Grid_fine, MapStack Map_Stack, double *ChiX, double *ChiY,
		double *W_real, double *Dev_Temp, double *bounds, double *W_initial, int simulation_num)
{
	// for normal map stack, bounds has the domain boundaries applied
	kernel_apply_map_stack_to_W_custom_part_1<<<Grid_fine.blocksPerGrid, Grid_fine.threadsPerBlock>>>(ChiX, ChiY, Dev_Temp, Map_Stack.Grid->NX, Map_Stack.Grid->NY, Map_Stack.Grid->h, Grid_fine.NX, Grid_fine.NY, Grid_fine.h, bounds[0], bounds[1], bounds[2], bounds[3]);

	// loop over all maps in map stack, where all maps are on host system
	// this could be parallelized
	for (int i_map = Map_Stack.map_stack_ctr-1; i_map >= 0; i_map--) {
		Map_Stack.copy_map_to_device(i_map);
		kernel_apply_map_stack_to_W_part_2<<<Grid_fine.blocksPerGrid, Grid_fine.threadsPerBlock>>>(Map_Stack.Dev_ChiX_stack, Map_Stack.Dev_ChiY_stack, Dev_Temp, Map_Stack.Grid->NX, Map_Stack.Grid->NY, Map_Stack.Grid->h, Grid_fine.NX, Grid_fine.NY);
	}

	// initial condition
	kernel_apply_map_stack_to_W_part_3<<<Grid_fine.blocksPerGrid, Grid_fine.threadsPerBlock>>>(W_real, Dev_Temp, Grid_fine.NX, Grid_fine.NY, Grid_fine.h, W_initial, simulation_num);
}


__global__ void kernel_apply_map_stack_to_W_custom_part_1(double *ChiX, double *ChiY, double *x_y, int NXc, int NYc, double hc, int NXs, int NYs, double hs, double xl, double xr, double yl, double yr)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NXs || iY >= NYs)
		return;
	
	int In = iY*NXs + iX;
	
	double htemp = (xr - xl)/NXs;

	//position
	double x = xl + iX*htemp;
	double y = yl + iY*htemp;
	
	device_diffeo_interpolate_2D(ChiX, ChiY, x, y, &x, &y, NXc, NYc, hc);
	
	// save in two points in array
	x_y[2*In  ] = x;
	x_y[2*In+1] = y;

}

__global__ void kernel_apply_map_stack_to_W_part_2(double *ChiX_stack, double *ChiY_stack, double *x_y, int NXc, int NYc, double hc, int NXs, int NYs)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NXs || iY >= NYs)
		return;
	
	int In = iY*NXs + iX;
	
	//for(int k = stack_length - 1; k >= 0; k--)
	device_diffeo_interpolate_2D(ChiX_stack, ChiY_stack, x_y[2*In], x_y[2*In+1], &x_y[2*In], &x_y[2*In+1], NXc, NYc, hc);
	
}

__global__ void kernel_apply_map_stack_to_W_part_3(double *ws, double *x_y, int NXs, int NYs, double hs, double *W_initial, int simulation_num)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NXs || iY >= NYs)
		return;
	
	int In = iY*NXs + iX;
	
	#ifndef DISCRET
		ws[In] = device_initial_W(x_y[2*In], x_y[2*In+1], simulation_num);
	#endif
	
	#ifdef DISCRET
		ws[In] = device_hermite_interpolate_2D(W_initial, x_y[2*In], x_y[2*In+1], NXs, NYs, hs);
	#endif
	
}


/*******************************************************************
*						 										   *
*******************************************************************/


__global__ void kernel_compare_vorticity_with_initial(double *ChiX_stack, double *ChiY_stack, double *ChiX, double *ChiY, double *error, int stack_length, int NXc, int NYc, double hc, int NXs, int NYs, double hs, int simulation_num)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);
	
	if(iX >= NXs || iY >= NYs)
		return;
	
	int In = iY*NXs + iX;
	long int N = NXc*NYc;	
	
	//position
	double x = iX*hs;
	double y = iY*hs;
	
	device_diffeo_interpolate_2D(ChiX, ChiY, x, y, &x, &y, NXc, NYc, hc);
	for(int k = stack_length - 1; k >= 0; k--)
		device_diffeo_interpolate_2D(&ChiX_stack[k*N*4], &ChiY_stack[k*N*4], x, y, &x, &y, NXc, NYc, hc);
	
	error[In] = fabs(device_initial_W(x, y, simulation_num) - device_initial_W(iX*hs, iY*hs, simulation_num));
}


// apply mollifier
__global__ void kernel_apply_map_and_sample_from_hermite(double *ChiX, double *ChiY, double *fs, double *H, int NXc, int NYc, double hc, int NXs, int NYs, double hs, int NXh, int NYh, double hh, int molly_stencil)
{
	//index
	int iX = (blockDim.x * blockIdx.x + threadIdx.x);
	int iY = (blockDim.y * blockIdx.y + threadIdx.y);

	if(iX >= NXs || iY >= NYs)
		return;

	int In = iY*NXs + iX;

	//position
	double x = iX*hs;
	double y = iY*hs;

	// mollification to act as a lowpass filter
	double x2, y2;

	/*                                      0  1/6  0
	 * mollifier of order 1 using stencil  1/6 1/3 1/6
	 * using 4 neighbouring points			0  1/6  0
	 */
	if (molly_stencil == 4) {
		// compute main points
		device_diffeo_interpolate_2D(ChiX, ChiY, x, y, &x2, &y2, NXc, NYc, hc);
		double moll_add = device_hermite_interpolate_2D(H, x2, y2, NXh, NYh, hh)/3.0;  // other values will be added on here
		for (int i_molly = 0; i_molly < 4; i_molly++) {
			// choose 4 points in between the grid: W, E, S, N
			x2 = x + hs/2.0*((i_molly/2+1)%2) * (-1 + 2*(i_molly%2));  // -1 +1  0  0
			y2 = y + hs/2.0*((i_molly/2  )%2) * (-1 + 2*(i_molly%2));  //  0  0 -1 +1

			device_diffeo_interpolate_2D(ChiX, ChiY, x2, y2, &x2, &y2, NXc, NYc, hc);
			moll_add += device_hermite_interpolate_2D(H, x2, y2, NXh, NYh, hh)/6.0;
		}
		fs[In] = moll_add;
	}
	/*                                     1/16 2/16 1/16
	 * mollifier of order 1 using stencil  2/16 4/16 2/16
	 * using 8 neighbouring points		   1/16 2/16 1/16
	 */
	else if (molly_stencil == 8) {
//		double moll_fac[9] = {1/16, 2/16, 1/16, 2/16, 4/16, 2/16, 1/16, 2/16, 1/16};  // molly factor for all points, chosen randomly
		// compute main points
		double moll_add = 0;  // other values will be added on here
		for (int i_molly = 0; i_molly < 9; i_molly++) {
			// choose 9 points in between the grid: SW, S, SE, W, C, E, NW, N, NE
			x2 = x + hs*(-1 + i_molly%3)/2.0;
			y2 = y + hs*(-1 + i_molly/3)/2.0;

			device_diffeo_interpolate_2D(ChiX, ChiY, x2, y2, &x2, &y2, NXc, NYc, hc);
			moll_add += (1 + (i_molly%3)%2) * (1 + (i_molly/3)%2) * device_hermite_interpolate_2D(H, x2, y2, NXh, NYh, hh)/16.0;
		}
		fs[In] = moll_add;
	}
	// else, assume no mollification
	else {
		double x2, y2;
		device_diffeo_interpolate_2D(ChiX, ChiY, x, y, &x2, &y2, NXc, NYc, hc);
		fs[In] = device_hermite_interpolate_2D(H, x2, y2, NXh, NYh, hh);
	}
}




/*******************************************************************
*						 Initial condition						   *
*******************************************************************/

__device__ double device_initial_W(double x, double y, int simulation_num)
{
// "4_nodes"		"quadropole"		"three_vortices"		"single_shear_layer"		"two_votices"

//	double ret = 0;
	switch (simulation_num) {
		case 0:  // 4_nodes
		{
			x -= floor(x/twoPI)*twoPI;
			y -= floor(y/twoPI)*twoPI;

			return cos(x) + cos(y) + 0.6*cos(2*x) + 0.2*cos(3*x);
			break;
		}
		case 1:  // quadropole
		{
			double ret = 0;
			for(int iy = -2; iy <= 2; iy++)
				for(int ix = -2; ix <= 2; ix++)
				{
					double dx = x - PI/2 + ix * 2*PI;
					double dy = y - PI/2 + iy * 2*PI;
					double A = 0.6258473;
					double s = 0.5;
					double B = A/(s*s*s*s) * (dx * dy) * (dx*dx + dy*dy - 6*s*s);
					double D = (dx*dx + dy*dy)/(2*s*s);
					ret += B * exp(-D);
				}
				return ret;
			break;
		}
		case 2:  // two vortices
		{
			double ret = 0;
			for(int iy = -1; iy <= 1; iy++)
				for(int ix = -1; ix <= 1; ix++)
				{
					ret += sin(0.5*(x + twoPI*ix))*sin(0.5*(x + twoPI*ix))*sin(0.5*((y + twoPI*iy) + twoPI*iy))*sin(0.5*((y + twoPI*iy) + twoPI*iy)) * (exp(-(((x + twoPI*ix) - PI)*((x + twoPI*ix) - PI) + ((y + twoPI*iy) - 0.33*twoPI)*((y + twoPI*iy) - 0.33*twoPI))*5) +
												exp(-(((x + twoPI*ix) - PI)*((x + twoPI*ix) - PI) + ((y + twoPI*iy) - 0.67*twoPI)*((y + twoPI*iy) - 0.67*twoPI))*5));		 //two votices of same size
				}
			return ret;
			break;
		}
		case 3:  // three vortices
		{
			//three vortices
			double ret = 0;
			double LX = PI/2;
			double LY = PI/(2.0*sqrt(2.0));

			for(int iy = -1; iy <= 1; iy++)
				for(int ix = -1; ix <= 1; ix++)
				{
					ret += sin(0.5*(x + twoPI*ix))*sin(0.5*(x + twoPI*ix))*sin(0.5*((y + twoPI*iy) + twoPI*iy))*sin(0.5*((y + twoPI*iy) + twoPI*iy)) *
								(
								+	exp(-(((x + twoPI*ix) - PI - LX)*((x + twoPI*ix) - PI - LX) + ((y + twoPI*iy) - PI)*((y + twoPI*iy) - PI))*5)
								+	exp(-(((x + twoPI*ix) - PI + LX)*((x + twoPI*ix) - PI + LX) + ((y + twoPI*iy) - PI)*((y + twoPI*iy) - PI))*5)
								-	exp(-(((x + twoPI*ix) - PI + LX)*((x + twoPI*ix) - PI + LX) + ((y + twoPI*iy) - PI - LY)*((y + twoPI*iy) - PI - LY))*5)
								);		 //two votices of same size
				}
			return ret;
			break;
		}
		case 4:  // single_shear_layer
		{
			//single shear layer
			double delta = 50;
			double delta2 = 0.01;
			double ret = 0;
			for(int iy = -1; iy <= 1; iy++)
				for(int iy = -1; iy <= 1; iy++)
					{
						ret +=    (1 + delta2 * cos(2*x))  *    exp( - delta * (y - PI) * (y - PI) );
					}
			ret /= 9;
			return ret;
			break;
		}
		case 5:  // turbulence_gaussienne
		{
			x -= floor(x/twoPI)*twoPI;
			y -= floor(y/twoPI)*twoPI;

			int NB_gaus = 8;		//NB_gaus = 6;sigma = 0.24;
			double sigma = 0.2;
			double ret = 0;
			for(int mu_x = 0; mu_x < NB_gaus; mu_x++){
				for(int mu_y = 0; mu_y < NB_gaus; mu_y++){
					ret += 1/(twoPI*sigma*sigma)*exp(-((x-mu_x*twoPI/(NB_gaus-1))*(x-mu_x*twoPI/(NB_gaus-1))/(2*sigma*sigma)+(y-mu_y*twoPI/(NB_gaus-1))*(y-mu_y*twoPI/(NB_gaus-1))/(2*sigma*sigma)));
				}
			}
			for(int mu_x = 0; mu_x < NB_gaus-1; mu_x++){
				for(int mu_y = 0; mu_y < NB_gaus-1; mu_y++){
					hiprandState_t state_x;
					hiprand_init((mu_x+1)*mu_y*mu_y, 0, 0, &state_x);
					double RAND_gaus_x = ((double)(hiprand(&state_x)%1000)-500)/100000;
					hiprandState_t state_y;
					hiprand_init((mu_y+1)*mu_x*mu_x, 0, 0, &state_y);
					double RAND_gaus_y = ((double)(hiprand(&state_y)%1000)-500)/100000;
					ret -= 1/(twoPI*sigma*sigma)*exp(-((x-(2*mu_x+1)*twoPI/(2*(NB_gaus-1))+RAND_gaus_x)*(x-(2*mu_x+1)*twoPI/(2*(NB_gaus-1))+RAND_gaus_x)/(2*sigma*sigma)+(y-(2*mu_y+1)*twoPI/(2*(NB_gaus-1))+RAND_gaus_y)*(y-(2*mu_y+1)*twoPI/(2*(NB_gaus-1))+RAND_gaus_y)/(2*sigma*sigma)));
				}
			}
			//hiprandState_t state;
			//hiprand_init(floor(y * 16384) * 16384 + floor(x * 16384), 0, 0, &state);
			//ret *= 1+((double)(hiprand(&state)%1000)-500)/100000;
			return ret - 0.008857380480028442;
			break;
		}
		// u(x,y)= - y 1/(nu^2 t^2) exp(-(x^2+y^2)/(4 nu t))
		// v(x,y)= + x 1/(nu^2 t^2) exp(-(x^2+y^2)/(4 nu t))
		case 6:  // shielded vortex
		{
			double nu = 2e-1;
			double nu_fac = 1 / (2*nu*nu);  // 1 / (2*nu*nu*nu)
			double nu_center = 4*nu;  // 4*nu
			double nu_scale = 4*nu;  // 4*nu

			// compute distance from center
			double x_r = PI-x; double y_r = PI-y;

			// build vorticity
			return nu_fac * (nu_center - x_r*x_r - y_r*y_r) * exp(-(x_r*x_r + y_r*y_r)/nu_scale);
			break;
		}
		default:  //default case goes to stationary
		{
			x = x - (x>0)*((int)(x/twoPI))*twoPI - (x<0)*((int)(x/twoPI)-1)*twoPI;
			y = y - (y>0)*((int)(y/twoPI))*twoPI - (y<0)*((int)(y/twoPI)-1)*twoPI;

			return cos(x)*cos(y);
			break;
		}
	}

}



__device__ double device_initial_W_discret(double x, double y, double *W_initial, int NX, int NY){
	// map position back into domain
	x = fmod(x, twoPI) - floor(x/twoPI)*twoPI;
	y = fmod(y, twoPI) - floor(y/twoPI)*twoPI;
	
	// compute index
	int In = floor(y/twoPI * NY) * NX + floor(x/twoPI * NX);
	//In = floor(x * NY) * NX + floor(y * NX); 
	
	return W_initial[In];
}
