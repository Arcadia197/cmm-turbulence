#include "hip/hip_runtime.h"
#include "cmm-io.h"


/*******************************************************************
*				     Creation of storage files					   *
*******************************************************************/

void create_directory_structure(SettingsCMM SettingsMain, double dt, int iterMax)
{
	string folder_data = SettingsMain.getWorkspace() + "data";
	struct stat st = {0};
	if (stat(folder_data.c_str(), &st) == -1) mkdir(folder_data.c_str(), 0777);

	//creating main folder
	string folder_name = SettingsMain.getWorkspace() + "data/" + SettingsMain.getFileName();
	mkdir(folder_name.c_str(), 0777);

	// create general subfolder for other data
	string folder_name_tdata = folder_name + "/Monitoring_data";
	mkdir(folder_name_tdata.c_str(), 0777);

	// create general subfolder for timesteps
	folder_name_tdata = folder_name + "/Time_data";
	mkdir(folder_name_tdata.c_str(), 0777);

	// create general subfolder for zoom
	if (SettingsMain.getZoom()) {
		folder_name_tdata = folder_name + "/Zoom_data";
		mkdir(folder_name_tdata.c_str(), 0777);
	}

	string fileName = folder_name + "/readme.txt";
	ofstream file(fileName.c_str(), ios::out);

	if (!file)
	{
		cout<<"Error writting files"<<fileName<<endl;
		exit(0);
	}
	else
	{
        file<<"Simulation name \t\t\t: "<<SettingsMain.getSimName()<<endl;
        switch (SettingsMain.getTimeIntegrationNum()) {
			case 10: { file<<"Time integration \t\t\t: Euler explicit"<<endl; break; }
			case 20: { file<<"Time integration \t\t\t: Adam Bashfords 2"<<endl; break; }
			case 21: { file<<"Time integration \t\t\t: Runge Kutta 2"<<endl; break; }
			case 30: { file<<"Time integration \t\t\t: Runge Kutta 3"<<endl; break; }
			case 40: { file<<"Time integration \t\t\t: Runge Kutta 4"<<endl; break; }
			case 31: { file<<"Time integration \t\t\t: Runge Kutta 3 (modified)"<<endl; break; }
			case 41: { file<<"Time integration \t\t\t: Runge Kutta 4 (modified)"<<endl; break; }
			default: { file<<"Time integration \t\t\t: Default (zero)"<<endl; break; }
		}
        file<<"Lagrange order \t: "<<SettingsMain.getLagrangeOrder()<<endl;

        file<<"N_coarse(resolution coarse grid) \t: "<<SettingsMain.getGridCoarse()<<endl;
		file<<"N_fine(resolution fine grid) \t\t: "<<SettingsMain.getGridFine()<<endl;
		file<<"N_psi(resolution psi grid) \t\t: "<<SettingsMain.getGridPsi()<<endl;
		file<<"N_vort(resolution vort for psi grid) \t: "<<SettingsMain.getGridVort()<<endl;
		file<<"time step dt \t\t: "<<dt<<endl;
		file<<"Final time \t\t: "<<SettingsMain.getFinalTime()<<endl;
		file<<"iter max \t\t: "<<iterMax<<endl;
		file<<"Incomppressibility Threshold \t: "<<SettingsMain.getIncompThreshold()<<endl;
		file<<"Map advection epsilon \t: "<<SettingsMain.getMapEpsilon()<<endl;
		file<<"Map update order \t: "<<SettingsMain.getMapUpdateOrder()<<endl;
		file<<"Cut Psi Frequencies at \t: "<<SettingsMain.getFreqCutPsi()<<endl;
		file<<"Molly stencil version \t: "<<SettingsMain.getMollyStencil()<<endl;

		if (SettingsMain.getZoom()) {
			file<<"Zoom enabled"<<endl;
			file<<"Zoom center x : "<<SettingsMain.getZoomCenterX()<<endl;
			file<<"Zoom center y : "<<SettingsMain.getZoomCenterY()<<endl;
			file<<"Zoom width x : "<<SettingsMain.getZoomWidthX()<<endl;
			file<<"Zoom width y : "<<SettingsMain.getZoomWidthY()<<endl;
			file<<"Zoom repetitions : "<<SettingsMain.getZoomRepetitions()<<endl;
			file<<"Zoom repetition factor : "<<SettingsMain.getZoomRepetitionsFactor()<<endl;
		}

        if (SettingsMain.getParticles()) {
        	file<<"Particles enabled"<<endl;
        	file<<"Amount of particles : "<<SettingsMain.getParticlesNum()<<endl;
            switch (SettingsMain.getParticlesTimeIntegrationNum()) {
    			case 10: { file<<"Particles Time integration : Euler explicit"<<endl; break; }
    			case 20: { file<<"Particles Time integration : Euler midpoint"<<endl; break; }
    			case 30: { file<<"Particles Time integration : Runge Kutta 3"<<endl; break; }
    			case 40: { file<<"Particles Time integration : Runge Kutta 4"<<endl; break; }
    			case 25: { file<<"Particles Time integration : Nicolas Euler midpoint"<<endl; break; }
    			case 35: { file<<"Particles Time integration : Nicolas Runge Kutta 3"<<endl; break; }
    			default: { file<<"Particles Time integration : Default (zero)"<<endl; break; }
    		}
            if (SettingsMain.getSaveFineParticles()) {
                file<<"Safe fine Particles enabled"<<endl;
                file<<"Amount of fine particles : "<<SettingsMain.getParticlesFineNum()<<endl;
            }
        }
        else file<<"Particles disabled"<<endl;

		file.close();
	}
}


// separate call for creating particle folder structure, as this is initialized later
void create_particle_directory_structure(SettingsCMM SettingsMain) {
	// create particle folders
    if (SettingsMain.getParticles()) {
    	string folder_name = SettingsMain.getWorkspace() + "data/" + SettingsMain.getFileName();

    	// main folder
        string fi = folder_name + "/Particle_data";
        mkdir(fi.c_str(), 0777);

        // folder for fluid particle data
        string fi_1 = fi + "/Fluid";
        mkdir(fi_1.c_str(), 0777);
        // folder for fine particle data
        if (SettingsMain.getSaveFineParticles()) {
			fi_1 = fi + "/Fluid_fine";
			mkdir(fi_1.c_str(), 0777);
        }

        // folder for tau_p particles together with fine folder
        for(int i = 1; i<SettingsMain.getParticlesTauNum(); i+=1){
            fi_1 = fi + "/Tau=" + to_str(SettingsMain.particles_tau[i]);
            mkdir(fi_1.c_str(), 0777);

            if (SettingsMain.getSaveFineParticles()) {
				fi_1 = fi + "/Tau=" + to_str(SettingsMain.particles_tau[i]) + "_fine";
				mkdir(fi_1.c_str(), 0777);
            }
        }
	}
}

/*******************************************************************
*					    Writting in binary						   *
*******************************************************************/


void writeAllRealToBinaryFile(int Len, double *var, SettingsCMM SettingsMain, string data_name)
{
	string fileName = SettingsMain.getWorkspace() + "data/" + SettingsMain.getFileName() + data_name + ".data";
	ofstream file(fileName.c_str(), ios::out | ios::binary);

	if(!file)
	{
		cout<<"Error saving file. Unable to open : "<<fileName<<endl;
		return;
	}
	else {
		file.write( (char*) var, Len*sizeof(double) );
//		for (int l=0; l<Len; l++) {
//			file.write( (char*) &var[l], sizeof(double) );
//		}
	}

	file.close();
}


bool readAllRealFromBinaryFile(int Len, double *var, string data_name)
{
	string fileName = data_name;
	ifstream file(fileName.c_str(), ios::in | ios::binary);
	bool open_file;

	if(!file)
	{
		cout<<"Error reading file. Unable to open : "<<fileName<<endl;
		open_file = false;
	}
	else {
		file.read( (char*) var, Len*sizeof(double) );
//		for (int l=0; l<Len; l++) {
//			file.read( (char*) &var[l], sizeof(double) );
//		}
		open_file = true;
	}

	file.close();
	return open_file;
}


/*******************************************************************
* Structures to create or save on timestep in either hdf5 or binary
*
* hdf5: create subgroup for the timestep and save values there
* add attributes to group
*
* binary: create subfolder for the timestep and save values there
* attributes are not directly given, maybe over a readme file in folder
*******************************************************************/

// hdf5 version
#ifdef HDF5_INCLUDE
	void writeTimeStep(string workspace, string file_name, string i_num, double *Host_save, double *Dev_W_coarse, double *Dev_W_fine, double *Dev_Psi_real, double *Dev_ChiX, double *Dev_ChiY, TCudaGrid2D *Grid_fine, TCudaGrid2D *Grid_coarse, TCudaGrid2D *Grid_psi) {

	}

// binary version
#else
	void writeTimeStep(SettingsCMM SettingsMain, std::string i_num, double *Host_save, double *Dev_W_coarse, double *Dev_W_fine,
			double *Dev_Psi_real, double *Dev_ChiX, double *Dev_ChiY, TCudaGrid2D Grid_fine, TCudaGrid2D Grid_coarse, TCudaGrid2D Grid_psi) {

		// create new subfolder for current timestep
		std::string sub_folder_name = "/Time_data/Time_" + i_num;
		std::string folder_name_now = SettingsMain.getWorkspace() + "data/" + SettingsMain.getFileName() + sub_folder_name;
		struct stat st = {0};
		if (stat(folder_name_now.c_str(), &st) == -1) mkdir(folder_name_now.c_str(), 0777);

		// execute binary save for all wanted variables
		std::string save_var = SettingsMain.getSaveVar();

		// Vorticity on coarse grid : W_coarse
		if (save_var.find("Vorticity") != std::string::npos or save_var.find("W") != std::string::npos) {
			hipMemcpy(Host_save, Dev_W_coarse, Grid_coarse.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(Grid_coarse.N, Host_save, SettingsMain, sub_folder_name + "/Vorticity_W_coarse");
		}
		// Vorticity on fine grid : W_fine
//		hipMemcpy(Host_save, Dev_W_fine, Grid_fine.sizeNReal, hipMemcpyDeviceToHost);
//	    writeAllRealToBinaryFile(Grid_fine.N, Host_save, SettingsMain, sub_folder_name + "/Vorticity_W_fine");

		// Stream function on psi grid : Psi_psi
		if (save_var.find("Stream") != std::string::npos or save_var.find("Psi") != std::string::npos) {
			hipMemcpy(Host_save, Dev_Psi_real, Grid_psi.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(Grid_psi.N, Host_save, SettingsMain, sub_folder_name + "/Stream_function_Psi_psi");
		}
		if (save_var.find("Stream_H") != std::string::npos or save_var.find("Psi_H") != std::string::npos) {
			hipMemcpy(Host_save, Dev_Psi_real, 4*Grid_psi.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(4*Grid_psi.N, Host_save, SettingsMain, sub_folder_name + "/Stream_function_Psi_H_psi");
		}

		// Velocity on psi grid : U_psi
		if (save_var.find("Velocity") != std::string::npos or save_var.find("U") != std::string::npos) {
			// Velocity in x direction
			hipMemcpy(Host_save, Dev_Psi_real+1*Grid_psi.N, Grid_psi.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(Grid_psi.N, Host_save, SettingsMain, sub_folder_name + "/Velocity_UX_psi");
			// Velocity in y direction
			hipMemcpy(Host_save, Dev_Psi_real+2*Grid_psi.N, Grid_psi.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(Grid_psi.N, Host_save, SettingsMain, sub_folder_name + "/Velocity_UY_psi");
		}

		// Map on coarse grid in Hermite or single version : Chi_coarse
		if (save_var.find("Map") != std::string::npos or save_var.find("Chi") != std::string::npos) {
			// Map in x direction on coarse grid : ChiX
			hipMemcpy(Host_save, Dev_ChiX, Grid_coarse.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(Grid_coarse.N, Host_save, SettingsMain, sub_folder_name + "/Map_ChiX_coarse");
			// Map in y direction on coarse grid : ChiY
			hipMemcpy(Host_save, Dev_ChiY, Grid_coarse.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(Grid_coarse.N, Host_save, SettingsMain, sub_folder_name + "/Map_ChiY_coarse");
		}
		if (save_var.find("Map_H") != std::string::npos or save_var.find("Chi_H") != std::string::npos) {
			// Map in x direction on coarse grid : ChiX
			hipMemcpy(Host_save, Dev_ChiX, 4*Grid_coarse.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(4*Grid_coarse.N, Host_save, SettingsMain, sub_folder_name + "/Map_ChiX_H_coarse");
			// Map in y direction on coarse grid : ChiY
			hipMemcpy(Host_save, Dev_ChiY, 4*Grid_coarse.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(4*Grid_coarse.N, Host_save, SettingsMain, sub_folder_name + "/Map_ChiY_H_coarse");
		}
	}
#endif


// script to save only one of the variables, needed because we need temporal arrays to save
void writeTimeVariable(SettingsCMM SettingsMain, string data_name, string i_num, double *Host_save, double *Dev_save, long int size_N, long int N) {
	// create new subfolder for current timestep, doesn't matter if we try to create it several times
	string sub_folder_name = "/Time_data/Time_" + i_num;
	string folder_name_now = SettingsMain.getWorkspace() + "data/" + SettingsMain.getFileName() + sub_folder_name;
	struct stat st = {0};
	if (stat(folder_name_now.c_str(), &st) == -1) mkdir(folder_name_now.c_str(), 0777);
//	mkdir(folder_name_now.c_str(), 0777);

	// copy and save
	hipMemcpy(Host_save, Dev_save, size_N, hipMemcpyDeviceToHost);
	writeAllRealToBinaryFile(N, Host_save, SettingsMain, sub_folder_name + "/" + data_name);
}

// script to save only one of the variables, but with offset
void writeTimeVariable(SettingsCMM SettingsMain, string data_name, string i_num, double *Host_save, double *Dev_save, long int size_N, long int N, int offset) {
	// create new subfolder for current timestep, doesn't matter if we try to create it several times
	string sub_folder_name = "/Time_data/Time_" + i_num;
	string folder_name_now = SettingsMain.getWorkspace() + "data/" + SettingsMain.getFileName() + sub_folder_name;
	struct stat st = {0};
	if (stat(folder_name_now.c_str(), &st) == -1) mkdir(folder_name_now.c_str(), 0777);
//	mkdir(folder_name_now.c_str(), 0777);

	// copy and save
	hipMemcpy2D(Host_save, sizeof(double), Dev_save, sizeof(double)*2,
			sizeof(double), N, hipMemcpyDeviceToHost);
	writeAllRealToBinaryFile(N, Host_save, SettingsMain, sub_folder_name + "/" + data_name);
}



/*
 * Write particle positions
 */
// will be with hdf5 version too at some point
void writeParticles(SettingsCMM SettingsMain, string i_num, double *Host_particles_pos, double *Dev_particles_pos) {
	// copy data to host
    hipMemcpy(Host_particles_pos, Dev_particles_pos, 2*SettingsMain.getParticlesNum()*SettingsMain.getParticlesTauNum()*sizeof(double), hipMemcpyDeviceToHost);
    //hipDeviceSynchronize();
    writeAllRealToBinaryFile(2*SettingsMain.getParticlesNum(), Host_particles_pos, SettingsMain, "/Particle_data/Fluid/Particles_pos_" + i_num);
    for(int i = 1; i < SettingsMain.getParticlesTauNum(); i+=1)
        writeAllRealToBinaryFile(2*SettingsMain.getParticlesNum(), Host_particles_pos + i * 2*SettingsMain.getParticlesNum(), SettingsMain, "/Particle_data/Tau=" + to_str(SettingsMain.particles_tau[i]) + "/Particles_pos_" + i_num);
}

void writeFineParticles(SettingsCMM SettingsMain, string i_num, double *Host_particles_fine_pos, int fine_particle_save_num) {
	writeAllRealToBinaryFile(2*fine_particle_save_num, Host_particles_fine_pos, SettingsMain, "/Particle_data/Fluid_fine/Particles_pos_" + i_num);

    for(int i = 1; i < SettingsMain.getParticlesTauNum(); i+=1) {
		writeAllRealToBinaryFile(2*fine_particle_save_num, Host_particles_fine_pos, SettingsMain, "/Particle_data/Tau="+to_str(SettingsMain.particles_tau[i])+"_fine/Particles_pos_" + i_num);
    }
}


// save the map stack, only save used maps though
void writeMapStack(SettingsCMM SettingsMain, MapStack Map_Stack) {
	// create new subfolder for mapstack, doesn't matter if we try to create it several times
	string sub_folder_name = "/MapStack";
	string folder_name_now = SettingsMain.getWorkspace() + "data/" + SettingsMain.getFileName() + sub_folder_name;
	struct stat st = {0};
	if (stat(folder_name_now.c_str(), &st) == -1) mkdir(folder_name_now.c_str(), 0777);

	// check if we have to save a stack for every stack
	int save_ctr;
	if (Map_Stack.map_stack_ctr / (double)Map_Stack.cpu_map_num > 0) {
		if (Map_Stack.map_stack_ctr > 1*Map_Stack.cpu_map_num) save_ctr = Map_Stack.cpu_map_num;
		else save_ctr = Map_Stack.map_stack_ctr - 0*Map_Stack.cpu_map_num;
		printf("Save %d maps of map stack 1\n",save_ctr);
		writeAllRealToBinaryFile(save_ctr*4*Map_Stack.Grid->N, Map_Stack.Host_ChiX_stack_RAM_0, SettingsMain, "/MapStack/MapStack_ChiX_0");
		writeAllRealToBinaryFile(save_ctr*4*Map_Stack.Grid->N, Map_Stack.Host_ChiY_stack_RAM_0, SettingsMain, "/MapStack/MapStack_ChiY_0");
	}
	else if (Map_Stack.map_stack_ctr / (double)Map_Stack.cpu_map_num > 1) {
		if (Map_Stack.map_stack_ctr > 2*Map_Stack.cpu_map_num) save_ctr = Map_Stack.cpu_map_num;
		else save_ctr = Map_Stack.map_stack_ctr - 1*Map_Stack.cpu_map_num;
		writeAllRealToBinaryFile(save_ctr*4*Map_Stack.Grid->N, Map_Stack.Host_ChiX_stack_RAM_1, SettingsMain, "/MapStack/MapStack_ChiX_1");
		writeAllRealToBinaryFile(save_ctr*4*Map_Stack.Grid->N, Map_Stack.Host_ChiY_stack_RAM_1, SettingsMain, "/MapStack/MapStack_ChiY_1");
	}
	else if (Map_Stack.map_stack_ctr / (double)Map_Stack.cpu_map_num > 2) {
		if (Map_Stack.map_stack_ctr > 3*Map_Stack.cpu_map_num) save_ctr = Map_Stack.cpu_map_num;
		else save_ctr = Map_Stack.map_stack_ctr - 2*Map_Stack.cpu_map_num;
		writeAllRealToBinaryFile(save_ctr*4*Map_Stack.Grid->N, Map_Stack.Host_ChiX_stack_RAM_2, SettingsMain, "/MapStack/MapStack_ChiX_2");
		writeAllRealToBinaryFile(save_ctr*4*Map_Stack.Grid->N, Map_Stack.Host_ChiY_stack_RAM_2, SettingsMain, "/MapStack/MapStack_ChiY_2");
	}
	else if (Map_Stack.map_stack_ctr / (double)Map_Stack.cpu_map_num > 3) {
		if (Map_Stack.map_stack_ctr > 4*Map_Stack.cpu_map_num) save_ctr = Map_Stack.cpu_map_num;
		else save_ctr = Map_Stack.map_stack_ctr - 3*Map_Stack.cpu_map_num;
		writeAllRealToBinaryFile(save_ctr*4*Map_Stack.Grid->N, Map_Stack.Host_ChiX_stack_RAM_3, SettingsMain, "/MapStack/MapStack_ChiX_3");
		writeAllRealToBinaryFile(save_ctr*4*Map_Stack.Grid->N, Map_Stack.Host_ChiY_stack_RAM_3, SettingsMain, "/MapStack/MapStack_ChiY_3");
	}
}


Logger::Logger(SettingsCMM SettingsMain)
{
	fileName = SettingsMain.getWorkspace() + "data/" + SettingsMain.getFileName() + "/log.txt";
	file.open(fileName.c_str(), ios::out);
//	file.open(fileName.c_str(), ios::out | ios::app);  // append to file for continuing simulation

	if(!file)
	{
		cout<<"Unable to open log file.. exitting\n";
		exit(0);
	}
	else
	{
		file<<SettingsMain.getFileName()<<endl;  // if file existed, this will basically overwrite it
		file.close();
	}
}


void Logger::push(string message)
{
	file.open(fileName.c_str(), ios::out | ios::app);

	if(file)
	{
		file<<"["<<currentDateTime()<<"]\t";
		file<<message<<endl;
		file.close();
	}
}


void Logger::push()
{
	push(buffer);
}


const std::string currentDateTime() {
    time_t     now = time(0);
    struct tm  tstruct;
    char       buf[80];
    tstruct = *localtime(&now);
    // Visit http://en.cppreference.com/w/cpp/chrono/c/strftime
    // for more information about date/time format
    strftime(buf, sizeof(buf), "%Y-%m-%d.%X", &tstruct);

    return buf;
}

// helper function to format time to readable format
string format_duration(double sec) {
	return to_str(floor(sec/3600.0)) + "h " + to_str(floor(std::fmod(sec, 3600)/60.0)) + "m " + to_str(std::fmod(sec, 60)) + "s";
}

