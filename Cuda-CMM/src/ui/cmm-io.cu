#include "hip/hip_runtime.h"
#include "cmm-io.h"


/*******************************************************************
*				     Creation of storage files					   *
*******************************************************************/

void create_directory_structure(SettingsCMM SettingsMain, double dt, int iterMax)
{
	string folder_data = SettingsMain.getWorkspace() + "data";
	struct stat st = {0};
	if (stat(folder_data.c_str(), &st) == -1) mkdir(folder_data.c_str(), 0777);

	//creating main folder
	string folder_name = SettingsMain.getWorkspace() + "data/" + SettingsMain.getFileName();
	mkdir(folder_name.c_str(), 0777);

	// create general subfolder for other data
	string folder_name_tdata = folder_name + "/Monitoring_data";
	mkdir(folder_name_tdata.c_str(), 0777);

	// create general subfolder for timesteps
	folder_name_tdata = folder_name + "/Time_data";
	mkdir(folder_name_tdata.c_str(), 0777);

	// create general subfolder for zoom
	if (SettingsMain.getZoom()) {
		folder_name_tdata = folder_name + "/Zoom_data";
		mkdir(folder_name_tdata.c_str(), 0777);
	}

	// empty all monitoring data so that we can later flush every value
	std::string monitoring_names[7] = {"/Error_incompressibility", "/Map_counter", "/Map_gaps",
			"/Timesteps", "/Timing_Values", "/Mesure", "/Mesure_fine",
	};
	for ( const auto &i_mon_names : monitoring_names) {
		std::string fileName = folder_name + "/Monitoring_data" + i_mon_names + ".data";
		ofstream file(fileName.c_str(), std::ios::out | std::ios::trunc);
		file.close();
	}
	// empty out mesure file for sample
	if (SettingsMain.getSampleOnGrid()) {
		std::string fileName = folder_name + "/Monitoring_data/Mesure_" + to_str(SettingsMain.getGridSample()) + ".data";
		ofstream file(fileName.c_str(), std::ios::out | std::ios::trunc);
		file.close();
	}
	if (SettingsMain.getForwardMap()) {
		std::string fileName = folder_name + "/Monitoring_data/Error_incompressibility_forward.data";
		ofstream file(fileName.c_str(), std::ios::out | std::ios::trunc);
		file.close();
		std::string fileName2 = folder_name + "/Monitoring_data/Error_invertibility.data";
		ofstream file2(fileName2.c_str(), std::ios::out | std::ios::trunc);
		file.close();
	}
}


// separate call for creating particle folder structure, as this is initialized later
void create_particle_directory_structure(SettingsCMM SettingsMain) {
	// create particle folders
    if (SettingsMain.getParticles()) {
    	string folder_name = SettingsMain.getWorkspace() + "data/" + SettingsMain.getFileName();

    	// main folder
        string fi = folder_name + "/Particle_data";
        mkdir(fi.c_str(), 0777);

        // folder for fluid particle data
        string fi_1 = fi + "/Fluid";
        mkdir(fi_1.c_str(), 0777);
        // folder for fine particle data
        if (SettingsMain.getSaveFineParticles()) {
			fi_1 = fi + "/Fluid_fine";
			mkdir(fi_1.c_str(), 0777);
        }

        // folder for tau_p particles together with fine folder
        for(int i = 1; i<SettingsMain.getParticlesTauNum(); i+=1){
            fi_1 = fi + "/Tau=" + to_str(SettingsMain.particles_tau[i]);
            mkdir(fi_1.c_str(), 0777);

            if (SettingsMain.getSaveFineParticles()) {
				fi_1 = fi + "/Tau=" + to_str(SettingsMain.particles_tau[i]) + "_fine";
				mkdir(fi_1.c_str(), 0777);
            }
        }
	}
}

/*******************************************************************
*					    Writting in binary						   *
*******************************************************************/


void writeAllRealToBinaryFile(int Len, double *var, SettingsCMM SettingsMain, string data_name)
{
	string fileName = SettingsMain.getWorkspace() + "data/" + SettingsMain.getFileName() + data_name + ".data";
	ofstream file(fileName.c_str(), ios::out | ios::binary);

	if(!file)
	{
		cout<<"Error saving file. Unable to open : "<<fileName<<endl;
		return;
	}
	else {
		file.write( (char*) var, Len*sizeof(double) );
	}

	file.close();
}


void writeAppendToBinaryFile(int Len, double *var, SettingsCMM SettingsMain, string data_name)
{
	string fileName = SettingsMain.getWorkspace() + "data/" + SettingsMain.getFileName() + data_name + ".data";
	ofstream file(fileName.c_str(), ios::out | ios::app | ios::binary);

	if(!file)
	{
		cout<<"Error saving file. Unable to open : "<<fileName<<endl;
		return;
	}
	else {
		file.write( (char*) var, Len*sizeof(double) );
	}

	file.close();
}


bool readAllRealFromBinaryFile(int Len, double *var, string data_name)
{
	string fileName = data_name;
	ifstream file(fileName.c_str(), ios::in | ios::binary);
	bool open_file;

	if(!file)
	{
		cout<<"Error reading file. Unable to open : "<<fileName<<endl;
		open_file = false;
	}
	else {
		file.read( (char*) var, Len*sizeof(double) );
		open_file = true;
	}

	file.close();
	return open_file;
}


/*******************************************************************
* Structures to create or save on timestep in either hdf5 or binary
*
* hdf5: create subgroup for the timestep and save values there
* add attributes to group
*
* binary: create subfolder for the timestep and save values there
* attributes are not directly given, maybe over a readme file in folder
*******************************************************************/

// hdf5 version
#ifdef HDF5_INCLUDE
void writeTimeStep(SettingsCMM SettingsMain, std::string i_num, TCudaGrid2D Grid_fine, TCudaGrid2D Grid_coarse, TCudaGrid2D Grid_psi,
		double *Host_save, double *Dev_W_coarse, double *Dev_W_fine, double *Dev_Psi_real,
		double *Dev_ChiX, double *Dev_ChiY, double *Dev_ChiX_f, double *Dev_ChiY_f) {
	}

// binary version
#else
	void writeTimeStep(SettingsCMM SettingsMain, std::string i_num, TCudaGrid2D Grid_fine, TCudaGrid2D Grid_coarse, TCudaGrid2D Grid_psi,
			double *Host_save, double *Dev_W_coarse, double *Dev_W_fine, double *Dev_Psi_real,
			double *Dev_ChiX, double *Dev_ChiY, double *Dev_ChiX_f, double *Dev_ChiY_f) {

		// create new subfolder for current timestep
		std::string sub_folder_name = "/Time_data/Time_" + i_num;
		std::string folder_name_now = SettingsMain.getWorkspace() + "data/" + SettingsMain.getFileName() + sub_folder_name;
		struct stat st = {0};
		if (stat(folder_name_now.c_str(), &st) == -1) mkdir(folder_name_now.c_str(), 0777);

		// execute binary save for all wanted variables
		std::string save_var = SettingsMain.getSaveVar();

		// Vorticity on coarse grid : W_coarse
		if (save_var.find("Vorticity") != std::string::npos or save_var.find("W") != std::string::npos) {
			hipMemcpy(Host_save, Dev_W_coarse, Grid_coarse.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(Grid_coarse.N, Host_save, SettingsMain, sub_folder_name + "/Vorticity_W_coarse");
		}
		// Vorticity on fine grid : W_fine
//		hipMemcpy(Host_save, Dev_W_fine, Grid_fine.sizeNReal, hipMemcpyDeviceToHost);
//	    writeAllRealToBinaryFile(Grid_fine.N, Host_save, SettingsMain, sub_folder_name + "/Vorticity_W_fine");

		// Stream function on psi grid : Psi_psi
		if (save_var.find("Stream") != std::string::npos or save_var.find("Psi") != std::string::npos) {
			hipMemcpy(Host_save, Dev_Psi_real, Grid_psi.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(Grid_psi.N, Host_save, SettingsMain, sub_folder_name + "/Stream_function_Psi_psi");
		}
		if (save_var.find("Stream_H") != std::string::npos or save_var.find("Psi_H") != std::string::npos) {
			hipMemcpy(Host_save, Dev_Psi_real, 4*Grid_psi.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(4*Grid_psi.N, Host_save, SettingsMain, sub_folder_name + "/Stream_function_Psi_H_psi");
		}

		// Velocity on psi grid : U_psi
		if (save_var.find("Velocity") != std::string::npos or save_var.find("U") != std::string::npos) {
			// Velocity in x direction
			hipMemcpy(Host_save, Dev_Psi_real+1*Grid_psi.N, Grid_psi.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(Grid_psi.N, Host_save, SettingsMain, sub_folder_name + "/Velocity_UX_psi");
			// Velocity in y direction
			hipMemcpy(Host_save, Dev_Psi_real+2*Grid_psi.N, Grid_psi.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(Grid_psi.N, Host_save, SettingsMain, sub_folder_name + "/Velocity_UY_psi");
		}

		// Backwards map on coarse grid in Hermite or single version : Chi_coarse
		if (save_var.find("Map") != std::string::npos or save_var.find("Chi") != std::string::npos) {
			// Map in x direction on coarse grid : ChiX
			hipMemcpy(Host_save, Dev_ChiX, Grid_coarse.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(Grid_coarse.N, Host_save, SettingsMain, sub_folder_name + "/Map_ChiX_coarse");
			// Map in y direction on coarse grid : ChiY
			hipMemcpy(Host_save, Dev_ChiY, Grid_coarse.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(Grid_coarse.N, Host_save, SettingsMain, sub_folder_name + "/Map_ChiY_coarse");
		}
		if (save_var.find("Map_H") != std::string::npos or save_var.find("Chi_H") != std::string::npos) {
			// Map in x direction on coarse grid : ChiX
			hipMemcpy(Host_save, Dev_ChiX, 4*Grid_coarse.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(4*Grid_coarse.N, Host_save, SettingsMain, sub_folder_name + "/Map_ChiX_H_coarse");
			// Map in y direction on coarse grid : ChiY
			hipMemcpy(Host_save, Dev_ChiY, 4*Grid_coarse.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(4*Grid_coarse.N, Host_save, SettingsMain, sub_folder_name + "/Map_ChiY_H_coarse");
		}

		// Forwards map on coarse grid in Hermite or single version : Chi_f_coarse
		if (save_var.find("Map_f") != std::string::npos or save_var.find("Chi_f") != std::string::npos) {
			// Map in x direction on coarse grid : ChiX
			hipMemcpy(Host_save, Dev_ChiX_f, Grid_coarse.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(Grid_coarse.N, Host_save, SettingsMain, sub_folder_name + "/Map_ChiX_f_coarse");
			// Map in y direction on coarse grid : ChiY
			hipMemcpy(Host_save, Dev_ChiY_f, Grid_coarse.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(Grid_coarse.N, Host_save, SettingsMain, sub_folder_name + "/Map_ChiY_f_coarse");
		}
		if (save_var.find("Map_f_H") != std::string::npos or save_var.find("Chi_f_H") != std::string::npos) {
			// Map in x direction on coarse grid : ChiX
			hipMemcpy(Host_save, Dev_ChiX_f, 4*Grid_coarse.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(4*Grid_coarse.N, Host_save, SettingsMain, sub_folder_name + "/Map_ChiX_f_H_coarse");
			// Map in y direction on coarse grid : ChiY
			hipMemcpy(Host_save, Dev_ChiY_f, 4*Grid_coarse.sizeNReal, hipMemcpyDeviceToHost);
			writeAllRealToBinaryFile(4*Grid_coarse.N, Host_save, SettingsMain, sub_folder_name + "/Map_ChiY_f_H_coarse");
		}
	}
#endif


// script to save only one of the variables, needed because we need temporal arrays to save
void writeTimeVariable(SettingsCMM SettingsMain, string data_name, string i_num, double *Host_save, double *Dev_save, long int size_N, long int N) {
	// create new subfolder for current timestep, doesn't matter if we try to create it several times
	string sub_folder_name = "/Time_data/Time_" + i_num;
	string folder_name_now = SettingsMain.getWorkspace() + "data/" + SettingsMain.getFileName() + sub_folder_name;
	struct stat st = {0};
	if (stat(folder_name_now.c_str(), &st) == -1) mkdir(folder_name_now.c_str(), 0777);
//	mkdir(folder_name_now.c_str(), 0777);

	// copy and save
	hipMemcpy(Host_save, Dev_save, size_N, hipMemcpyDeviceToHost);
	writeAllRealToBinaryFile(N, Host_save, SettingsMain, sub_folder_name + "/" + data_name);
}

// script to save only one of the variables, but with offset
void writeTimeVariable(SettingsCMM SettingsMain, string data_name, string i_num, double *Host_save, double *Dev_save, long int size_N, long int N, int offset) {
	// create new subfolder for current timestep, doesn't matter if we try to create it several times
	string sub_folder_name = "/Time_data/Time_" + i_num;
	string folder_name_now = SettingsMain.getWorkspace() + "data/" + SettingsMain.getFileName() + sub_folder_name;
	struct stat st = {0};
	if (stat(folder_name_now.c_str(), &st) == -1) mkdir(folder_name_now.c_str(), 0777);
//	mkdir(folder_name_now.c_str(), 0777);

	// copy and save
	hipMemcpy2D(Host_save, sizeof(double), Dev_save, sizeof(double)*2,
			sizeof(double), N, hipMemcpyDeviceToHost);
	writeAllRealToBinaryFile(N, Host_save, SettingsMain, sub_folder_name + "/" + data_name);
}



/*
 * Write particle positions
 */
// will be with hdf5 version too at some point
void writeParticles(SettingsCMM SettingsMain, string i_num, double *Host_particles_pos, double *Dev_particles_pos) {
	// copy data to host
    hipMemcpy(Host_particles_pos, Dev_particles_pos, 2*SettingsMain.getParticlesNum()*SettingsMain.getParticlesTauNum()*sizeof(double), hipMemcpyDeviceToHost);
    //hipDeviceSynchronize();
    writeAllRealToBinaryFile(2*SettingsMain.getParticlesNum(), Host_particles_pos, SettingsMain, "/Particle_data/Fluid/Particles_pos_" + i_num);
    for(int i = 1; i < SettingsMain.getParticlesTauNum(); i+=1)
        writeAllRealToBinaryFile(2*SettingsMain.getParticlesNum(), Host_particles_pos + i * 2*SettingsMain.getParticlesNum(), SettingsMain, "/Particle_data/Tau=" + to_str(SettingsMain.particles_tau[i]) + "/Particles_pos_" + i_num);
}

void writeFineParticles(SettingsCMM SettingsMain, string i_num, double *Host_particles_fine_pos, int fine_particle_save_num) {
	writeAllRealToBinaryFile(2*fine_particle_save_num, Host_particles_fine_pos, SettingsMain, "/Particle_data/Fluid_fine/Particles_pos_" + i_num);

    for(int i = 1; i < SettingsMain.getParticlesTauNum(); i+=1) {
		writeAllRealToBinaryFile(2*fine_particle_save_num, Host_particles_fine_pos, SettingsMain, "/Particle_data/Tau="+to_str(SettingsMain.particles_tau[i])+"_fine/Particles_pos_" + i_num);
    }
}


// save the map stack, only save used maps though
void writeMapStack(SettingsCMM SettingsMain, MapStack Map_Stack) {
	// create new subfolder for mapstack, doesn't matter if we try to create it several times
	string sub_folder_name = "/MapStack";
	string folder_name_now = SettingsMain.getWorkspace() + "data/" + SettingsMain.getFileName() + sub_folder_name;
	struct stat st = {0};
	if (stat(folder_name_now.c_str(), &st) == -1) mkdir(folder_name_now.c_str(), 0777);

	// check if we have to save a stack for every stack
	int save_ctr;
	if (Map_Stack.map_stack_ctr / (double)Map_Stack.cpu_map_num > 0) {
		if (Map_Stack.map_stack_ctr > 1*Map_Stack.cpu_map_num) save_ctr = Map_Stack.cpu_map_num;
		else save_ctr = Map_Stack.map_stack_ctr - 0*Map_Stack.cpu_map_num;
		printf("Save %d maps of map stack 1\n",save_ctr);
		writeAllRealToBinaryFile(save_ctr*4*Map_Stack.Grid->N, Map_Stack.Host_ChiX_stack_RAM_0, SettingsMain, "/MapStack/MapStack_ChiX_0");
		writeAllRealToBinaryFile(save_ctr*4*Map_Stack.Grid->N, Map_Stack.Host_ChiY_stack_RAM_0, SettingsMain, "/MapStack/MapStack_ChiY_0");
	}
	else if (Map_Stack.map_stack_ctr / (double)Map_Stack.cpu_map_num > 1) {
		if (Map_Stack.map_stack_ctr > 2*Map_Stack.cpu_map_num) save_ctr = Map_Stack.cpu_map_num;
		else save_ctr = Map_Stack.map_stack_ctr - 1*Map_Stack.cpu_map_num;
		writeAllRealToBinaryFile(save_ctr*4*Map_Stack.Grid->N, Map_Stack.Host_ChiX_stack_RAM_1, SettingsMain, "/MapStack/MapStack_ChiX_1");
		writeAllRealToBinaryFile(save_ctr*4*Map_Stack.Grid->N, Map_Stack.Host_ChiY_stack_RAM_1, SettingsMain, "/MapStack/MapStack_ChiY_1");
	}
	else if (Map_Stack.map_stack_ctr / (double)Map_Stack.cpu_map_num > 2) {
		if (Map_Stack.map_stack_ctr > 3*Map_Stack.cpu_map_num) save_ctr = Map_Stack.cpu_map_num;
		else save_ctr = Map_Stack.map_stack_ctr - 2*Map_Stack.cpu_map_num;
		writeAllRealToBinaryFile(save_ctr*4*Map_Stack.Grid->N, Map_Stack.Host_ChiX_stack_RAM_2, SettingsMain, "/MapStack/MapStack_ChiX_2");
		writeAllRealToBinaryFile(save_ctr*4*Map_Stack.Grid->N, Map_Stack.Host_ChiY_stack_RAM_2, SettingsMain, "/MapStack/MapStack_ChiY_2");
	}
	else if (Map_Stack.map_stack_ctr / (double)Map_Stack.cpu_map_num > 3) {
		if (Map_Stack.map_stack_ctr > 4*Map_Stack.cpu_map_num) save_ctr = Map_Stack.cpu_map_num;
		else save_ctr = Map_Stack.map_stack_ctr - 3*Map_Stack.cpu_map_num;
		writeAllRealToBinaryFile(save_ctr*4*Map_Stack.Grid->N, Map_Stack.Host_ChiX_stack_RAM_3, SettingsMain, "/MapStack/MapStack_ChiX_3");
		writeAllRealToBinaryFile(save_ctr*4*Map_Stack.Grid->N, Map_Stack.Host_ChiY_stack_RAM_3, SettingsMain, "/MapStack/MapStack_ChiY_3");
	}
}


Logger::Logger(SettingsCMM SettingsMain)
{
	fileName = SettingsMain.getWorkspace() + "data/" + SettingsMain.getFileName() + "/log.txt";
	file.open(fileName.c_str(), ios::out);
//	file.open(fileName.c_str(), ios::out | ios::app);  // append to file for continuing simulation

	if(!file)
	{
		cout<<"Unable to open log file.. exitting\n";
		exit(0);
	}
	else
	{
		file<<SettingsMain.getFileName()<<endl;  // if file existed, this will basically overwrite it
		file.close();
	}
}


void Logger::push(string message)
{
	file.open(fileName.c_str(), ios::out | ios::app);

	if(file)
	{
		file<<"["<<currentDateTime()<<"]\t";
		file<<message<<endl;
		file.close();
	}
}


void Logger::push()
{
	push(buffer);
}


const std::string currentDateTime() {
    time_t     now = time(0);
    struct tm  tstruct;
    char       buf[80];
    tstruct = *localtime(&now);
    // Visit http://en.cppreference.com/w/cpp/chrono/c/strftime
    // for more information about date/time format
    strftime(buf, sizeof(buf), "%Y-%m-%d.%X", &tstruct);

    return buf;
}

// helper function to format time to readable format
string format_duration(double sec) {
	return to_str(floor(sec/3600.0)) + "h " + to_str(floor(std::fmod(sec, 3600)/60.0)) + "m " + to_str(std::fmod(sec, 60)) + "s";
}

