#include "hip/hip_runtime.h"
#include "cudagrid2d.h"


TCudaGrid2D::TCudaGrid2D(int NX, int NY, double xRange)
{
	this->NX = NX;
	this->NY = NY;
	
	this->h = xRange/(float)NX;
	
	this->N = NX*NY;
	this->sizeNReal = sizeof(double)*N;
	this->sizeNComplex = sizeof(hipfftDoubleComplex)*N;

	//block & grid
	threadsPerBlock.x = BLOCK_SIZE;
	threadsPerBlock.y = BLOCK_SIZE;
	threadsPerBlock.z = 1;

	blocksPerGrid.x = ceil((float)NX/threadsPerBlock.x);
	blocksPerGrid.y = ceil((float)NY/threadsPerBlock.y);
	blocksPerGrid.z = 1;

	// debug information about grid, maybe add a verbose parameter?
//	printf("Grid      : (%d, %d)\n", NX, NY);
//	printf("Block Dim : (%d, %d, %d)\n", threadsPerBlock.x, threadsPerBlock.y, threadsPerBlock.z);
//	printf("Grid Dim  : (%d, %d, %d)\n\n", blocksPerGrid.x, blocksPerGrid.y, blocksPerGrid.z);
}


void get_max_min(TCudaGrid2D *G, double *var, double *min, double *max)
{
	//calculating max and min
	double var_min, var_max;
	
		for(int i=0; i<G->N; i++)
		{
			if(i==0)
			{
				var_min = var[i];
				var_max = var[i];
			}
			
			if(var_min > var[i])
				var_min = var[i];
				
			if(var_max < var[i])
				var_max = var[i];
		}
	
	*min = var_min;
	*max = var_max;
}


void Host_get_max_min(int len, double *Var_min, double *Var_max, double *min_f, double *max_f)
{
	//calculating max and min

	double min = Var_min[0];
	double max = Var_max[0];
	
	for(int i=0;i<len;i++)
	{			
		if(min > Var_min[i])
			min = Var_min[i];
			
		if(max < Var_max[i])
			max = Var_max[i];
	}

	*min_f = min;
	*max_f = max;
}


__global__ void Dev_get_max_min(int len, double *var, double *min, double *max)
{
    
	int In = threadIdx.x + blockDim.x * blockIdx.x;
	int Di = blockDim.x * gridDim.x;
	
	int pos = len / Di * In, step_pos = len / Di;
	//calculating max and min
	double var_min, var_max;

	var_min = var[pos];
	var_max = var[pos];
	
	for(int i=pos;i<pos + step_pos;i++)
	{
		if(var_min > var[i])
			var_min = var[i];
			
		if(var_max < var[i])
			var_max = var[i];
	}
	
	min[In] = var_min;
	max[In] = var_max;
}
