#include "hip/hip_runtime.h"
#include "cmm-grid2d.h"


TCudaGrid2D::TCudaGrid2D (int NX, int NY, double *bounds)
{
	this->NX = NX;
	this->NX_fft = (int)(NX/2.0+1);
	this->NY = NY;

	this->h = (bounds[1] - bounds[0]) / (float)NX;  // for quadratic problems, is used everywhere so changing it is tedious

	this->hx = (bounds[1] - bounds[0]) / (float)NX;
	this->hy = (bounds[3] - bounds[2]) / (float)NY;

	for (int i_b = 0; i_b < 4; ++i_b) {
		this->bounds[i_b] = bounds[i_b];
	}

	this->N = NX*NY;
	this->Nfft = NX_fft*NY;

	this->sizeNReal = sizeof(double)*N;
	this->sizeNComplex = sizeof(hipfftDoubleComplex)*N;
	this->sizeNfft = sizeof(hipfftDoubleComplex)*Nfft;  // fft D2Z and Z2D size

	//block & grid
	threadsPerBlock.x = BLOCK_SIZE;
	threadsPerBlock.y = BLOCK_SIZE;
	threadsPerBlock.z = 1;

	blocksPerGrid.x = ceil(NX/(double)threadsPerBlock.x);
	blocksPerGrid.y = ceil(NY/(double)threadsPerBlock.y);
	blocksPerGrid.z = 1;

	fft_blocks.x = ceil((NX+1)/2.0/(double)threadsPerBlock.x);
	fft_blocks.y = ceil(NY/(double)threadsPerBlock.y);
	fft_blocks.z = 1;
}


MapStack::MapStack(TCudaGrid2D *Grid, int cpu_map_num)
{

	this->Grid = Grid;

	this->cpu_map_num = cpu_map_num;

	// set values
	map_stack_ctr = 0;
	Nb_array_RAM = 4;

	// initialize device map stack
	hipMalloc((void **) &Dev_ChiX_stack, 4*Grid->sizeNReal);
	hipMalloc((void **) &Dev_ChiY_stack, 4*Grid->sizeNReal);

	Host_ChiX_stack_RAM_0 = new double[cpu_map_num * 4*Grid->sizeNReal];
	Host_ChiY_stack_RAM_0 = new double[cpu_map_num * 4*Grid->sizeNReal];
	Host_ChiX_stack_RAM_1 = new double[cpu_map_num * 4*Grid->sizeNReal];
	Host_ChiY_stack_RAM_1 = new double[cpu_map_num * 4*Grid->sizeNReal];
	Host_ChiX_stack_RAM_2 = new double[cpu_map_num * 4*Grid->sizeNReal];
	Host_ChiY_stack_RAM_2 = new double[cpu_map_num * 4*Grid->sizeNReal];
	Host_ChiX_stack_RAM_3 = new double[cpu_map_num * 4*Grid->sizeNReal];
	Host_ChiY_stack_RAM_3 = new double[cpu_map_num * 4*Grid->sizeNReal];
}


// copy inserted map into last position at map stack, after 1/4 of all total maps, start using new array
void MapStack::copy_map_to_host(double *Dev_ChiX, double *Dev_ChiY) {
    switch(map_stack_ctr / cpu_map_num){
        case 0:
            hipMemcpy(&Host_ChiX_stack_RAM_0[(map_stack_ctr%cpu_map_num)*4*Grid->N], Dev_ChiX, 4*Grid->sizeNReal, hipMemcpyDeviceToHost);
            hipMemcpy(&Host_ChiY_stack_RAM_0[(map_stack_ctr%cpu_map_num)*4*Grid->N], Dev_ChiY, 4*Grid->sizeNReal, hipMemcpyDeviceToHost);
            break;
        case 1:
            hipMemcpy(&Host_ChiX_stack_RAM_1[(map_stack_ctr%cpu_map_num)*4*Grid->N], Dev_ChiX, 4*Grid->sizeNReal, hipMemcpyDeviceToHost);
            hipMemcpy(&Host_ChiY_stack_RAM_1[(map_stack_ctr%cpu_map_num)*4*Grid->N], Dev_ChiY, 4*Grid->sizeNReal, hipMemcpyDeviceToHost);
            break;
        case 2:
            hipMemcpy(&Host_ChiX_stack_RAM_2[(map_stack_ctr%cpu_map_num)*4*Grid->N], Dev_ChiX, 4*Grid->sizeNReal, hipMemcpyDeviceToHost);
            hipMemcpy(&Host_ChiY_stack_RAM_2[(map_stack_ctr%cpu_map_num)*4*Grid->N], Dev_ChiY, 4*Grid->sizeNReal, hipMemcpyDeviceToHost);
            break;

        case 3:
            hipMemcpy(&Host_ChiX_stack_RAM_3[(map_stack_ctr%cpu_map_num)*4*Grid->N], Dev_ChiX, 4*Grid->sizeNReal, hipMemcpyDeviceToHost);
            hipMemcpy(&Host_ChiY_stack_RAM_3[(map_stack_ctr%cpu_map_num)*4*Grid->N], Dev_ChiY, 4*Grid->sizeNReal, hipMemcpyDeviceToHost);
            break;
    }
    map_stack_ctr++;
}
// copy map to device, map_num decides which map to take out
void MapStack::copy_map_to_device(int map_num) {
	switch (map_num / cpu_map_num) {
		case 0: {
			hipMemcpy(Dev_ChiX_stack, &Host_ChiX_stack_RAM_0[(map_num%cpu_map_num)*Grid->N*4], 4*Grid->sizeNReal, hipMemcpyHostToDevice);
			hipMemcpy(Dev_ChiY_stack, &Host_ChiY_stack_RAM_0[(map_num%cpu_map_num)*Grid->N*4], 4*Grid->sizeNReal, hipMemcpyHostToDevice);
			break;
		}
		case 1: {
			hipMemcpy(Dev_ChiX_stack, &Host_ChiX_stack_RAM_1[(map_num%cpu_map_num)*Grid->N*4], 4*Grid->sizeNReal, hipMemcpyHostToDevice);
			hipMemcpy(Dev_ChiY_stack, &Host_ChiY_stack_RAM_1[(map_num%cpu_map_num)*Grid->N*4], 4*Grid->sizeNReal, hipMemcpyHostToDevice);
			break;
		}
		case 2: {
			hipMemcpy(Dev_ChiX_stack, &Host_ChiX_stack_RAM_2[(map_num%cpu_map_num)*Grid->N*4], 4*Grid->sizeNReal, hipMemcpyHostToDevice);
			hipMemcpy(Dev_ChiY_stack, &Host_ChiY_stack_RAM_2[(map_num%cpu_map_num)*Grid->N*4], 4*Grid->sizeNReal, hipMemcpyHostToDevice);
			break;
		}
		case 3: {
			hipMemcpy(Dev_ChiX_stack, &Host_ChiX_stack_RAM_3[(map_num%cpu_map_num)*Grid->N*4], 4*Grid->sizeNReal, hipMemcpyHostToDevice);
			hipMemcpy(Dev_ChiY_stack, &Host_ChiY_stack_RAM_3[(map_num%cpu_map_num)*Grid->N*4], 4*Grid->sizeNReal, hipMemcpyHostToDevice);
			break;
		}
	}
}

// free those resources
void MapStack::free_res() {
	delete [] Host_ChiX_stack_RAM_0;
	delete [] Host_ChiY_stack_RAM_0;
	delete [] Host_ChiX_stack_RAM_1;
	delete [] Host_ChiY_stack_RAM_1;
	delete [] Host_ChiX_stack_RAM_2;
	delete [] Host_ChiY_stack_RAM_2;
	delete [] Host_ChiX_stack_RAM_3;
	delete [] Host_ChiY_stack_RAM_3;
	hipFree(Dev_ChiX_stack);
	hipFree(Dev_ChiY_stack);
}
